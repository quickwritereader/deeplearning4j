#include "hip/hip_runtime.h"
/*
 *  ******************************************************************************
 *  *
 *  *
 *  * This program and the accompanying materials are made available under the
 *  * terms of the Apache License, Version 2.0 which is available at
 *  * https://www.apache.org/licenses/LICENSE-2.0.
 *  *
 *  * See the NOTICE file distributed with this work for additional
 *  * information regarding copyright ownership.
 *  * Unless required by applicable law or agreed to in writing, software
 *  * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  * License for the specific language governing permissions and limitations
 *  * under the License.
 *  *
 *  * SPDX-License-Identifier: Apache-2.0
 *  *****************************************************************************
 */
/* Copyright 2016 The TensorFlow Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

//
//  @author George A. Shulinok <sgazeos@gmail.com>
//
#include <ops/declarable/helpers/image_resize.h>
#include <exceptions/cuda_exception.h>
#include <array/NDArrayFactory.h>

namespace sd {
namespace ops {
namespace helpers {


////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// computeInterpolationWeights kernel
//      outSize - output length
//      inSize - input size
//      scale - input scale
//      interporationData - result
//
    template <class Scaler>
    static SD_KERNEL void computeInterpolationWeights(sd::LongType outSize,
                                              sd::LongType inSize,
                                              double scale,
                                              sd::LongType channels,
                                              BilinearInterpolationData* interpolationData) {
        interpolationData[outSize].bottomIndex = 0;
        interpolationData[outSize].topIndex = 0;
        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;
        Scaler scaler;
        for (sd::LongType i = outSize - tid; i >= 0; i -= step) {
            double in = scaler(i, scale);
//            interpolationData[i].bottomIndex = static_cast<sd::LongType>(in);
//            interpolationData[i].topIndex = sd::math::sd_min(interpolationData[i].bottomIndex + 1, inSize - 1);
//            interpolationData[i].interpolarValue = in - interpolationData[i].bottomIndex;
            double const in_f = sd::math::p_floor<double>(in);
            double const in_c = sd::math::p_ceil<double>(in);
            interpolationData[i].bottomIndex = sd::math::sd_max(static_cast<sd::LongType>(in_f), (sd::LongType)0LL);//static_cast<sd::LongType>(in);
            interpolationData[i].topIndex = sd::math::sd_min(static_cast<sd::LongType>(in_c), inSize - 1);
            interpolationData[i].interpolarValue = in - in_f;

            if (channels) {
                math::atomics::sd_atomicMul(&interpolationData[i].bottomIndex, channels);
                math::atomics::sd_atomicMul(&interpolationData[i].topIndex, channels);
            }
        }
    }
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// resize image with bilinear interpolation algorithm
//
    static void resizeImage(sd::LaunchContext* context, NDArray const* images, sd::LongType batchSize, sd::LongType inHeight, sd::LongType inWidth, sd::LongType outHeight,
                     sd::LongType outWidth, sd::LongType channels,
                     BilinearInterpolationData* xs_,
                     BilinearInterpolationData* ys_,
                     NDArray* output);

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// resize image with bilinear interpolation algorithm kernel
//
    template <typename T, typename Z>
    static SD_KERNEL void resizeImageKernel(T const* input, sd::LongType const* inputShape, Z* outputYptr,
            sd::LongType const* outputShape, sd::LongType batchSize, sd::LongType outWidth, sd::LongType outHeight, sd::LongType channels,
            sd::LongType inRowSize, sd::LongType outRowSize, sd::LongType inBatchNumValues,
            BilinearInterpolationData* xs_, BilinearInterpolationData* ys_) {

        for (auto batch = blockIdx.x; batch < batchSize; batch += gridDim.x ) { // blockIdx.x as batch index
            auto pX = input + batch * inBatchNumValues;
            for (sd::LongType y = threadIdx.x; y < outHeight; y += blockDim.x) {
                const T* ys_input_lower_ptr = pX + ys_[y].bottomIndex * inRowSize;
                const T* ys_input_upper_ptr = pX + ys_[y].topIndex * inRowSize;
                double yVal = ys_[y].interpolarValue;
                auto pZ = outputYptr + (batch * outHeight + y) * outRowSize;
                for (sd::LongType x = 0; x < outWidth; x++) {
                    auto xsBottom = xs_[x].bottomIndex;
                    auto xsTop = xs_[x].topIndex;
                    auto xVal = xs_[x].interpolarValue;
                    // process interpolation for all channels
                    for (int c = 0; c < channels; c++) {
                        Z topLeft(ys_input_lower_ptr[xsBottom + c]);
                        Z topRight(ys_input_lower_ptr[xsTop + c]);
                        Z bottomLeft(ys_input_upper_ptr[xsBottom + c]);
                        Z bottomRight(ys_input_upper_ptr[xsTop + c]);
                        Z top = topLeft + (topRight - topLeft) * xVal;
                        Z bottom = bottomLeft + (bottomRight - bottomLeft) * xVal;
                        Z resVal = Z(top + (bottom - top) * yVal);
                        pZ[x * channels + c] = resVal;
                    }
                }
            }
        }
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// resize image with
    template <typename T, typename F>
    static void resizeImage_(sd::LaunchContext* context, NDArray const* images, sd::LongType batchSize, sd::LongType inHeight, sd::LongType inWidth, sd::LongType outHeight,
                     sd::LongType outWidth, sd::LongType channels,
                     BilinearInterpolationData* xs_,
                     BilinearInterpolationData* ys_,
                     NDArray* output) {
        sd::LongType inRowSize = inWidth * channels;
        sd::LongType inBatchNumValues = inHeight * inRowSize;
        sd::LongType outRowSize = outWidth * channels;
        auto stream = context->getCudaStream();
        T const* pInput = images->getDataBuffer()->specialAsT<T>(); //reinterpret_cast<T const *>(images->specialBuffer()); // this works only with 'c' direction
        F* pOutput = output->dataBuffer()->specialAsT<F>();//reinterpret_cast<F *>(output->specialBuffer());
        dim3 batchSizeBlock(batchSize, 1, 1);
        dim3 pictureBlock(outHeight, outWidth, channels);
        resizeImageKernel<T,F><<<256, 256, 256, *stream>>>(pInput, images->specialShapeInfo(), pOutput,
                output->specialShapeInfo(), batchSize, outWidth, outHeight, channels, inRowSize, outRowSize,
                inBatchNumValues, xs_, ys_);

        auto err = hipStreamSynchronize(*stream);
        if (err != 0) {
            throw cuda_exception::build("helpers::resizeImage_: Cannot synchronize kernel execution", err);
        }
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template <typename T, typename F>
    static sd::Status resizeBilinearFunctor_(sd::LaunchContext* context, NDArray const* images, int const width,
            int const height, bool const alignCorners, bool const halfPixelCenter, NDArray* output) {
        const sd::LongType batchSize = images->sizeAt(0);
        const sd::LongType inHeight = images->sizeAt(1);
        const sd::LongType inWidth = images->sizeAt(2);
        const sd::LongType channels = images->sizeAt(3);

        const sd::LongType outHeight = output->sizeAt(1);
        const sd::LongType outWidth = output->sizeAt(2);

        // Handle no-op resizes efficiently.
        if (outHeight == inHeight && outWidth == inWidth) {
            output->assign(images);
            return sd::Status::OK;
        }

        float heightScale = ImageResizerState::calculateResizeScale(inHeight, outHeight, alignCorners);
        float widthScale = ImageResizerState::calculateResizeScale(inWidth, outWidth, alignCorners);

        BilinearInterpolationData* xs_;// = xs.data();
        BilinearInterpolationData* ys_;// = xs.data();

        hipError_t err = hipMalloc(&xs_, sizeof(BilinearInterpolationData) * (outWidth + 1));
        if (err != 0) {
            throw cuda_exception::build("helpers::resize_image: Cannot allocate memory for vertical parts rectangulars", err);
        }

        err = hipMalloc(&ys_, sizeof(BilinearInterpolationData) * (outHeight + 1));
        if (err != 0) {
            throw cuda_exception::build("helpers::resize_image: Cannot allocate memory for horizontal parts rectangulars", err);
        }
        auto stream = context->getCudaStream();
        // Compute the cached interpolation weights on the x and y dimensions.
        if (halfPixelCenter) {
            computeInterpolationWeights <
            HalfPixelScaler ><<<256, 512, 512, *stream>>>(outHeight, inHeight, heightScale, 0, ys_);
            computeInterpolationWeights <
            HalfPixelScaler ><<<256, 512, 512, *stream>>>(outWidth, inWidth, widthScale, channels, xs_);
        }
        else {
            computeInterpolationWeights <
            LegacyScaler ><<<256, 512, 512, *stream>>>(outHeight, inHeight, heightScale, 0, ys_);
            computeInterpolationWeights <
            LegacyScaler ><<<256, 512, 512, *stream>>>(outWidth, inWidth, widthScale, channels, xs_);
        }

        NDArray::prepareSpecialUse({output}, {images});
        resizeImage_<T,F>(context, images, batchSize, inHeight, inWidth, outHeight, outWidth, channels, xs_, ys_, output);
        err = hipStreamSynchronize(*stream);
        NDArray::registerSpecialUse({output}, {images});

        err = hipFree(xs_);
        if (err != 0) {
            throw cuda_exception::build("helpers::resize_image: Cannot deallocate memory for vertical parts rectangulars", err);
        }

        err = hipFree(ys_);
        if (err != 0) {
            throw cuda_exception::build("helpers::resize_image: Cannot deallocate memory for horizontical parts rectangulars", err);
        }

        return sd::Status::OK;
    }


    typedef float (*MODE_FUNC) (float);

    SD_DEVICE MODE_FUNC mode_functions[4] = { sd::math::p_floor<float>, 
                                               sd::math::p_round_prefer_floor<float>,
                                               sd::math::p_round_prefer_ceil<float>,
                                               sd::math::p_ceil<float> };

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// resize by interpolation nearest neighbor algorithm kernel
//
    template <typename T, typename Scaler>
    static SD_KERNEL void resizeNeighborKernel(T const* input, sd::LongType const* inputShape, T* output, sd::LongType const* outputShape,
            sd::LongType batchSize, sd::LongType inWidth, sd::LongType inHeight, sd::LongType outWidth, sd::LongType outHeight, sd::LongType channels, double widthScale, double heightScale, NearestMode nearestMode) {

                constexpr bool halfPixelCenter = std::is_same<Scaler, HalfPixelScaler>::value || std::is_same<Scaler, HalfPixelScalerNN>::value;
        MODE_FUNC modeFunc;
        switch (nearestMode)
        {
            case NearestMode::FLOOR :
                modeFunc = mode_functions[0];
                break;
            case NearestMode::ROUND_PREFER_FLOOR :
                modeFunc = mode_functions[1];
                break;
            case NearestMode::ROUND_PREFER_CEIL :
                modeFunc = mode_functions[2];
                break;
            case NearestMode::CEIL :
                modeFunc = mode_functions[3];
                break;
            default:
                modeFunc = mode_functions[0] ;
        }
        Scaler scaler;
        // if(threadIdx.x==0){


        // }
        //for (int b = blockIdx.x; b < batchSize; b += gridDim.x)
        if (blockIdx.x < batchSize)
        {
            auto b = blockIdx.x;
            for (int y = threadIdx.x; y < outHeight; y += blockDim.x) {
                auto posY = static_cast<sd::LongType>(modeFunc(scaler(y, heightScale)));
                sd::LongType inY = sd::math::sd_min(posY, inHeight - 1);
                if (halfPixelCenter) {
                    inY = sd::math::sd_max(0LL, inY);
                }

                for (int x = threadIdx.y; x < outWidth; x += blockDim.y) {
                    auto posX = static_cast<sd::LongType>(modeFunc(scaler(x, widthScale)));
                    sd::LongType inX = sd::math::sd_min(posX, inWidth - 1);
                    if (halfPixelCenter) {
                        inX = sd::math::sd_max(0LL, inX);
                    }

                    auto start = blockIdx.z * blockDim.z + threadIdx.z;
                    auto step = blockDim.z * gridDim.z;

                    for (sd::LongType e = start; e < channels; e += step) {
                        sd::LongType posX[] = {b, inY, inX, e};
                        sd::LongType posZ[] = {b, y, x, e};
                        auto xIndex = shape::getOffset(inputShape, posX);
                        auto zIndex = shape::getOffset(outputShape, posZ);
                        output[zIndex] = input[xIndex];
                    }
                }
            }
        }

    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// resizeNeighborFunctor - main algorithm by nearest neighbor
//
    template<typename T>
    sd::Status resizeNeighborFunctor_(sd::LaunchContext* context, NDArray const *images, int const width, int const height, CoordinateTransformationMode coorMode, NearestMode nearestMode, bool alignCorner, NDArray *output) {
        const sd::LongType batchSize = images->sizeAt(0);
        const sd::LongType inHeight = images->sizeAt(1);
        const sd::LongType inWidth = images->sizeAt(2);
        const sd::LongType channels = images->sizeAt(3);

        const sd::LongType outHeight = output->sizeAt(1);
        const sd::LongType outWidth = output->sizeAt(2);

        // Handle no-op resizes efficiently.
        if (outHeight == inHeight && outWidth == inWidth) {
            output->assign(images);
            return sd::Status::OK;
        }

        float heightScale = ImageResizerState::calculateResizeScale(inHeight, outHeight, alignCorner);
        float widthScale = ImageResizerState::calculateResizeScale(inWidth, outWidth, alignCorner);

        auto imagesBuffer = images->getDataBuffer()->specialAsT<T>();//reinterpret_cast<T const*>(images->specialBuffer());
        auto outputBuffer = output->dataBuffer()->specialAsT<T>();//reinterpret_cast<T*>(output->specialBuffer());
        auto stream = context->getCudaStream();

        NDArray::prepareSpecialUse({output}, {images});
        switch (coorMode)
        {
        case ASYMMETRIC:
            resizeNeighborKernel<T, LegacyScaler><<<batchSize, outHeight * outWidth, 512, *stream>>>(imagesBuffer, images->specialShapeInfo(), outputBuffer, output->specialShapeInfo(),
                batchSize, inWidth, inHeight, outWidth, outHeight, channels, widthScale, heightScale, nearestMode);
            break;
        case HALF_PIXEL:
            resizeNeighborKernel<T, HalfPixelScaler><<<batchSize, outHeight * outWidth, 512, *stream>>>(imagesBuffer, images->specialShapeInfo(), outputBuffer, output->specialShapeInfo(),
                batchSize, inWidth, inHeight, outWidth, outHeight, channels, widthScale, heightScale, nearestMode);
            break;
        case HALF_PIXEL_NN:
            resizeNeighborKernel<T, HalfPixelScalerNN><<<batchSize, outHeight * outWidth, 512, *stream>>>(imagesBuffer, images->specialShapeInfo(), outputBuffer, output->specialShapeInfo(),
                batchSize, inWidth, inHeight, outWidth, outHeight, channels, widthScale, heightScale, nearestMode);
            break;
        default:
            resizeNeighborKernel<T, HalfPixelScaler><<<batchSize, outHeight * outWidth, 512, *stream>>>(imagesBuffer, images->specialShapeInfo(), outputBuffer, output->specialShapeInfo(),
                batchSize, inWidth, inHeight, outWidth, outHeight, channels, widthScale, heightScale, nearestMode);
            break;
        };

        NDArray::registerSpecialUse({output}, {images});

        return sd::Status::OK;
    }
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// resizeImage - resize bilinear algorithm caller
//
    void resizeImage(sd::LaunchContext* context, NDArray const* images, sd::LongType batchSize, sd::LongType inHeight,
            sd::LongType inWidth, sd::LongType outHeight, sd::LongType outWidth, sd::LongType channels, BilinearInterpolationData* xs_,
            BilinearInterpolationData* ys_, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(images->dataType(), output->dataType(),
                resizeImage_, (context, images, batchSize, inHeight, inWidth, outHeight, outWidth, channels,
                        xs_, ys_, output), SD_NUMERIC_TYPES, SD_FLOAT_TYPES);
    }

    BUILD_DOUBLE_TEMPLATE(template void resizeImage_,(sd::LaunchContext* context, NDArray const* images,
            sd::LongType batchSize, sd::LongType inHeight, sd::LongType inWidth, sd::LongType outHeight, sd::LongType outWidth,
            sd::LongType channels, BilinearInterpolationData* xs_, BilinearInterpolationData* ys_, NDArray* output),
            SD_NUMERIC_TYPES, SD_FLOAT_TYPES);

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    sd::Status resizeBilinearFunctor(sd::LaunchContext* context, NDArray const* images, int width, int height,
            bool const alignCorners, bool const halfPixelCenter, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(images->dataType(), output->dataType(), return resizeBilinearFunctor_, (context, images,
                width, height, alignCorners, halfPixelCenter, output), SD_NUMERIC_TYPES, SD_FLOAT_TYPES);
    }
//    BUILD_SINGLE_TEMPLATE(template sd::Status resizeBilinearFunctor_, (sd::LaunchContext* context,
//            NDArray const* images, int const width, int const height, bool const alignCorners,
//            bool const halfPixelCenter, NDArray* output), SD_COMMON_TYPES);

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
sd::Status resizeNeighborFunctor(sd::LaunchContext * context, NDArray const *images, int const width, int const height,
    CoordinateTransformationMode coorMode, NearestMode nearestMode, bool alignCorner, NDArray *output) {
    BUILD_SINGLE_SELECTOR(images->dataType(), return resizeNeighborFunctor_, (context, images, width, height, coorMode, nearestMode, alignCorner, output), SD_COMMON_TYPES);
}
//    BUILD_SINGLE_TEMPLATE(template sd::Status Logger::logStatusMsg, (sd::LaunchContext* context, NDArray const* images,
//            int width, int height, bool const alignCorners, bool const halfPixelCenter, NDArray* output), SD_COMMON_TYPES);

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Bicubic interpolation
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


    static SD_KERNEL void initCoefTableKernel(const float a, float* table, sd::LongType tableSize) {
        KeysCubicKernelFunc<float> kernel(a);
        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;
        for (int i = start; i <= tableSize; i += step) {
            float x = i * 1.0 / tableSize;
            table[i * 2] = kernel.calc_less1pt0(x);
            x += 1.0;
            table[i * 2 + 1] = kernel.calc_less2pt0(x);
        }
    }


    float* initCoeffsTable(const double a, hipStream_t* stream) {
        // Allocate and initialize coefficients table using Bicubic
        // convolution algorithm.
        // https://en.wikipedia.org/wiki/Bicubic_interpolation
        float* coeffs_table; // = new float[(kTableSize + 1) * 2];
        auto err = hipMalloc(&coeffs_table, sizeof(float) * ((kTableSize + 1) * 2));
        if (err != 0) {
            throw cuda_exception::build("helpers::initCoeffsTable: Cannot allocate memory for vertical parts rectangulars", err);
        }


        initCoefTableKernel<<<128,128,128, *stream>>>(static_cast<float>(a), coeffs_table, kTableSize);
        err = hipStreamSynchronize(*stream);
        if (err != 0) {
            throw cuda_exception::build("helpers::initCoeffsTable: Cannot syncronize kernel", err);
        }

        return coeffs_table;
    }
//    SD_HOST_DEVICE const  float* getCoeffsTable(const bool use_keys_cubic) {
//            // Static so that we initialize it on first use
//            if (use_keys_cubic) {
//                // http://ieeexplore.ieee.org/document/1163711/
//                // R. G. Keys. Cubic convolution interpolation for digital image
//                // processing. IEEE Transactions on Acoustics, Speech, and Signal
//                // Processing, 29(6):1153–1160, 1981.
//                //static const float* coeffs_table = initCoeffsTable(-0.5f, stream);
//                return sCoeffsTableHalf;
//            } else {
//                //static const float* coeffs_table = initCoeffsTable(-0.75f, stream);
//                return sCoeffsTableThreeFourth;
//            }
//        }


    static SD_KERNEL void accumulateChannelsKernel(WeightsAndIndices* pXWais, sd::LongType outWidth, sd::LongType channels) {
        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (auto x = start; x < outWidth; x += step) {
            pXWais[x]._index0 *= channels;
            pXWais[x]._index1 *= channels;
            pXWais[x]._index2 *= channels;
            pXWais[x]._index3 *= channels;
        }
    }

    template<typename Scaler>
    static SD_KERNEL void advanceWeightsAndIndicesKernel(float const* cacheTable, CachedInterpolationCalculator* calc, WeightsAndIndices* pXWais, sd::LongType inWidth, float widthScale,
            sd::LongType outWidth, sd::LongType channels, bool exclude_outside) {
        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (auto x = start; x < outWidth; x += step) {
            getWeightsAndIndices<Scaler>(cacheTable, widthScale, x, inWidth, pXWais + x, exclude_outside);
        }
        __syncthreads();
        if(start == 0){
            //update only in one thread
            for (auto i = 0; i < outWidth; i++) {
                pXWais[i]._advance = calc->Advance(pXWais[i]._index0, pXWais[i]._index1, pXWais[i]._index2,
                    pXWais[i]._index3);
            }
        }
 
    }
    // resizerState and xWais are device allocated
    template<typename Scaler>
    static void computeXWeightsAndIndices(float const* coeffsTable, const ImageResizerState& resizerState,
                                          WeightsAndIndices* pXWais, bool exclude_outside) {

        auto stream = resizerState.stream;
        auto outWidth = resizerState.outWidth;
        CachedInterpolationCalculator calc; // = new CachedInterpolationCalculator;
        CachedInterpolationCalculator* pCalcD;
        auto err = hipMalloc(&pCalcD, sizeof(CachedInterpolationCalculator));
        if (err != 0) {
            cuda_exception::build("helpers::computeXWeightsAndIndices: Cannot allocated device memory for interpolate calculator", err);
        }
        err = hipMemcpyAsync(pCalcD, &calc, sizeof(CachedInterpolationCalculator), hipMemcpyHostToDevice, *stream);
        if (err != 0) {
            cuda_exception::build("helpers::computeXWeightsAndIndices: Cannot set up device memory for interpolate calculator", err);
        }

        advanceWeightsAndIndicesKernel<Scaler><<<128, 128, 128, *stream>>>(coeffsTable, pCalcD, pXWais, resizerState.inWidth, resizerState.widthScale, outWidth, resizerState.channels, exclude_outside);
        err = hipFree(pCalcD);
        if (err != 0) {
            cuda_exception::build("helpers::computeXWeightsAndIndices: Cannot deallocated device memory for interpolate calculator", err);
        }
        err = hipStreamSynchronize(*stream);
        if (err != 0) {
            cuda_exception::build("helpers::computeXWeightsAndIndices: Cannot synchronize stream after advance weights and indicers", err);
        }
        // Scale the values so they can be used as offsets into buffers.
        accumulateChannelsKernel<<<128, 128, 512, *stream>>>(pXWais, outWidth, resizerState.wStride);
        err = hipStreamSynchronize(*stream);
        if (err != 0) {
            cuda_exception::build("helpers::computeXWeightsAndIndices: Cannot synchronize stream after accumulate channels", err);
        }

    }


    template <typename T, typename Scaler>
    static SD_KERNEL void bicubicInterpolateWithCachingKernel(float const* cachedTable, T const* inputPtr, ImageResizerState* pResizerState, WeightsAndIndices* xWais, bool exclude_outside, float* outputPtr) {
//        auto numChannels = pResizerState->channels;
        const auto batchStride = pResizerState->bStride;
        const auto hStride = pResizerState->hStride;
        const auto cStride = pResizerState->cStride;
        for (sd::LongType b = blockIdx.x; b < pResizerState->batchSize; b += gridDim.x) {
            auto pInput = inputPtr + b * batchStride;

            float* cachedValue;
            for (sd::LongType y = threadIdx.x; y < pResizerState->outHeight; y += blockDim.x) {
                if (threadIdx.x == 0) {
                    extern __shared__ char sharedChar[];
                    cachedValue = reinterpret_cast<float*>(sharedChar);
                }
                auto pos = (b * pResizerState->outHeight + y) * pResizerState->outWidth * pResizerState->channels;
                auto pOutput = &outputPtr[pos];
                struct WeightsAndIndices yWai;

                getWeightsAndIndices<Scaler>(cachedTable, pResizerState->heightScale, y, pResizerState->inHeight, &yWai, exclude_outside);

                // Make pointers represent offsets of data in inputBPtr.
                const T* y_ptr_0 = pInput + yWai._index0 * hStride;
                const T* y_ptr_1 = pInput + yWai._index1 * hStride;
                const T* y_ptr_2 = pInput + yWai._index2 * hStride;
                const T* y_ptr_3 = pInput + yWai._index3 * hStride;

                if (pResizerState->channels == 100) {
                    // Manually unroll case of 3 channels.
                    float cached_value_0[4] = {0};
                    float cached_value_1[4] = {0};
                    float cached_value_2[4] = {0};
                    for (sd::LongType x = 0; x < pResizerState->outWidth; ++x) {
                        const WeightsAndIndices& xWai = xWais[x];
                        // Shift values in cached_value_* to fill first '_advance' values.
                        switch (xWai._advance) {
                            case 3:
                                cached_value_0[0] = cached_value_0[1];
                                cached_value_0[1] = cached_value_0[2];
                                cached_value_0[2] = cached_value_0[3];
                                cached_value_1[0] = cached_value_1[1];
                                cached_value_1[1] = cached_value_1[2];
                                cached_value_1[2] = cached_value_1[3];
                                cached_value_2[0] = cached_value_2[1];
                                cached_value_2[1] = cached_value_2[2];
                                cached_value_2[2] = cached_value_2[3];
                                break;
                            case 2:
                                cached_value_0[0] = cached_value_0[2];
                                cached_value_0[1] = cached_value_0[3];
                                cached_value_1[0] = cached_value_1[2];
                                cached_value_1[1] = cached_value_1[3];
                                cached_value_2[0] = cached_value_2[2];
                                cached_value_2[1] = cached_value_2[3];
                                break;
                            case 1: {
                                cached_value_0[0] = cached_value_0[3];
                                cached_value_1[0] = cached_value_1[3];
                                cached_value_2[0] = cached_value_2[3];
                                break;
                            }
                        }

                        // Set the remaining '4-_advance' values by computing.
                        switch (xWai._advance) {
                            case 0:
                                cached_value_0[0] = computeYInterpolation(0,          0, yWai, y_ptr_0, y_ptr_1, y_ptr_2, y_ptr_3, xWai);
                                cached_value_1[0] = computeYInterpolation(0,    cStride, yWai, y_ptr_0, y_ptr_1, y_ptr_2, y_ptr_3, xWai);
                                cached_value_2[0] = computeYInterpolation(0, 2 *cStride, yWai, y_ptr_0, y_ptr_1, y_ptr_2, y_ptr_3, xWai);
                            case 1:
                                cached_value_0[1] = computeYInterpolation(1,          0, yWai, y_ptr_0, y_ptr_1, y_ptr_2, y_ptr_3, xWai);
                                cached_value_1[1] = computeYInterpolation(1,    cStride, yWai, y_ptr_0, y_ptr_1, y_ptr_2, y_ptr_3, xWai);
                                cached_value_2[1] = computeYInterpolation(1, 2 *cStride, yWai, y_ptr_0, y_ptr_1, y_ptr_2, y_ptr_3, xWai);
                            case 2:
                                cached_value_0[2] = computeYInterpolation(2,          0, yWai, y_ptr_0, y_ptr_1, y_ptr_2, y_ptr_3, xWai);
                                cached_value_1[2] = computeYInterpolation(2,    cStride, yWai, y_ptr_0, y_ptr_1, y_ptr_2, y_ptr_3, xWai);
                                cached_value_2[2] = computeYInterpolation(2, 2 *cStride, yWai, y_ptr_0, y_ptr_1, y_ptr_2, y_ptr_3, xWai);
                            case 3:
                                cached_value_0[3] = computeYInterpolation(3,          0, yWai, y_ptr_0, y_ptr_1, y_ptr_2, y_ptr_3, xWai);
                                cached_value_1[3] = computeYInterpolation(3,    cStride, yWai, y_ptr_0, y_ptr_1, y_ptr_2, y_ptr_3, xWai);
                                cached_value_2[3] = computeYInterpolation(3, 2 *cStride, yWai, y_ptr_0, y_ptr_1, y_ptr_2, y_ptr_3, xWai);
                        //        break;
                        }
                        pOutput[x * pResizerState->channels + 0] = compute(cached_value_0, xWai._weight0, xWai._weight1,
                                        xWai._weight2, xWai._weight3);
                        pOutput[x * pResizerState->channels + 1] = compute(cached_value_1, xWai._weight0, xWai._weight1,
                                        xWai._weight2, xWai._weight3);
                        pOutput[x * pResizerState->channels + 2] = compute(cached_value_2, xWai._weight0, xWai._weight1,
                                        xWai._weight2, xWai._weight3);
                    }
                } else {
                    for (sd::LongType x = 0; x < pResizerState->outWidth; ++x) {
                        const WeightsAndIndices& xWai = xWais[x];
                        // Shift values in cachedValue to fill first '_advance' values.
                        switch (xWai._advance) {
                            case 3:
                                for (sd::LongType c = 0; c < pResizerState->channels; ++c) {
                                    cachedValue[4 * c + 0] = cachedValue[4 * c + 1];
                                    cachedValue[4 * c + 1] = cachedValue[4 * c + 2];
                                    cachedValue[4 * c + 2] = cachedValue[4 * c + 3];
                                }
                                break;
                            case 2:
                                for (sd::LongType c = 0; c < pResizerState->channels; ++c) {
                                    cachedValue[4 * c + 0] = cachedValue[4 * c + 2];
                                    cachedValue[4 * c + 1] = cachedValue[4 * c + 3];
                                }
                                break;
                            case 1: {
                                for (sd::LongType c = 0; c < pResizerState->channels; ++c) {
                                    cachedValue[4 * c + 0] = cachedValue[4 * c + 3];
                                }
                                break;
                            }
                        }

                        // Set the remaining '4-_advance' values by computing.
                        switch (xWai._advance) {
                            case 0:
                                for (sd::LongType c = 0; c < pResizerState->channels; ++c) {
                                    cachedValue[4 * c + 0] = computeYInterpolation(0, c * cStride, yWai, y_ptr_0, y_ptr_1, y_ptr_2, y_ptr_3, xWai);
                                }
                            case 1:
                                for (sd::LongType c = 0; c < pResizerState->channels; ++c) {
                                    cachedValue[4 * c + 1] = computeYInterpolation(1, c * cStride, yWai, y_ptr_0, y_ptr_1, y_ptr_2, y_ptr_3, xWai);
                                }
                            case 2:
                                for (sd::LongType c = 0; c < pResizerState->channels; ++c) {
                                    cachedValue[4 * c + 2] = computeYInterpolation(2, c * cStride, yWai, y_ptr_0, y_ptr_1, y_ptr_2, y_ptr_3, xWai);
                                }
                            case 3:
                                for (sd::LongType c = 0; c < pResizerState->channels; ++c) {
                                    cachedValue[4 * c + 3] = computeYInterpolation(3, c * cStride, yWai, y_ptr_0, y_ptr_1, y_ptr_2, y_ptr_3, xWai);
                                }
                               // break;
                        }
                        for (sd::LongType c = 0; c < pResizerState->channels; ++c) {
                            auto res= compute(&cachedValue[4 * c], xWai._weight0, xWai._weight1, xWai._weight2, xWai._weight3);
                            pOutput[x * pResizerState->channels + c] =res;
                            
                        }
                    }
                }
            }
        }

    }


    template <typename T, typename Scaler>
    static void
    bicubicInterpolateWithCaching(NDArray const* image, const ImageResizerState& resizerState, const double coefficient, bool exclude_outside, NDArray* output){
        const auto numChannels = resizerState.channels;
        auto stream = resizerState.stream; //output->getContext()->getCudaStream();
        ImageResizerState* resizerStateD;
        auto err = hipMalloc(&resizerStateD, sizeof(ImageResizerState));
        if (err != 0) {
            throw cuda_exception::build("helpers::bicubicInterpolateWithCaching: Cannot allocate memory for resizerState", err);
        }
        err = hipMemcpyAsync(resizerStateD, &resizerState, sizeof(ImageResizerState), hipMemcpyHostToDevice, *stream);
        if (err != 0) {
            throw cuda_exception::build("helpers::bicubicInterpolateWithCaching: Cannot set up memory for resizerState", err);
        }

//        float* cachedValue = nullptr;
//        size_t cachedSize = sizeof(float) * (numChannels == 3 ? 0 : 4 * numChannels);
//        if (cachedSize) {
//            err = hipMalloc(reinterpret_cast<void**>(&cachedValue), cachedSize);
//            if (err != 0) {
//                throw cuda_exception::build(
//                        "helpers::bicubicInterpolateWithCaching: Cannot allocate memory for cached values", err);
//            }
//            err = hipMemset(cachedValue, 0, cachedSize);
//            if (err != 0) {
//                throw cuda_exception::build(
//                        "helpers::bicubicInterpolateWithCaching: Cannot set up memory for cached values", err);
//            }
//        }

        WeightsAndIndices* xWais; //(resizerState.outWidth);
        err = hipMalloc(&xWais, sizeof(WeightsAndIndices) * resizerState.outWidth);
        if (err != 0) {
            throw cuda_exception::build("helpers::bicubicInterpolateWithCaching: Cannot allocate memory for weights and indices", err);
        }

        auto coeffsTable = initCoeffsTable(coefficient, stream); //halfPixelCenters?initCoeffsTable(-0.5, stream): initCoeffsTable(-0.75, stream);
        if (err != 0) {
            throw cuda_exception::build("helpers::bicubicInterpolateWithCaching: computeXWeigtsAndInidces finished with error", err);
        }
        computeXWeightsAndIndices<Scaler>(coeffsTable, resizerState, xWais, exclude_outside);
        err = hipStreamQuery(*stream);
        if (err != 0) {
            throw cuda_exception::build("helpers::bicubicInterpolateWithCaching: computeXWeigtsAndInidces finished with error", err);
        }

        const T* pInput = image->getDataBuffer()->specialAsT<T>();
        float* pOutput = output->dataBuffer()->specialAsT<float>(); //_data.data();
        bicubicInterpolateWithCachingKernel<T, Scaler><<<128, 1, 512, *stream>>>(coeffsTable, pInput,
                resizerStateD, xWais, exclude_outside, pOutput);
        err = hipStreamSynchronize(*stream);
        if (err != 0) {
            throw cuda_exception::build("helpers::bicubicInterpolateWithCaching: Kernels finished with error", err);
        }

        err = hipFree(resizerStateD);
        if (err != 0) {
            throw cuda_exception::build("helpers::bicubicInterpolateWithCaching: Cannot deallocate memory for resizerState", err);
        }
//        if (cachedSize)
//        err = hipFree(cachedValue);
//        if (err != 0) {
//            throw cuda_exception::build("helpers::bicubicInterpolateWithCaching: Cannot deallocate memory for cached values", err);
//        }

        err = hipFree(xWais);
        if (err != 0) {
            throw cuda_exception::build("helpers::bicubicInterpolateWithCaching: Cannot deallocate memory for weights and indices", err);
        }

        err = hipFree(coeffsTable);
        if (err != 0) {
            throw cuda_exception::build("helpers::bicubicInterpolateWithCaching: Cannot deallocate memory for coefficients table", err);
        }

    }
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template <typename T>
    sd::Status resizeBicubicFunctor_(sd::LaunchContext * context, NDArray const* image, int width, int height,
                              bool preserveAspectRatio, bool antialias, NDArray* output) {
        return sd::Status::OK;
    }

    sd::Status resizeBicubicFunctor(sd::LaunchContext * context, NDArray const* image, int width, int height,
                             bool preserveAspectRatio, bool antialias, NDArray* output) {
        BUILD_SINGLE_SELECTOR(image->dataType(), return resizeBicubicFunctor_, (context, image,
                width, height, preserveAspectRatio, antialias, output), SD_NUMERIC_TYPES);
    }
    BUILD_SINGLE_TEMPLATE(template sd::Status resizeBicubicFunctor_, (sd::LaunchContext * context, NDArray const* image, int width, int height,
            bool preserveAspectRatio, bool antialias, NDArray* output), SD_NUMERIC_TYPES);
// ------------------------------------------------------------------------------------------------------------------ //

    static SD_KERNEL void fillInterpolationCache(CachedInterpolation* xCached, sd::LongType cacheLen, sd::LongType inWidth, float widthScale) {
        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto increment = blockDim.x * gridDim.x;

        for (auto x = start; x < cacheLen; x += increment) {
            auto& xCache = xCached[x];
            const float inX = x * widthScale;
            const float inX1 = (x + 1) * widthScale;

            sd::LongType v = math::sd_floor<float, sd::LongType>(inX);
            xCache.start = v;
            xCache.startScale = v < inX ? (v + 1 > inX1 ? widthScale : v + 1 - inX) : (v + 1 > inX1 ? inX1 - v : 1.f);
            v = math::sd_ceil<float, sd::LongType>(inX1);
            xCache.end = v--;
            xCache.endMinusOneScale = v < inX ? (v + 1 > inX1 ? widthScale : v + 1 - inX) : (v + 1 > inX1 ? inX1 - v : 1.f);
            xCache.needsBounding = bound(xCache.start, inWidth) != xCache.start || bound(xCache.end - 1, inWidth) != (xCache.end - 1);
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //

    template <typename T>
    static SD_KERNEL void resizeAreaKernel(ImageResizerState const* pSt, CachedInterpolation const* caches, float scale,
            T const* inputPtr, sd::LongType const* inputShape, float* outputPtr, sd::LongType const* outputShape, ScaleCache<T>* cachePool) { //batch * outWidth * outHeight

        for (auto batch = blockIdx.x; batch < pSt->batchSize; batch += gridDim.x) {
            for (auto y = threadIdx.x; y < pSt->outHeight; y += blockDim.x) {
                const float inY = y * pSt->heightScale;
                const float inY1 = (y + 1) * pSt->heightScale;
                // The start and end height indices of all the cells that could
                // contribute to the target cell.
                const sd::LongType yStart = math::sd_floor<float, sd::LongType>(inY);
                const sd::LongType yEnd = math::sd_ceil<float, sd::LongType>(inY1);
                auto scalesDim = yEnd - yStart;
                auto yScaleCache = cachePool + (batch * pSt->outHeight + y) * pSt->outWidth;

                //auto startPtr = sharedPtr + y * scalesDim * sizeof(float);
                //float* yScales = yScalesShare + y * sizeof(float) * scalesDim;//reinterpret_cast<float*>(startPtr); //shared + y * scalesDim * y + scalesDim * sizeof(T const *) [scalesDim];
                //T const** yPtrs = yPtrsShare + y * sizeof(T const*) * scalesDim; //[scalesDim];
                //yPtrs = reinterpret_cast<T const**>(sharedBuf);
                float* output = outputPtr + (batch * pSt->outHeight  +  y)  * pSt->channels * pSt->outWidth;
                //int k = 0;
                for (sd::LongType i = yStart, k = 0; i < yEnd; ++i, ++k) {
                    float scaleY;
                    if (i < inY) {
                        scaleY = (i + 1 > inY1 ? pSt->heightScale : i + 1 - inY);
                    } else {
                        scaleY = (i + 1 > inY1 ? inY1 - i : 1.0);
                    }
                    yScaleCache[k].yScale = scaleY;
                    yScaleCache[k].yPtr = inputPtr + (batch * pSt->bStride + bound(i, pSt->inHeight) * pSt->hStride);
                }

                if (pSt->channels == 3) {
                    for (sd::LongType x = 0; x < pSt->outWidth; ++x) {
                        const CachedInterpolation& xCache = caches[x];
                        computePatchSumOf3Channels<T>(scale, *pSt, yScaleCache, scalesDim, xCache, output);
                        output += pSt->channels;
                    }
                } else {
                    for (sd::LongType x = 0; x < pSt->outWidth; ++x) {
                        const CachedInterpolation &xCache = caches[x];
                        computePatchSum<T>(scale, *pSt, yScaleCache, scalesDim, xCache, output);
                        output += pSt->channels;
                    }
                }
            }
        }
    }

    template <typename T>
    static void resizeArea(hipStream_t* stream, ImageResizerState const& st, CachedInterpolation* cache,
            NDArray const* input, NDArray* output) {

        T const* inputPtr = reinterpret_cast<T const*>(input->specialBuffer());
//        float* yScales;
//        T const** yPtrs;
        float scale = 1.f / (st.heightScale * st.widthScale);
        auto outputPtr = reinterpret_cast<float*>(output->specialBuffer()); // output is always float. TO DO: provide another float types also with  template <typename X, typename Z> declaration
        ImageResizerState* pSt;
        auto err = hipMalloc(&pSt, sizeof(ImageResizerState));
        if (err != 0) {
            throw cuda_exception::build("helpers::resizeArea: Cannot allocate memory for ImageResizerState", err);
        }

        err = hipMemcpyAsync(pSt, &st, sizeof(ImageResizerState), hipMemcpyHostToDevice, *stream);
        if (err != 0) {
            throw cuda_exception::build("helpers::resizeArea: Cannot copy to device memory", err);
        }
        ScaleCache<T>* cachePool;
        auto cachePoolSize = sizeof(ScaleCache<T>) * st.batchSize * st.outWidth * st.outHeight;
        err = hipMalloc(&cachePool, cachePoolSize);
        if (err != 0) {
            throw cuda_exception::build("helpers::resizeArea: Cannot allocate memory for cache", err);
        }
        resizeAreaKernel<T><<<128, 128, 2048, *stream>>>(pSt, cache, scale, inputPtr, input->specialShapeInfo(), outputPtr,
                output->specialShapeInfo(), cachePool);
        err = hipStreamSynchronize(*stream);
        if (err != 0) {
            throw cuda_exception::build("helpers::resizeArea: An error occured with kernel running", err);
        }
        err = hipFree(cachePool);
        if (err != 0) {
            throw cuda_exception::build("helpers::resizeArea: Cannot deallocate memory for cache", err);
        }
        err = hipFree(pSt);
        if (err != 0) {
            throw cuda_exception::build("helpers::resizeArea: Cannot deallocate memory for ImageResizeState", err);
        }
    }
// ------------------------------------------------------------------------------------------------------------------ //
    template <typename T>
    sd::Status resizeAreaFunctor_(sd::LaunchContext* context, NDArray const* image, int const width, int const height,
                              bool const alignCorners, NDArray* output) {

        ImageResizerState st(alignCorners, false); // Create resize info
        auto res = st.validateAndCalculateOutputSize(image, width, height);
        auto stream = context->getCudaStream();
        if (sd::Status::OK == res) {
            CachedInterpolation* xCached;
            //(st.outWidth);
            auto err = hipMalloc(&xCached, sizeof(CachedInterpolation) * st.outWidth);
            if (err != 0) {
                throw cuda_exception::build("helpers::resizeAreaFunctor_: Cannot allocate memory for cached interpolations", err);
            }
            NDArray::prepareSpecialUse({output}, {image});
            fillInterpolationCache<<<128, 128, 256, *stream>>>(xCached, st.outWidth, st.inWidth, st.widthScale);
            resizeArea<T>(stream, st, xCached, image, output);
            err = hipStreamSynchronize(*stream);
            if (err != 0) {
                throw cuda_exception::build("helpers::resizeAreaFunctor_: Error occured when kernel was running", err);
            }
            err = hipFree(xCached);
            if (err != 0) {
                throw cuda_exception::build("helpers::resizeAreaFunctor_: Cannot deallocate memory for cached interpolations", err);
            }
            NDArray::registerSpecialUse({output}, {image});
        }

        return res;
    }
    sd::Status resizeAreaFunctor(sd::LaunchContext * context, NDArray const* image, int const width, int const height,
                              bool const alignCorners, NDArray* output) {
        BUILD_SINGLE_SELECTOR(image->dataType(), return resizeAreaFunctor_, (context, image, width, height, alignCorners, output), SD_NUMERIC_TYPES);
    }

// ------------------------------------------------------------------------------------------------------------------ //
// simplified bicubic resize without antialiasing
//
    template <typename T>
    sd::Status resizeBicubicFunctorA_(sd::LaunchContext * context, NDArray const* image, int const width, int const height,
                            bool const alignCorners, CoordinateTransformationMode coorMode, bool exclude_outside, double coefficient, NDArray* output) {
        ImageResizerState st(alignCorners, coorMode == HALF_PIXEL, context->getCudaStream()); // align_corners, half_pixel_align
        NDArray::prepareSpecialUse({output}, {image});
        sd::Status res = st.validateAndCreateOutput(image, width, height);
        if (res == sd::Status::OK){
            switch (coorMode)
            {
            case ASYMMETRIC:
                bicubicInterpolateWithCaching<T, LegacyScaler>(image, st, coefficient, exclude_outside, output);
                break;
            case HALF_PIXEL:
                bicubicInterpolateWithCaching<T, HalfPixelScaler>(image, st, coefficient, exclude_outside, output);
                break;
            case HALF_PIXEL_NN:
                bicubicInterpolateWithCaching<T, HalfPixelScalerNN>(image, st, coefficient, exclude_outside, output);
                break;
            default:
                break;
            }
        }
        NDArray::registerSpecialUse({output}, {image});
        return res;
    }
    sd::Status resizeBicubicFunctorA(sd::LaunchContext * context, NDArray const* image, int const width, int const height,
                            bool const alignCorners, CoordinateTransformationMode coorMode, bool exclude_outside, double coefficient, NDArray* output) {
        BUILD_SINGLE_SELECTOR(image->dataType(), return resizeBicubicFunctorA_, (context, image, width, height, alignCorners, coorMode, exclude_outside, coefficient, output), SD_NUMERIC_TYPES);
    }
// ------------------------------------------------------------------------------------------------------------------ //
    sd::Status resizeImagesFunctor(sd::LaunchContext * context, NDArray const* image, int const width, int const height,
        ImageResizeMethods method, bool alignCorners, NDArray* output) {
        switch (method) {
        case kResizeBilinear:
            return resizeBilinearFunctor(context, image, width, height, alignCorners, false, output);
        case kResizeNearest:
            return resizeNeighborFunctor(context, image, width, height, CoordinateTransformationMode::ASYMMETRIC, 
            alignCorners ? NearestMode::ROUND_PREFER_CEIL : NearestMode::FLOOR, alignCorners, output);
        case kResizeBicubic:
            return resizeBicubicFunctor(context, image, width, height, alignCorners, false, output);
        case kResizeArea:
            return resizeAreaFunctor(context, image, width, height, alignCorners, output);
        default:
            throw std::runtime_error("helper::resizeImagesFunctor: Wrong resize method.");
    }
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // --------------------------------------------------------------------------------------------------------------- //
    // Crop and Resize helper implementation
    // -------------------------------------------------------------------------------------------------------------- //
    // cropAndResize kernel   type of input(images) and output should be the same
    //
    template <typename T, typename Z, typename I>
    static SD_KERNEL void cropAndResizeKernel(T const *images, sd::LongType const* imagesShape, Z const* boxes, sd::LongType const* boxesShape,
            I const* indices, sd::LongType const* indexShape, I const* cropSize, sd::LongType const* cropShape, int method,
            double extrapolationVal, T* output, sd::LongType const* outputShape, int numBoxes, int cropHeight, int cropWidth,
            int batchSize, int imageHeight, int imageWidth, int depth) {

        for (int b = blockIdx.x; b < numBoxes; b += gridDim.x)
        {
            sd::LongType x1Pos[] = {b, 1};
            sd::LongType y1Pos[] = {b, 0};
            sd::LongType y2Pos[] = {b, 2};
            sd::LongType x2Pos[] = {b, 3};
            Z y1 = boxes[shape::getOffset(boxesShape, y1Pos)];//->t<T>(b, 0)];
            Z x1 = boxes[shape::getOffset(boxesShape, x1Pos)];
            Z y2 = boxes[shape::getOffset(boxesShape, y2Pos)];
            Z x2 = boxes[shape::getOffset(boxesShape, x2Pos)];

            int bIn = indices[b];
            if (bIn >= batchSize) {
                continue;
            }

            Z heightScale = (cropHeight > 1) ? (y2 - y1) * (imageHeight - 1) / Z(cropHeight - 1) : Z(0);
            Z widthScale = (cropWidth > 1) ? (x2 - x1) * (imageWidth - 1) / Z(cropWidth - 1) : Z(0);

            for (int y = threadIdx.x; y < cropHeight; y += blockDim.x) {
                const float inY = (cropHeight > 1)
                                  ? y1 * (imageHeight - 1) + y * heightScale
                                  : 0.5 * (y1 + y2) * (imageHeight - 1);
                if (inY < 0 || inY > imageHeight - 1) {
                    for (int x = threadIdx.y; x < cropWidth; x += blockDim.y) {
                        auto start = blockIdx.z * blockDim.x + threadIdx.z;
                        auto step = blockDim.z * gridDim.z;
                        for (int d = start; d < depth; d += step) {
                            sd::LongType zPos[] = {b, y, x, d};
                            auto zIndex = shape::getOffset(outputShape, zPos);
                            output[zIndex] = (Z)extrapolationVal;
                            //crops->p(b, y, x, d, extrapolationVal);
                        }
                    }
                    continue;
                }

                if (method == 0 /* bilinear */) {
                    const int topYIndex = sd::math::p_floor(inY);
                    const int bottomYIndex = sd::math::p_ceil(inY);
                    const float y_lerp = inY - topYIndex;

                    for (int x = 0; x < cropWidth; ++x) {
                        const float in_x = (cropWidth > 1)
                                           ? x1 * (imageWidth - 1) + x * widthScale
                                           : 0.5 * (x1 + x2) * (imageWidth - 1);
                        if (in_x < 0 || in_x > imageWidth - 1) {
                            auto start = blockIdx.z * blockDim.x + threadIdx.z;
                            auto step = blockDim.z * gridDim.z;
                            for (int d = start; d < depth; d += step) {
                                sd::LongType zPos[] = {b, y, x, d};
                                auto zIndex = shape::getOffset(outputShape, zPos);
                                output[zIndex] = (Z)extrapolationVal;
//                                crops->p(b, y, x, d, extrapolationVal);
                            }
                            continue;
                        }
                        int left_x_index = math::p_floor(in_x);
                        int right_x_index = math::p_ceil(in_x);
                        T x_lerp = in_x - left_x_index;

                        auto start = blockIdx.z * blockDim.x + threadIdx.z;
                        auto step = blockDim.z * gridDim.z;
                        for (int d = start; d < depth; d += step) {
                            sd::LongType topLeftPos[] = {bIn, topYIndex, left_x_index, d};
                            sd::LongType topRightPos[] = {bIn, topYIndex, right_x_index, d};
                            sd::LongType bottomLeftPos[] = {bIn, bottomYIndex, left_x_index, d};
                            sd::LongType bottomRightPos[] = {bIn, bottomYIndex, right_x_index, d};
                            const T topLeft(images[shape::getOffset(imagesShape, topLeftPos)]); //->e<float>(bIn, topYIndex, left_x_index, d));
                            const T topRight(images[shape::getOffset(imagesShape, topRightPos)]); //->e<float>(bIn, topYIndex, right_x_index, d));
                            const T bottomLeft(images[shape::getOffset(imagesShape, bottomLeftPos)]);//->e<float>(bIn, bottomYIndex, left_x_index, d));
                            const T bottomRight(images[shape::getOffset(imagesShape, bottomRightPos)]); //->e<float>(bIn, bottomYIndex, right_x_index, d));
                            const T top = topLeft + (topRight - topLeft) * x_lerp;
                            const T bottom = bottomLeft + (bottomRight - bottomLeft) * x_lerp;
                            sd::LongType zPos[] = {b, y, x, d};
                            auto zIndex = shape::getOffset(outputShape, zPos);
                            output[zIndex] = Z(top + (bottom - top) * y_lerp);
                        }
                    }
                } else {  // method is "nearest neighbor"
                    for (int x = 0; x < cropWidth; ++x) {
                        const float inX = (cropWidth > 1)
                                          ? x1 * (imageWidth - 1) + x * widthScale
                                          : 0.5 * (x1 + x2) * (imageWidth - 1);
                        if (inX < 0 || inX > imageWidth - 1) {
                            auto start = blockIdx.z * blockDim.x + threadIdx.z;
                            auto step = blockDim.z * gridDim.z;
                            for (int d = start; d < depth; d += step) {
                                sd::LongType zPos[] = {b, y, x, d};
                                auto zIndex = shape::getOffset(outputShape, zPos);
                                output[zIndex] = (Z)extrapolationVal;
                            }
                            continue;
                        }
                        const int closestXIndex = roundf(inX);
                        const int closestYIndex = roundf(inY);
                        auto start = blockIdx.z * blockDim.x + threadIdx.z;
                        auto step = blockDim.z * gridDim.z;
                        for (int d = start; d < depth; d += step) {
                            sd::LongType zPos[] = {b, y, x, d};
                            sd::LongType xPos[] = {bIn, closestYIndex, closestXIndex, d};
                            auto zIndex = shape::getOffset(outputShape, zPos);
                            auto xIndex = shape::getOffset(imagesShape, xPos);
                            output[zIndex] = images[xIndex];
                        }
                    }
                }
            }
        }

    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// cropAndResizeFunctor main algorithm
//      context - launch context
//      images - batch of images (4D tensor - [batch, width, height, pixels])
//      boxes - 2D tensor with boxes for crop
//      indices - 2D int tensor with indices of boxes to crop
//      cropSize - 2D int tensor with crop box sizes
//      method - (one of 0 - bilinear, 1 - nearest)
//      extrapolationVal - double value of extrapolation
//      crops - output (4D tensor - [batch, outWidth, outHeight, pixels])
//
    template <typename T, typename Z, typename I>
    void cropAndResizeFunctor_(sd::LaunchContext* context, NDArray const *images, NDArray const *boxes, NDArray const *indices,
                                      NDArray const *cropSize, int method, double extrapolationVal, NDArray *crops) {
        const int batchSize = images->sizeAt(0);
        const int imageHeight = images->sizeAt(1);
        const int imageWidth = images->sizeAt(2);

        const int numBoxes = crops->sizeAt(0);
        const int cropHeight = crops->sizeAt(1);
        const int cropWidth = crops->sizeAt(2);
        const int depth = crops->sizeAt(3);
        auto stream = context->getCudaStream();
        T const* imagesBuf = reinterpret_cast<T const*>(images->specialBuffer());
        Z const* boxesBuf = reinterpret_cast<Z const*>(boxes->specialBuffer());
        I const* indexBuf = reinterpret_cast<I const*>(indices->specialBuffer());
        I const* cropSizes = reinterpret_cast<I const*>(cropSize->specialBuffer());
        T* outBuf = reinterpret_cast<T*>(crops->specialBuffer());

        int threadsPerBlock = math::sd_max(imageHeight * imageWidth, cropHeight * cropWidth);
        if(threadsPerBlock > SD_MAX_NUM_THREADS/4)
            threadsPerBlock = SD_MAX_NUM_THREADS/4;

        NDArray::prepareSpecialUse({crops}, {images, boxes, indices, cropSize});
        cropAndResizeKernel<T,Z,I><<<batchSize, threadsPerBlock, 256, *stream>>>(imagesBuf, images->specialShapeInfo(), boxesBuf, boxes->specialShapeInfo(), indexBuf, indices->specialShapeInfo(),
                cropSizes, cropSize->specialShapeInfo(), method, extrapolationVal, outBuf, crops->specialShapeInfo(), numBoxes, cropHeight, cropWidth, batchSize, imageHeight, imageWidth, depth);
        NDArray::registerSpecialUse({crops}, {images, boxes, indices, cropSize});
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    void cropAndResizeFunctor(sd::LaunchContext * context, NDArray const *images, NDArray const *boxes, NDArray const *indices, NDArray const *cropSize, int method, double extrapolationVal, NDArray *crops) {
        BUILD_TRIPLE_SELECTOR(images->dataType(), boxes->dataType(), indices->dataType(), cropAndResizeFunctor_,
                              (context, images, boxes, indices, cropSize, method, extrapolationVal, crops), SD_NUMERIC_TYPES, SD_FLOAT_TYPES, SD_INTEGER_TYPES);
        //
    }
    BUILD_TRIPLE_TEMPLATE(template void cropAndResizeFunctor_,
                          (sd::LaunchContext * context, NDArray const* images, NDArray const* boxes, NDArray const* indices, NDArray const* cropSize, int method, double extrapolationVal, NDArray* crops),
                          SD_NUMERIC_TYPES, SD_FLOAT_TYPES, SD_INTEGER_TYPES);
}
}
}
