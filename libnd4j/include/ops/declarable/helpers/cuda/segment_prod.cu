#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//
#include <ops/declarable/helpers/segment.h>
#include <ops/declarable/helpers/segment_common.h>
#include <array/NDArrayFactory.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <helpers/PointersManager.h>
#include <helpers/ConstantTadHelper.h>

namespace sd {
namespace ops {
namespace helpers {
    // -------------------------------------------------------------------------------------------------------------- //
    // Segment Prod ops linear kernels
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static SD_KERNEL void segmentProdLinearKernel(void* input, sd::LongType const* inputShape, int* starts, int* lengths,
            sd::LongType numOfClasses, void* output, sd::LongType const* outputShape) {

        __shared__ sd::LongType xLen, zLen;
        __shared__ T* x;
        __shared__ T* z;

        if (threadIdx.x == 0) {
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);
        }
        __syncthreads();

        for(auto segment = blockIdx.x; segment < numOfClasses; segment += gridDim.x) {
            auto zIndex = shape::getIndexOffset(segment, outputShape);
            auto start = starts[segment];
            auto finish = start + lengths[segment];
            if (lengths[segment] == 0) {
                continue;
            }
            for (auto e = start + threadIdx.x; e < finish; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputShape);
                sd::math::atomics::sd_atomicMul(&z[segment], x[xIndex]);
            }
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static SD_KERNEL void unsortedSegmentProdLinearKernel(T* input, sd::LongType const* inputShape, I* indices, sd::LongType const* indicesShape, int* starts, int* lengths, sd::LongType numOfClasses, T* output, sd::LongType const* outputShape) {
        __shared__ sd::LongType xLen, zLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);
        }
        __syncthreads();
        auto start = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;
        for (auto idx = start; idx < xLen; idx += step) {
            auto xIndex = shape::getIndexOffset(idx, inputShape);
            auto yIndex = shape::getIndexOffset(idx, indicesShape);
            auto segment = indices[yIndex];
            auto zIndex = shape::getIndexOffset(segment, outputShape);
            if (lengths[segment] == 0) {
                continue;
            }
            sd::math::atomics::sd_atomicMul(&output[zIndex], input[xIndex]);
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    // SegmentProd kernel
    template <typename T, typename I>
    static SD_KERNEL void segmentProdTadKernel(void* inputBuf, sd::LongType const* inputShape, sd::LongType const* inputTads,
            sd::LongType const* inputTadOffsets, I* indices, int* starts, int* lengths, sd::LongType numOfClasses, void* outputBuf,
            sd::LongType const* outputShape, sd::LongType const* outputTads, sd::LongType const* outputTadOffsets) {

        __shared__ sd::LongType len, total;

        if (threadIdx.x == 0) {
            total = shape::sizeAt(inputShape, 0);
            len = shape::length(inputTads);
        }
        __syncthreads();

        for (auto idx = blockIdx.x; idx < total; idx += gridDim.x) {
            auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
            auto segment = indices[idx]; // / threadsPerSegment;
            auto z = reinterpret_cast<T *>(outputBuf) + outputTadOffsets[segment];
            auto start = starts[segment];
            auto finish = start + lengths[segment];
            if (lengths[segment] == 0) continue;
            for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputTads);
                auto zIndex = shape::getIndexOffset(e, outputTads);
                sd::math::atomics::sd_atomicMul(&z[zIndex], x[xIndex]);
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static void segmentProdFunctor_(sd::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        sd::LongType numClasses = indices->e<sd::LongType>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses}, context);
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses}, context);
        output->assign(1);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);

        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        fillUpSegments(indices, numClasses, classesRangesBegs, classesRangesLens);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());

        if (input->isVector()) {
            segmentProdLinearKernel<T,I><<<128, 256, 128, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            auto inputTads = packX.specialShapeInfo();
            auto inputTadOffsets = packX.specialOffsets();
            auto outputTads = packZ.specialShapeInfo();
            auto outputTadOffsets = packZ.specialOffsets();
            segmentProdTadKernel<T,I><<<128, 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //
    void segmentProdFunctor(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), segmentProdFunctor_, (context, input, indices, output), SD_NUMERIC_TYPES, SD_INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices});
    }

    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static void unsortedSegmentProdFunctor_(sd::LaunchContext* context, NDArray* input, NDArray* indices, sd::LongType numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses}, context);
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses}, context);
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(sd::BroadcastOpsTuple::Assign(), &row, &classes);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numOfClasses, indices->lengthOf(), numOfClasses * 32 + 32);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        fillUpSegments(indices, numOfClasses, classesRangesBegs, classesRangesLens);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        output->assign(1);

        if (input->isVector()) {
            unsortedSegmentProdLinearKernel<T,I><<<128, 256, 256, *stream>>>(
                    input->dataBuffer()->specialAsT<T>(), input->specialShapeInfo(),
                    indices->dataBuffer()->specialAsT<I>(), indices->specialShapeInfo(), begins, lengths, numOfClasses,
                    output->dataBuffer()->specialAsT<T>(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            auto inputTads = packX.specialShapeInfo();
            auto inputTadOffsets = packX.specialOffsets();
            auto outputTads = packZ.specialShapeInfo();
            auto outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            segmentProdTadKernel<T,I><<<128, 256, 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //
    void unsortedSegmentProdFunctor(sd::LaunchContext* context , NDArray* input, NDArray* indices, sd::LongType numOfClasses, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices});
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentProdFunctor_, (context, input, indices, numOfClasses, output),
                              SD_NUMERIC_TYPES, SD_INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices});
    }

    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static SD_KERNEL void segmentProdBPLinearKernel(void* inputBuf, sd::LongType const* inputShape, void* forwardOutput,
                                                     sd::LongType const* forwardShape, void* eps, sd::LongType const* epsShape, void* indicesBuf, sd::LongType const* indicesShape,
                                                     void* outputBuf, sd::LongType const* outputShape) {
        __shared__ T* x;
        __shared__ T* gradIn;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ sd::LongType xLen, gradLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            gradIn = reinterpret_cast<T*>(forwardOutput);
            gradOut = reinterpret_cast<T*>(eps);
            gradLen = shape::length(epsShape);
        }
        __syncthreads();

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = gridDim.x * blockDim.x;

        for (auto e = start; e < xLen; e += step) {

            auto zOffset = shape::getIndexOffset(e, outputShape);
            auto xOffset = shape::getIndexOffset(e, inputShape);
            auto yOffset = shape::getIndexOffset(e, indicesShape);
            auto classIndex = y[yOffset];
            auto gradOffsetI = shape::getIndexOffset(classIndex, forwardShape);
            auto gradOffsetO = shape::getIndexOffset(classIndex, epsShape);

            z[zOffset] = gradOut[gradOffsetO]  * gradIn[gradOffsetI] / x[xOffset];
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static SD_KERNEL void segmentProdBPTadKernel(void* inputBuf, sd::LongType const* inputShape, void* forwardOutput,
                                                  sd::LongType const* forwardShape, void* eps, sd::LongType const* epsShape, void* indicesBuf, sd::LongType const* indicesShape,
                                                  void* outputBuf, sd::LongType const* outputShape, sd::LongType const* inputTad,
                                                  sd::LongType const* inputOffsets, sd::LongType const* gradInTad, sd::LongType const* gradInOffsets,
                                                  sd::LongType const* gradOutTad, sd::LongType const* gradOutOffsets, sd::LongType const* outTad,
                                                  sd::LongType const* outOffsets) {
        __shared__ T* x;
        __shared__ T* gradIn;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ sd::LongType xLen, yLen, gradLen, currentLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            yLen = shape::length(indicesShape);
            gradOut = reinterpret_cast<T*>(eps);
            gradIn = reinterpret_cast<T*>(forwardOutput);
            gradLen = shape::length(epsShape);
            currentLen = shape::length(outTad);
        }
        __syncthreads();

        for (auto i = blockIdx.x; i < yLen; i += gridDim.x) {
            auto yIndex = shape::getIndexOffset(i, indicesShape);
            auto segment = y[yIndex];
            T* current = x + inputOffsets[i];
            T* currentOut = z + outOffsets[i];
            T* in = gradIn + gradInOffsets[segment];
            T* outGrad = gradOut + gradOutOffsets[segment];

            for (auto e = threadIdx.x; e < currentLen; e += blockDim.x) {
                currentOut[e] = outGrad[e] * in[e] / current[e];
            }
        }
    }

    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    sd::Status segmentProdFunctorBP_(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray tempRes(gradOut->ordering(), gradOut->getShapeAsVector(), DataTypeUtils::fromT<T>(), context);//->shapeInfo(), context);
        segmentProdFunctor_<T, I>(context, input, indices, &tempRes);
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        if (input->isVector()) {
            sd::LongType loopSize = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<sd::LongType>(loop_size - 1);
            segmentProdBPLinearKernel<T,I><<<gradOut->lengthOf(), loopSize, 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            auto packGradIn = sd::ConstantTadHelper::getInstance().tadForDimensions(tempRes.shapeInfo(), dimensions);
            auto packGradOut = sd::ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
            auto inputTads = packX.specialShapeInfo();
            auto inputTadOffsets = packX.specialOffsets();
            auto outputTads = packZ.specialShapeInfo();
            auto outputTadOffsets = packZ.specialOffsets();
            auto gradInTads = packGradIn.specialShapeInfo();
            auto gradInTadOffsets = packGradIn.specialOffsets();
            auto gradOutTads = packGradOut.specialShapeInfo();
            auto gradOutTadOffsets = packGradOut.specialOffsets();

            segmentProdBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                    inputTads, inputTadOffsets, gradInTads, gradInTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
        return sd::Status::OK;
    }

    // -------------------------------------------------------------------------------------------------------------- //

    sd::Status segmentProdFunctorBP(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return segmentProdFunctorBP_, (context, input,
                indices, gradOut, output), SD_FLOAT_TYPES, SD_INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
    }

    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static sd::Status unsortedSegmentProdFunctorBP_(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, sd::LongType numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();

        NDArray tempRes(gradOut->ordering(), gradOut->getShapeAsVector(), DataTypeUtils::fromT<T>(), context);//->shapeInfo(), context);
        unsortedSegmentProdFunctor_<T, I>(context, input, indices, numOfClasses, &tempRes);
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        if (input->isVector()) {
            sd::LongType loopSize = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<sd::LongType>(loop_size - 1);
            segmentProdBPLinearKernel<T,I><<<gradOut->lengthOf(), loopSize, 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            auto packGradIn = sd::ConstantTadHelper::getInstance().tadForDimensions(tempRes.shapeInfo(), dimensions);
            auto packGradOut = sd::ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
            auto inputTads = packX.specialShapeInfo();
            auto inputTadOffsets = packX.specialOffsets();
            auto outputTads = packZ.specialShapeInfo();
            auto outputTadOffsets = packZ.specialOffsets();
            auto gradInTads = packGradIn.specialShapeInfo();
            auto gradInTadOffsets = packGradIn.specialOffsets();
            auto gradOutTads = packGradOut.specialShapeInfo();
            auto gradOutTadOffsets = packGradOut.specialOffsets();

            segmentProdBPTadKernel<T,I><<<indices->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                    inputTads, inputTadOffsets, gradInTads, gradInTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
        return sd::Status::OK;
    }

    // -------------------------------------------------------------------------------------------------------------- //
    sd::Status unsortedSegmentProdFunctorBP(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, sd::LongType numOfClasses, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentProdFunctorBP_, (context, input, indices, gradOut, numOfClasses, output), SD_FLOAT_TYPES, SD_INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
    }

    // -------------------------------------------------------------------------------------------------------------- //

}
}
}
