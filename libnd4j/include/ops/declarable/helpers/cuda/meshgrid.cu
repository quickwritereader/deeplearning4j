#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//

#include<ops/declarable/helpers/meshgrid.h>
#include <helpers/PointersManager.h>
#include <helpers/ConstantTadHelper.h>
#include <array/ResultSet.h>
#include <numeric>

namespace sd       {
namespace ops       {
namespace helpers {

    template <typename T>
    static SD_DEVICE void assign_(void *vx, sd::LongType *xShapeInfo, void *vz, sd::LongType *zShapeInfo) {
        auto x = reinterpret_cast<T*>(vx);
        auto z = reinterpret_cast<T*>(vz);

        auto tid = threadIdx.x + blockIdx.x * blockDim.x;

        auto xEws = shape::elementWiseStride(xShapeInfo);
        auto zEws = shape::elementWiseStride(zShapeInfo);

        auto xOrder = shape::order(xShapeInfo);
        auto zOrder = shape::order(zShapeInfo);

        __shared__ sd::LongType length;

        if (threadIdx.x == 0) {
            length = shape::length(xShapeInfo);
        }
        __syncthreads();

        if (xEws > 0 && zEws > 0 && xOrder == zOrder) {
            for (int i = threadIdx.x; i < length; i += blockDim.x) {
                z[i * zEws] = x[i * xEws];
            }
        } else {
            for (int i = threadIdx.x; i < length; i += blockDim.x) {
                auto xOffset = shape::getIndexOffset(i, xShapeInfo);
                auto zOffset = shape::getIndexOffset(i, zShapeInfo);

                z[zOffset] = x[xOffset];
            }
        }

    }

    template <typename T>
    static SD_KERNEL void meshgridKernel(int rank, void **outBuffers, sd::LongType **tadShapes, sd::LongType **tadOffsets, sd::LongType *numTads, void **inBuffers, sd::LongType **inShapes) {
        // for all arrays
        for (int i = blockIdx.x; i < rank; i += gridDim.x) {

            // for all tads in this array
            for(sd::LongType j = 0; j < numTads[i]; j++) {
                assign_<T>(inBuffers[i], inShapes[i], reinterpret_cast<T*>(outBuffers[i]) + tadOffsets[i][j], tadShapes[i]);
            }
            __syncthreads();
        }
    }

    template <typename T>
    static void meshgrid_(sd::LaunchContext * context, const std::vector<NDArray*>& inArrs, const std::vector<NDArray*>& outArrs, const bool swapFirst2Dims) {
        const int rank = inArrs.size();
        int inIndices[SD_MAX_RANK];
        std::iota(inIndices, inIndices + rank, 0);
        if(swapFirst2Dims && rank > 1) {
            inIndices[0] = 1;
            inIndices[1] = 0;
        }

        PointersManager pm(context, "meshgrid");
        std::vector<const void *> hInBuffers(rank);
        std::vector<void *> hOutBuffers(rank);
        std::vector<const sd::LongType *> hInShapes(rank);

        std::vector<const sd::LongType *> hOutTadShapes(rank);
        std::vector<const sd::LongType *> hOutTadOffsets(rank);

        std::vector<sd::LongType> hNumTads(rank);

        for(int i = 0; i < rank; ++i) {
            hInBuffers[i] = inArrs[i]->specialBuffer();
            hInShapes[i] = inArrs[i]->specialShapeInfo();

            hOutBuffers[i] = outArrs[i]->specialBuffer();


            auto pack = ConstantTadHelper::getInstance().tadForDimensions(outArrs[i]->shapeInfo(), {inIndices[i]});
            hOutTadShapes[i] = pack.specialShapeInfo();
            hOutTadOffsets[i] = pack.specialOffsets();
            hNumTads[i] = pack.numberOfTads();


            //auto list = outArrs[i]->allTensorsAlongDimension({inIndices[i]});
            //for(int j = 0; j < list->size(); ++j)
            //    list->at(j)->assign(inArrs[i]);

            //delete list;
        }

        auto dInBuffers = reinterpret_cast<void **>(pm.replicatePointer(hInBuffers.data(), hInBuffers.size() * sizeof(void *)));
        auto dOutBuffers = reinterpret_cast<void **>(pm.replicatePointer(hOutBuffers.data(), hOutBuffers.size() * sizeof(void *)));


        auto dInShapes = reinterpret_cast<sd::LongType **>(pm.replicatePointer(hInShapes.data(), hInShapes.size() * sizeof(sd::LongType *)));
        auto dOutTadShapes = reinterpret_cast<sd::LongType **>(pm.replicatePointer(hOutTadShapes.data(), hOutTadShapes.size() * sizeof(sd::LongType *)));
        auto dOutTadOffsets = reinterpret_cast<sd::LongType **>(pm.replicatePointer(hOutTadOffsets.data(), hOutTadOffsets.size() * sizeof(sd::LongType *)));

        auto dNumTads = reinterpret_cast<sd::LongType *>(pm.replicatePointer(hNumTads.data(), hNumTads.size() * sizeof(sd::LongType)));


        meshgridKernel<T><<<256, 256, 1024, *context->getCudaStream()>>>(rank, dOutBuffers, dOutTadShapes, dOutTadOffsets, dNumTads, dInBuffers, dInShapes);

        pm.synchronize();
    }

    //////////////////////////////////////////////////////////////////////////
    void meshgrid(sd::LaunchContext * context, const std::vector<NDArray*>& inArrs, const std::vector<NDArray*>& outArrs, const bool swapFirst2Dims) {

        BUILD_SINGLE_SELECTOR(inArrs.at(0)->dataType(), meshgrid_, (context, inArrs, outArrs, swapFirst2Dims), SD_NUMERIC_TYPES);

        for (auto v:outArrs)
            v->tickWriteDevice();
    }

}
}
}

