#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author raver119@gmail.com
//
#include <ops/declarable/helpers/top_k.h>
#include <helpers/MmulHelper.h>
#include <array/NDArrayFactory.h>

#include <helpers/ConstantTadHelper.h>
#include <helpers/ShapeUtils.h>
//#include <ops/declarable/generic/helpers/BroadcastHelper.h>
#include <hipsolver.h>
#include <exceptions/cuda_exception.h>

namespace sd {
namespace ops {
namespace helpers {

// ------------------------------------------------------------------------------------------------------------------ //
//  invert the second diagonal for lower diagonal matrix
    template<typename T>
    static SD_KERNEL void
    invertKernelLow(void *invertedBuf, const sd::LongType *invertedShape, const void *inputBuf, const sd::LongType *inputShape, sd::LongType n) {
        auto inverted = reinterpret_cast<T *>(invertedBuf);
        auto input = reinterpret_cast<const T*>(inputBuf);

        auto start = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (int i = start + 1; i < n; i += step) {
            sd::LongType pos[] = {i, i - 1};
            sd::LongType posX[] = {i, i};
            sd::LongType posY[] = {i - 1, i - 1};
            auto xIndex = shape::getOffset(inputShape, pos);
            auto dxIndex = shape::getOffset(inputShape, posX);
            auto dyIndex = shape::getOffset(inputShape, posY);
            auto zIndex = shape::getOffset(invertedShape, pos);
            // invert lower triangular matrix
            inverted[zIndex] = -input[xIndex] / (input[dxIndex] * input[dyIndex]);
//            math::atomics::sd_atomicAdd(&inverted[zIndex], - input[xIndex] * inverted[iIndex] / input[dIndex]);
        }
    }
// ------------------------------------------------------------------------------------------------------------------ //
// invert diagonal vals to upper diagonal matrix
    template<typename T>
    static SD_KERNEL void
    upvertKernel(void *invertedBuf, const sd::LongType *invertedShape, const void *inputBuf, const sd::LongType *inputShape, sd::LongType n) {
        auto inverted = reinterpret_cast<T *>(invertedBuf);
        auto input = reinterpret_cast<const T *>(inputBuf);

        auto start = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (int i = start; i < n; i += step) {
            sd::LongType pos[] = {i, i};
            auto xIndex = shape::getOffset(inputShape, pos);
            auto zIndex = shape::getOffset(invertedShape, pos);

            // invert diagonal elements
            inverted[zIndex] /= input[xIndex];
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
//  invert upper second diagonal
    template<typename T>
    static SD_KERNEL void
    upvertKernelUp(void *invertedBuf, const sd::LongType *invertedShape, const void *inputBuf, const sd::LongType *inputShape, sd::LongType n) {

        __shared__ T* inverted;
        __shared__ const T* input;
        if (threadIdx.x == 0) {
            inverted = reinterpret_cast<T *>(invertedBuf);
            input = reinterpret_cast<const T *>(inputBuf);
        }
        __syncthreads();

        auto start = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (int i = start; i < n - 1; i += step) {
            sd::LongType pos[] = {i, i + 1};
            sd::LongType posX[] = {i + 1, i + 1};
            auto xIndex = shape::getOffset(inputShape, pos);
            auto iIndex = shape::getOffset(invertedShape, posX);
            auto zIndex = shape::getOffset(invertedShape, pos);
            // invert upper matrix
            math::atomics::sd_atomicAdd(&inverted[zIndex], -input[xIndex] * inverted[iIndex]); // / input[yIndex]);
            //inputMatrix->t<T>(i, i + 1) * invertedMatrix->t<T>(i + 1, i + 1) / inputMatrix->t<T>(i, i)
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
    template<typename T>
    static SD_KERNEL void
    invertLowKernel(void *invertedBuf, const sd::LongType *invertedShape, const void *inputBuf, const sd::LongType *inputShape, sd::LongType n) {

        auto input = reinterpret_cast<const T *>(inputBuf);
        auto inverted = reinterpret_cast<T *>(invertedBuf);


        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = gridDim.x * blockDim.x;

        for (int i = tid + 2; i < n; i += step) {
            for (int j = i - 2; j >= 0; --j)
                for (int k = 0; k < i; k++) {
                    sd::LongType posZ[] = {i, j};
                    sd::LongType posY[] = {k, j};
                    sd::LongType posX[] = {i, k};
                    sd::LongType posD[] = {i, i};

                    auto xIndex = shape::getOffset(inputShape, posX);
                    auto yIndex = shape::getOffset(invertedShape, posY);
                    auto dIndex = shape::getOffset(inputShape, posD);
                    auto zIndex = shape::getOffset(invertedShape, posZ);
                    // invert non-diagonal elements
                    math::atomics::sd_atomicAdd(&inverted[zIndex], -inverted[yIndex] * input[xIndex] / input[dIndex]);
                }
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
// Invertion of upper triangular matrix non-diagonal elements when main and second diagonals already processed
    template<typename T>
    static SD_KERNEL void
    invertUpKernel(
            void *invertedBuf, const sd::LongType *invertedShape,
            const void *inputBuf, const sd::LongType *inputShape,
            sd::LongType n) {

        auto inverted = reinterpret_cast<T *>(invertedBuf);;
        auto input = reinterpret_cast<const T *>(inputBuf);

        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (int i = (int)n - tid - 2; i >= 0; i -= step) {
            for (int j = i + 2; j < (int)n; j++)
                for (int k = i; k < (int)n; k++) {
                    sd::LongType posZ[] = {i, j};
                    sd::LongType posY[] = {k, j};
                    sd::LongType posX[] = {i, k};
                    // inversion with Joardan Gauss transformation
                    auto xIndex = shape::getOffset(inputShape, posX);
                    auto yIndex = shape::getOffset(invertedShape, posY);
                    auto zIndex = shape::getOffset(invertedShape, posZ);
                    // invert upper non-diagonal elements
                    math::atomics::sd_atomicAdd(&inverted[zIndex], -inverted[yIndex] * input[xIndex]);
                }
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
// procedure to invert lower-triangular matrix.
// In current case lower triangular matrix has main diagonal with general values
//
    template<typename T>
    static void invertLowerMatrix_(LaunchContext *context, NDArray *inputMatrix, NDArray *invertedMatrix) {
        int n = inputMatrix->rows();
        invertedMatrix->setIdentity();

        if (inputMatrix->isIdentityMatrix()) return;

        auto stream = context->getCudaStream();

        // invert lower matrix
        // invert main diagonal
        upvertKernel<T><<<1, n, 512, *stream>>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(), inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
        // invert the second diagonal
        invertKernelLow<T><<<1, n, 512, *stream>>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(), inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
        // invert non-diagonal elements
        invertLowKernel<T><<<n, n, 512, *stream>>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(), inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
    }

// ------------------------------------------------------------------------------------------------------------------ //
// caller for invert lower matrix routine
    void invertLowerMatrix(LaunchContext *context, NDArray *inputMatrix, NDArray *invertedMatrix) {
        NDArray::prepareSpecialUse({invertedMatrix}, {inputMatrix});
        BUILD_SINGLE_SELECTOR(inputMatrix->dataType(), invertLowerMatrix_, (context, inputMatrix, invertedMatrix), SD_FLOAT_NATIVE);
        NDArray::registerSpecialUse({invertedMatrix}, {inputMatrix});
    }

// ------------------------------------------------------------------------------------------------------------------ //
// procedure to invert upper-triangular matrix.
// In current case upper triangular matrix has main diagonal with all ones on it.
    template<typename T>
    static void invertUpperMatrix_(LaunchContext *context, NDArray* inputMatrix, NDArray* invertedMatrix) {
        int n = inputMatrix->rows();
        invertedMatrix->setIdentity();
        auto stream = context->getCudaStream();
        if (inputMatrix->isIdentityMatrix()) { // the inverse for I is I
            return;
        }

        // invert upper matrix
        // invert the second diagonal
        upvertKernelUp<T><<<1, n, 512, *stream >>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(),
                inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);

        // invert other elements
        invertUpKernel<T><<<n, n, 512, *stream >>>(invertedMatrix->specialBuffer(), invertedMatrix->specialShapeInfo(),inputMatrix->specialBuffer(), inputMatrix->specialShapeInfo(), n);
    }

// ------------------------------------------------------------------------------------------------------------------ //
//  invertion of upper triangular matrix - runner routine
    void invertUpperMatrix(LaunchContext *context, NDArray *inputMatrix, NDArray *invertedMatrix) {
        NDArray::prepareSpecialUse({invertedMatrix}, {inputMatrix});
        BUILD_SINGLE_SELECTOR(invertedMatrix->dataType(), invertUpperMatrix_, (context, inputMatrix, invertedMatrix), SD_FLOAT_NATIVE);
        NDArray::prepareSpecialUse({invertedMatrix}, {inputMatrix});
    }

// ------------------------------------------------------------------------------------------------------------------ //
    // determinant kernel - accumulation product of all values on the main diagonal
    template<typename T>
    static SD_KERNEL void determinantKernel(T *compound, T *result, sd::LongType len) {
        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;
        for (auto i = start; i < len; i += step) {
            auto pos = i * len + i; //shape::getOffset(0, shape::shapeOf(shape), shape::stride(shape), di, 2);
            // multiply all diagonal elements
            math::atomics::sd_atomicMul(&result[0], compound[pos]);
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
    // determinant logarithm - accumulation sum of all logarithm values on the main diagonal. All in logarithic values
    // should be positive
    template<typename T>
    static SD_KERNEL void determinantLogKernel(T *compound, T *result, sd::LongType len) {
        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;
        for (auto i = start; i < len; i += step) {
            auto pos = i * len + i; //shape::getOffset(0, shape::shapeOf(shape), shape::stride(shape), di, 2);
            // sum logs of all diagonal elements
            math::atomics::sd_atomicAdd(result, math::sd_log<T,T>(math::sd_abs(compound[pos])));
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
    // kernel to copy matrix with given shape to compound tensor with given pos
    // output - a N-D tensor buffer with rank not less than 2, input - 2D square n x n matrix with n = rowLen
    template<typename T, typename F>
    static SD_KERNEL void
    fillMatrix(void *output, const sd::LongType *outShape, const void *input, const sd::LongType *inputShape, sd::LongType pos, sd::LongType rowLen) {
        __shared__ F *matrix;
        __shared__ const T *inputBuf;
        __shared__ sd::LongType inputLen;
        __shared__ sd::LongType n2;

        if (threadIdx.x == 0) {
            matrix = reinterpret_cast<F*>(output);
            inputBuf = reinterpret_cast<const T*>(input);
            inputLen = shape::length(inputShape);
            n2 = rowLen * rowLen;
        }
        __syncthreads();

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (int k = pos + start, j = start; j < n2; k += step, j += step) {
            auto xIndex = shape::getIndexOffset(k, inputShape);
            matrix[j] = (F) inputBuf[xIndex];
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
// same as above, but without type conversion
    template<typename T>
    static SD_KERNEL void
    returnMatrix(void *output, const sd::LongType *outputShape, const void *input, const sd::LongType *inputShape, sd::LongType pos, sd::LongType rowLen) {
        __shared__ sd::LongType outputLen;
        __shared__ sd::LongType n2;
        auto matrix = reinterpret_cast<const T *>(input);
        auto outputBuf = reinterpret_cast<T *>(output);

        if (threadIdx.x == 0) {

            outputLen = shape::length(inputShape);
            n2 = rowLen * rowLen;
        }
        __syncthreads();
        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (int k = pos + start, j = start; j < n2; k += step, j += step) {
            auto zIndex = shape::getIndexOffset(k, outputShape);
            outputBuf[zIndex] = matrix[j];
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
    // fill up permutaion matrix kernel. Permutation matrix filled with zeros and ones
    template<typename F>
    static SD_KERNEL void fillUpPermutation(void *output, const sd::LongType *shape, int *source, int rowNum) {
        F *permutation = reinterpret_cast<F *>(output);

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;
        for (auto i = start; i < rowNum; i += step) {
            int val = source[i] - 1;
            sd::LongType posF[] = {i, val};
            auto pos = shape::getOffset(shape, posF);
            permutation[pos] = F(1.f);
        }
    }

// ------------------------------------------------------------------------------------------------------------------ //
    // LUP decomposition runner - using CUBLAS SOLVER
    // if permutation is given, then using LUP decomposition, LU decomposition otherwise
    // L - lower triangular, U - upper triangular, P - permutation matricies
    // PA = LU
    //
    // input - A matrix nxn
    // compound - C matrix L + U - I, or main diagonal and lower - L matrix, from the 2nd diagonal - U matrix
    template<typename T, typename I>
    static void lup_(LaunchContext *context, NDArray *input, NDArray *compound, NDArray *permutation) {
        auto stream = context->getCudaStream();
        auto n = input->rows();
        std::lock_guard<std::mutex> lock(*LaunchContext::deviceMutex());

        hipsolverHandle_t* cusolverH = (hipsolverHandle_t*)context->getCusolverHandle(); //nullptr;
        // create solver handle
        hipsolverStatus_t status; //hipsolverDnCreate(&cusolverH);
//        if (HIPSOLVER_STATUS_SUCCESS != status) {
//            throw cuda_exception::build("Cannot create cuSolver handle", status);
//        }
        // set solver stream
        status = hipsolverSetStream(*cusolverH, *stream);
        if (HIPSOLVER_STATUS_SUCCESS != status) {
            throw cuda_exception::build("Cannot set up stream for cuda solver", status);
        }
        int lwork = 0;
        int *d_info = nullptr;
        // allocate memory for permutation vector
        auto err = hipMalloc((void **) &d_info, sizeof(int));
        if (err) {
            throw cuda_exception::build("helpers::lup_: Cannot allocate memory for solver info buffer", err);
        }

        DataType dtype = input->dataType();
        switch (dtype) { // there are two implementations with cublas for LUP decomposition - double and float

            case DataType::DOUBLE: {
                double *d_work = nullptr;
                // compute internal buffer size
                double *matrix = reinterpret_cast<double *>(input->specialBuffer());
                status = hipsolverDnDgetrf_bufferSize(
                        *cusolverH,
                        n,
                        n,
                        matrix,
                        n,
                        &lwork);
                if (HIPSOLVER_STATUS_SUCCESS != status) {
                    throw cuda_exception::build("helpers::lup_: Cannot create cuSolver handle", status);
                }

                err = hipMalloc((void **) &d_work, sizeof(float) * lwork);
                if (err) {
                    throw cuda_exception::build("helpers::lup_: Cannot allocate memory for solver data buffer",
                                                err);
                }

                if (permutation == nullptr) {
                    status = hipsolverDnDgetrf(
                            *cusolverH,
                            n,
                            n,
                            matrix,
                            n,
                            d_work,
                            nullptr,
                            d_info);

                    if (status != HIPSOLVER_STATUS_SUCCESS) {
                        throw cuda_exception::build("helpers::lup_: LU factorization is failed due ",
                                                    status);
                    }
                }
                else {
                    NDArray permutVector('c', {n}, sd::DataType::INT32, context);
                    int* permutationBuf = permutVector.dataBuffer()->specialAsT<int>();
                    status = hipsolverDnDgetrf(
                            *cusolverH,
                            n,
                            n,
                            matrix,
                            n,
                            d_work,
                            permutationBuf,
                            d_info);
                    if (status != HIPSOLVER_STATUS_SUCCESS) {
                        throw cuda_exception::build("helpers::lup_: LU factorization is failed due ",
                                                    status);
                    }

                    if (permutation->rankOf() == 2) {
                        fillUpPermutation<double> <<< n, n, 1024, *stream >>>
                                                                  (permutation->specialBuffer(), permutation->specialShapeInfo(), permutationBuf, n);
                    }
                    else {
                        permutVector.tickWriteDevice();
                        input->tickWriteDevice();
                        compound->assign(input);
                        permutation->assign(permutVector);
                    }
                }
                err = hipFree(d_work);
                if (err) {
                    throw cuda_exception::build("helpers::lup_: Cannot deallocate memory for solver data buffer",
                                                err);
                }
            }
                break;
            case DataType::FLOAT32: {
                float *matrix = reinterpret_cast<float*>(input->specialBuffer());
                float *d_work = nullptr;

                status = hipsolverDnSgetrf_bufferSize(
                        *cusolverH,
                        n,
                        n,
                        matrix,
                        n,
                        &lwork);
                if (HIPSOLVER_STATUS_SUCCESS != status) {
                    throw cuda_exception::build("helpers::lup_: Cannot create cuSolver handle", status);
                }

                err = hipMalloc((void **) &d_work, sizeof(float) * lwork);
                if (err) {
                    throw cuda_exception::build("helpers::lup_: Cannot allocate memory for solver data buffer",
                                                err);
                }

                if (permutation == nullptr)
                    status = hipsolverDnSgetrf(
                            *cusolverH,
                            n,
                            n,
                            matrix,
                            n,
                            d_work,
                            nullptr,
                            d_info);
                else {
                    NDArray permutVector('c', {n}, DataType::INT32, context);
                    int *permutationBuf = reinterpret_cast<int *>(permutVector.specialBuffer());
                    status = hipsolverDnSgetrf(
                            *cusolverH,
                            n,
                            n,
                            matrix,
                            n,
                            d_work,
                            permutationBuf,
                            d_info);
                    if (permutation->rankOf() == 2) {
                        fillUpPermutation<I> <<< n, n, 128, *stream >>>
                                                             (permutation->specialBuffer(), permutation->specialShapeInfo(), permutationBuf, n);
                        permutation->tickWriteDevice();
                    }
                    else {
                        input->tickWriteDevice();
                        compound->assign(input);
                        permutation->assign(permutVector);
                    }
                }
                err = hipFree(d_work);
                if (err) {
                    throw cuda_exception::build("helpers::lup_: Cannot deallocate memory for solver data buffer",
                                                err);
                }

            }
        }
        if (HIPSOLVER_STATUS_SUCCESS != status) {
            throw cuda_exception::build("helpers::lup_: Cannot make LU decomposition", status);
        }
        err = hipFree(d_info);
        if (err) {
            throw cuda_exception::build("helpers::lup_: Cannot deallocate memory for solver info buffer", err);
        }
//        hipsolverDnDestroy(cusolverH);
//        NDArray::registerSpecialUse({input}, {input});
        input->tickWriteDevice();
    }
// ------------------------------------------------------------------------------------------------------------------ //

    BUILD_DOUBLE_TEMPLATE(template void lup_,(LaunchContext * context, NDArray * input, NDArray * output, NDArray * permutation), SD_FLOAT_NATIVE, SD_INDEXING_TYPES);

    template <typename T>
    static SD_DEVICE void  swapRows(T* matrix, const sd::LongType* shape, sd::LongType theFirst, sd::LongType theSecond, sd::LongType n) {
        if (theFirst != theSecond) {
            for (auto i = 0; i < n; i++) {
                sd::LongType theFirstPos[] = {theFirst, i};
                sd::LongType theSecondPos[] = {theSecond, i};
                auto theFirstIndex = shape::getOffset(shape, theFirstPos, 0);
                auto theSecondIndex = shape::getOffset(shape, theSecondPos, 0);
                math::sd_swap(matrix[theFirstIndex], matrix[theSecondIndex]);
            }
        }
    }

    template <typename T>
    static SD_DEVICE void processColumns(sd::LongType currentRow, sd::LongType rowNum, T* compoundBuf, const sd::LongType* compoundShape) {
        sd::LongType xDiag[] = {currentRow, currentRow};
        auto diagIndex = shape::getOffset(compoundShape, xDiag, 0);
        for (auto j = currentRow + 1; j < rowNum; j++) {
            sd::LongType xRow[] = {j, currentRow};
            auto rowIndex = shape::getOffset(compoundShape, xRow, 0);
            compoundBuf[rowIndex] /= compoundBuf[diagIndex]; //output->t<T>(i, i);
            for (auto k = currentRow + 1; k < rowNum; k++) {
                sd::LongType yRow[] = {j, k};
                sd::LongType yCol[] = {currentRow, k};
                auto rowIndexY = shape::getOffset(compoundShape, yRow, 0);
                auto colIndex = shape::getOffset(compoundShape, yCol, 0);
                compoundBuf[rowIndexY] -= compoundBuf[rowIndex] * compoundBuf[colIndex];
            }
        }
    }

    template <typename T>
    SD_DEVICE sd::LongType argmaxCol(sd::LongType column, T* compoundBuffer, const sd::LongType* compoundShape) {
        auto rowNum = shape::sizeAt(compoundShape, 0);
        sd::LongType xInitial[] = {column, column};
        auto xInitialIndex = shape::getOffset(compoundShape, xInitial, 0);
        auto maxValue = T(0); //sd::math::sd_abs(compoundBuffer[xInitialIndex]);
        auto result = -1LL;

        for (auto rowCounter = column; rowCounter < rowNum; rowCounter++) {
            sd::LongType xPos[] = {rowCounter, column};
            auto xIndex = shape::getOffset(compoundShape, xPos, 0);
            if (sd::math::sd_abs(compoundBuffer[xIndex]) > maxValue) {
                maxValue = sd::math::sd_max(maxValue, sd::math::sd_abs(compoundBuffer[xIndex]));
                result = rowCounter;
            }
        }
        return result;
    }

        template <typename T, typename I>
    static SD_DEVICE int  luNN(T* matrix, const sd::LongType* shape, I* permutation, const sd::LongType* permuShape, sd::LongType n) {

        for (auto i = 0; i < n - 1; i++) {
            auto pivotIndex = argmaxCol(i, matrix, shape);
            if (pivotIndex < 0) {
                return -1;//throw std::runtime_error("helpers::luNN_: input matrix is singular.");
            }
            math::sd_swap(permutation[shape::getIndexOffset(i, permuShape)], permutation[shape::getIndexOffset(pivotIndex, permuShape)]);
            swapRows(matrix, shape, (sd::LongType)i, pivotIndex, n);

            processColumns(i, n, matrix, shape);
        }
        return 0;
    }

    template <typename T, typename I>
    static SD_KERNEL void luBatchedKernel(
            T* outputBuf, const sd::LongType* outputShape,
            I* permutations, const sd::LongType* permuShape,
            const sd::LongType* outputTadShape, const sd::LongType* outputTadOffsets,
            const sd::LongType* permuTadShape, const sd::LongType* permuTadOffsets,
            sd::LongType batchNum) {

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = blockDim.x * gridDim.x;

        for (auto b = start; b < batchNum; b += step) {
            T* matrix = outputBuf + outputTadOffsets[b];
            I* permutation = permutations + permuTadOffsets[b];

            if (0 != luNN(matrix, outputTadShape, permutation, permuTadShape, shape::length(permuTadShape))) break;
        }
    }

    template <typename T, typename I>
    static void lu_(LaunchContext * context, NDArray* input, NDArray* output, NDArray* permutationVectors) {
        auto n = input->sizeAt(-1);
        auto stream = context->getCudaStream();
        NDArray iota('c', {n}, permutationVectors->dataType(), context);// = NDArrayFactory::create(); // <int>('c', {n});
        iota.linspace(0); iota.syncToDevice();

        output->assign(input); // fill up output tensor with zeros
//        output->tickWriteDevice();
        permutationVectors->applyTrueBroadcast(sd::BroadcastOpsTuple::Assign(), iota, *permutationVectors, true, nullptr);
//        permutationVectors->tickWriteDevice();
        auto tads = ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), {-2, -1});
        auto permutaionTads = ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), {-1});
        auto batchNum = tads.numberOfTads();
        luBatchedKernel<T,I><<<batchNum, 256, 1024, *stream>>>(reinterpret_cast<T*>(output->platformBuffer()),
                output->specialShapeInfo(), reinterpret_cast<I*>(permutationVectors->platformBuffer()),
                permutationVectors->specialShapeInfo(), tads.specialShapeInfo(), tads.specialOffsets(),
                permutaionTads.specialShapeInfo(), permutaionTads.specialOffsets(), batchNum);
    }

    void lu(LaunchContext* context, NDArray* input, NDArray* output, NDArray* permutations) {
        NDArray::prepareSpecialUse({output, permutations}, {input});
        BUILD_DOUBLE_SELECTOR(input->dataType(), permutations->dataType(), lu_, (context, input, output, permutations), SD_FLOAT_NATIVE, SD_INDEXING_TYPES);
        NDArray::registerSpecialUse({output, permutations}, {input});
    }
// ------------------------------------------------------------------------------------------------------------------ //
    template<typename T>
    static sd::Status determinant_(sd::LaunchContext *context, NDArray *input, NDArray *output) {
        sd::LongType n = input->sizeAt(-1);
        sd::LongType n2 = n * n;
        std::vector<int> dims();
        auto packX = ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), {input->rankOf() - 2, input->rankOf() - 1});
        //auto packZ = ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), {output->rankOf() - 1});
//        DataType dtype = input->dataType();
//        if (dtype != DataType::DOUBLE)
//            dtype = DataType::FLOAT32;
        auto matrix = NDArrayFactory::create(input->ordering(), {n, n}, DataTypeUtils::fromT<T>(), context); //, block.getWorkspace());
        auto det = NDArrayFactory::create<T>(1, context);
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {input});
        dim3 launchDims(256, 256, 1024);
        output->assign(1.f);
        for (int e = 0; e < output->lengthOf(); e++) {
            sd::LongType pos = e * n2;
//            if (matrix.dataType() == input->dataType())
            fillMatrix<T, T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), pos, n);
//            else
//                fillMatrix<T, float><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->special(), pos, n);
            lup_<T, int>(context, &matrix, nullptr, nullptr);
//            else
//                lup_<float>(context, &matrix, nullptr, nullptr);
            auto offset = shape::getIndexOffset(e, output->shapeInfo());
            auto inputBuf = reinterpret_cast<T *>(matrix.specialBuffer());
            auto outputBuf = reinterpret_cast<T *>(output->specialBuffer()) + offset;
//            if (matrix.dataType() == input->dataType())
            determinantKernel<T><<< launchDims.x, launchDims.y, launchDims.z, *stream>>>(inputBuf, outputBuf, n);
//            else
//                determinantKernel<T, float><<<launchDims.x, launchDims.y, launchDims.z, *stream >>> (inputBuf, outputBuf, n);
        }
        NDArray::registerSpecialUse({output}, {input});

        return sd::Status::OK;
    }

        sd::Status determinant(sd::LaunchContext *context, NDArray *input, NDArray *output) {
            NDArray::prepareSpecialUse({output}, {input});
            BUILD_SINGLE_SELECTOR(input->dataType(), return determinant_, (context, input, output), SD_FLOAT_NATIVE);
            NDArray::registerSpecialUse({output}, {input});
        }

        template<typename T>
        sd::Status logAbsDeterminant_(LaunchContext *context, NDArray *input, NDArray *output) {
            sd::LongType n = input->sizeAt(-1);
            sd::LongType n2 = n * n;
            std::vector<int> dims();
            auto packX = ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), {input->rankOf() - 2, input->rankOf() - 1});
            //auto packZ = ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), {output->rankOf() - 1});
            DataType dtype = input->dataType();
            if (dtype != DataType::DOUBLE)
                dtype = DataType::FLOAT32;

            auto matrix = NDArrayFactory::create(input->ordering(), {n, n}, dtype, context); //, block.getWorkspace());
            auto det = NDArrayFactory::create<T>(1, context);
            auto stream = context->getCudaStream();
            NDArray::prepareSpecialUse({output}, {input});
            dim3 launchDims(256, 256, 1024);
            output->assign(0.f);
            for (int e = 0; e < output->lengthOf(); e++) {
                sd::LongType pos = e * n2;
//            if (matrix.dataType() == input->dataType())
                fillMatrix<T, T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), pos, n);
//            else
//                fillMatrix<T, float><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->special(), pos, n);

//            if (matrix.dataType() == input->dataType())
                lup_<T, int>(context, &matrix, nullptr, nullptr);
//            else
//                lup_<float>(context, &matrix, nullptr, nullptr);
                auto offset = shape::getIndexOffset(e, output->shapeInfo());
                auto inputBuf = reinterpret_cast<T *>(matrix.specialBuffer());
                auto outputBuf = reinterpret_cast<T *>(output->specialBuffer()) + offset;
//            if (matrix.dataType() == input->dataType())
                determinantLogKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(inputBuf, outputBuf, n);
//            else
//                determinantLogKernel<T, float><<<launchDims.x, launchDims.y, launchDims.z, *stream >>> (inputBuf, outputBuf, n);
            }
            NDArray::registerSpecialUse({output}, {input});

            return sd::Status::OK;
        }

        sd::Status logAbsDeterminant(sd::LaunchContext *context, NDArray *input, NDArray *output) {
            NDArray::prepareSpecialUse({output}, {input});
            BUILD_SINGLE_SELECTOR(input->dataType(), return logAbsDeterminant_, (context, input, output), SD_FLOAT_NATIVE);
            NDArray::registerSpecialUse({output}, {input});
        }

        template<typename T>
        static SD_KERNEL void
        fillLowerUpperKernel(
                void *lowerBuf, const sd::LongType *lowerShape,
                void *upperBuf, const sd::LongType *upperShape,
                void *matrixBuf, const sd::LongType *matrixShape,
                sd::LongType n) {

            __shared__ T *lowerMatrix;
            __shared__ T *upperMatrix;
            __shared__ T *matrix;

            if (threadIdx.x == 0) {
                lowerMatrix = reinterpret_cast<T *>(lowerBuf);
                upperMatrix = reinterpret_cast<T *>(upperBuf);
                matrix = reinterpret_cast<T *>(matrixBuf);
            }
            __syncthreads();

            for (int k = blockIdx.x; k < n; k += gridDim.x) {  // and then put all values under main diagonal on to it
                for (int j = threadIdx.x; j < n; j += blockDim.x) {
                    sd::LongType posX[] = {k, j};
                    sd::LongType posD[] = {j, j};
                    auto xPos = shape::getOffset(lowerShape, posX);
                    auto yPos = shape::getOffset(upperShape, posX);
                    auto iPos = shape::getOffset(matrixShape, posX);
                    auto dPos = shape::getOffset(matrixShape, posD);
                    if (k >= j)
                        lowerMatrix[xPos] = matrix[iPos];//(k, j);
                    else
                        upperMatrix[yPos] = matrix[iPos]; //k, j);
                }
            }
        }

        template<typename T>
        static sd::Status inverse_(sd::LaunchContext *context, NDArray *input, NDArray *output) {
            auto n = input->sizeAt(-1);
            auto n2 = n * n;
            auto dtype = DataTypeUtils::fromT<T>(); //input->dataType();
//            if (dtype != DataType::DOUBLE)
//                dtype = DataType::FLOAT32;
            NDArray matrix = NDArrayFactory::create('c', {n, n}, dtype, context);
            NDArray upper = NDArrayFactory::create('c', {n, n}, dtype, context);
            NDArray lower = NDArrayFactory::create('c', {n, n}, dtype, context);
            NDArray compound = NDArrayFactory::create('c', {n, n}, dtype, context);
            NDArray permutation = NDArrayFactory::create('c', {n, n}, dtype, context);
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(),
                                                                                  {input->rankOf() - 2,
                                                                                   input->rankOf() - 1});
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(),
                                                                                  {output->rankOf() - 2,
                                                                                   output->rankOf() - 1});
            auto stream = context->getCudaStream();

            for (auto i = 0LL; i < packX.numberOfTads(); i++) {
                fillMatrix<T, T><<<1, n2, 1024, *stream>>>(matrix.specialBuffer(), matrix.specialShapeInfo(), input->specialBuffer(), input->specialShapeInfo(), i * n2, n);
                matrix.tickWriteDevice();
                //compound.assign(matrix);
//            if (matrix.dataType() == input->dataType())
                lup_<T, int>(context, &matrix, nullptr, nullptr);
                fillLowerUpperKernel<T><<<n, n, 1024, *stream>>>(lower.specialBuffer(), lower.specialShapeInfo(), upper.specialBuffer(), upper.specialShapeInfo(), matrix.specialBuffer(), matrix.specialShapeInfo(), n);
                lower.tickWriteDevice();
                upper.tickWriteDevice();
//                lower.printIndexedBuffer("LOWER");
//                upper.printIndexedBuffer("UPPER");
                matrix.assign(0);
                invertUpperMatrix(context, &upper, &matrix); // U^{-1}
                matrix.tickWriteDevice();
//                matrix.printIndexedBuffer("Upper Inverted");
                compound.assign(0);
                invertLowerMatrix(context, &lower, &compound); // L{-1}
                compound.tickWriteDevice();
//                compound.printIndexedBuffer("Lower Inverted");
//                matrix.tickWriteDevice();
//                compound.tickWriteDevice();
                sd::MmulHelper::mmul(&matrix, &compound, &upper, 1.0, 0.0);
                upper.tickWriteDevice();
//                upper.printIndexedBuffer("Full inverted");
                returnMatrix<T><<<1, n2, 1024, *stream>>>(output->specialBuffer(), output->specialShapeInfo(), upper.specialBuffer(), upper.specialShapeInfo(), i * n2, n);
            }
            return sd::Status::OK;
        }

        sd::Status inverse(sd::LaunchContext *context, NDArray *input, NDArray *output) {
            NDArray::prepareSpecialUse({output}, {input});
            BUILD_SINGLE_SELECTOR(input->dataType(), return inverse_, (context, input, output), SD_FLOAT_NATIVE);
            NDArray::registerSpecialUse({output}, {input});
        }

        bool checkCholeskyInput(sd::LaunchContext *context, NDArray const *input) {
            return true;
        }

        template<typename F>
        SD_KERNEL void fillBatchKernel(F **dArrayBatch, F *buf, const sd::LongType *offsets, sd::LongType batchSize) {
            auto start = blockIdx.x * blockDim.x + threadIdx.x;
            auto step = blockDim.x * gridDim.x;

            for (auto i = start; i < batchSize; i += step) {
                dArrayBatch[i] = buf + offsets[i];
            }
        }

        template<typename F>
        SD_KERNEL void
        adjustResultsKernel(F *dArray, const sd::LongType *shape, const sd::LongType *offsets, sd::LongType batchSize, sd::LongType n) {
            //auto i = blockIdx.x * blockDim.x + threadIdx.x;
            sd::LongType *shapeOf = shape::shapeOf(shape);
            sd::LongType *strideOf = shape::stride(shape);

            for (auto i = blockIdx.x; i < batchSize; i += gridDim.x) {
                auto current = dArray + offsets[i];
                for (auto r = threadIdx.x; r < n; r += blockDim.x) {
                    for (auto c = r + 1; c < n; c++) {
                        sd::LongType posRC[] = {r, c};
                        auto pos = r * n + c; //shape::getOffset(0, shapeOf, strideOf, posRC, 2);
                        current[pos] = 0.;
                    }
                }
            }
        }

        template<typename F>
        sd::Status cholesky__(LaunchContext *context, NDArray *input, NDArray *output, bool inplace) {
            if (!inplace)
                output->assign(input);
            auto tempOutput =output->dup();
            hipsolverHandle_t handle = nullptr;
            auto n = input->sizeAt(-1);
            auto n2 = n * n;
            NDArray::prepareSpecialUse({output}, {input});
            auto status = hipsolverDnCreate(&handle);
            if (HIPSOLVER_STATUS_SUCCESS != status) {
                throw cuda_exception::build("helpers::cholesky_: Cannot create solver handle", status);
            }
            F **dArrayBatch = nullptr;
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(tempOutput.shapeInfo(),
                                                                                  {tempOutput.rankOf() - 2,
                                                                                   tempOutput.rankOf() - 1});
            const sd::LongType batchSize = packX.numberOfTads();
            int *dInfoArray = nullptr;
            auto err = hipMalloc((void **) &dArrayBatch, sizeof(F *) * batchSize);
            if (err) {
                throw cuda_exception::build("helpers::cholesky_: Cannot allocate memory for solver batch data buffer",
                                            err);
            }
            err = hipMalloc((void **) &dInfoArray, sizeof(int) * batchSize);
            if (err) {
                throw cuda_exception::build("helpers::cholesky_: Cannot allocate memory for solver errors buffer", err);
            }
            auto stream = context->getCudaStream();
            fillBatchKernel<F><<<1, batchSize, 128, *stream>>>(dArrayBatch, reinterpret_cast<F *>(tempOutput.specialBuffer()), packX.specialOffsets(), batchSize);

            status = hipsolverSetStream(handle, *stream);
            if (HIPSOLVER_STATUS_SUCCESS != status) {
                throw cuda_exception::build("helpers::cholesky_: Cannot set stream to solver handle", status);
            }
            const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;
            if (input->dataType() == DataType::DOUBLE)
                status = hipsolverDnDpotrfBatched(
                        handle,
                        uplo,
                        n,
                        (double **) dArrayBatch,
                        n,
                        dInfoArray,
                        batchSize);
            else
                status = hipsolverDnSpotrfBatched(
                        handle,
                        uplo,
                        n,
                        (float **) dArrayBatch,
                        n,
                        dInfoArray,
                        batchSize);

            if (HIPSOLVER_STATUS_SUCCESS != status) {
                throw cuda_exception::build("helpers::cholesky_: Cholesky factorization failed for batch", status);
            }
            adjustResultsKernel<F><<<batchSize, n2, 128, *stream>>>(reinterpret_cast<F *>(tempOutput.specialBuffer()), packX.specialShapeInfo(), packX.specialOffsets(), batchSize, n);

            err = hipFree(dArrayBatch);
            if (err) {
                throw cuda_exception::build("helpers::cholesky_: Cannot deallocate memory for solver batch data buffer",
                                            err);
            }
            err = hipFree(dInfoArray);
            if (err) {
                throw cuda_exception::build("helpers::cholesky_: Cannot allocate memory for solver errors buffer", err);
            }

            if (!inplace)
                output->assign(tempOutput);
            else
                input->assign(tempOutput);

            NDArray::registerSpecialUse({output}, {input});
            return sd::Status::OK;
        }

//    template <typename T>
        sd::Status cholesky_(LaunchContext *context, NDArray *input, NDArray *output, bool inplace) {
            NDArray::prepareSpecialUse({output}, {input});
            if (input->dataType() == DataType::DOUBLE)
                cholesky__<double>(context, input, output, inplace);
            else if (input->dataType() == DataType::FLOAT32)
                cholesky__<float>(context, input, output, inplace);
            else {
                std::unique_ptr<NDArray> tempOutput(
                        NDArrayFactory::create_('c', input->getShapeAsVector(), DataType::FLOAT32, context));
                tempOutput->assign(input);
                cholesky__<float>(context, tempOutput.get(), tempOutput.get(), true);
                output->assign(tempOutput.get());
            }
            NDArray::registerSpecialUse({output}, {input});
            return sd::Status::OK;
        }

        sd::Status cholesky(sd::LaunchContext *context, NDArray *input, NDArray *output, bool inplace) {
//        BUILD_SINGLE_SELECTOR(input->dataType(), return cholesky_, (context, input, output, inplace), SD_FLOAT_TYPES);
            return cholesky_(context, input, output, inplace);
        }
//    BUILD_SINGLE_TEMPLATE(template sd::Status cholesky_, (LaunchContext* context, NDArray* input, NDArray* output, bool inplace), SD_FLOAT_TYPES);
        BUILD_SINGLE_TEMPLATE(template sd::Status inverse_, (sd::LaunchContext * context, NDArray * input, NDArray * output),
                              SD_FLOAT_NATIVE);

        template<typename T>
        SD_KERNEL void logDetKernel(
                const T *inputBuf, const sd::LongType *inputShape,
                sd::LongType batchNum,
                const sd::LongType *tadShape, const sd::LongType *tadOffsets,
                T *outputBuf, const sd::LongType *outputShape) {

            __shared__ int n;
            if (threadIdx.x == 0) {
                n = shape::sizeAt(inputShape, -1); // * shape::sizeAt(inputShape, -1);
            }
            __syncthreads();

            auto output = outputBuf;
            auto input = inputBuf;

            for (auto i = blockIdx.x; i < batchNum; i += gridDim.x) {
                auto current = input + tadOffsets[i];

                auto zIndex = shape::getIndexOffset(i, outputShape);
                for (auto e = threadIdx.x; e < n; e += blockDim.x) {
                    sd::LongType diag[] = {e, e};
                    auto xIndex = shape::getOffset(tadShape, diag);
                    math::atomics::sd_atomicAdd(&output[zIndex],math::sd_log<T, T>(current[xIndex] * current[xIndex]));
                }
            }
        }

        template<typename T>
        sd::Status logdetFunctor_(sd::LaunchContext *context, NDArray *input, NDArray *output) {
            NDArray::prepareSpecialUse({output}, {input});
            auto n2 = input->sizeAt(-1) * input->sizeAt(-2);
            auto stream = context->getCudaStream();
            NDArray tempOutput(*input);

            cholesky(context, input, &tempOutput, false);

            auto outputBuf = output->dataBuffer()->specialAsT<T>(); //reinterpret_cast<T*>(output->specialBuffer()); // + e * n2; // + e * n2;
            auto inputBuf = tempOutput.dataBuffer()->specialAsT<T>(); //reinterpret_cast<T*>(tempOutput.specialBuffer());
            output->nullify();
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(tempOutput.shapeInfo(),
                                                                                  {tempOutput.rankOf() - 2,
                                                                                   tempOutput.rankOf() - 1});
            logDetKernel<T><<<128, 512, 256, *stream>>>(inputBuf, tempOutput.specialShapeInfo(),
                    packX.numberOfTads(), packX.specialShapeInfo(),
                    packX.specialOffsets(), outputBuf, output->specialShapeInfo());
            output->tickWriteDevice();
            NDArray::registerSpecialUse({output}, {input});
            return sd::Status::OK;
        }

        sd::Status logdetFunctor(sd::LaunchContext *context, NDArray *input, NDArray *output) {
            BUILD_SINGLE_SELECTOR(output->dataType(), return logdetFunctor_, (context, input, output), SD_FLOAT_NATIVE);
        }

        /*
         * lup - batched input, batched outputs
         * */
         sd::Status lup(LaunchContext *context, NDArray *input, NDArray *compound, NDArray *permutation) {
            BUILD_DOUBLE_SELECTOR(input->dataType(), permutation->dataType(), lup_,(context, input, compound, permutation), SD_FLOAT_NATIVE, SD_INDEXING_TYPES);
            return sd::Status::OK;
        }

//        BUILD_SINGLE_TEMPLATE(template sd::Status logdetFunctor_,
//                              (sd::LaunchContext * context, NDArray * input, NDArray * output), SD_FLOAT_NATIVE);
    }
}
}
