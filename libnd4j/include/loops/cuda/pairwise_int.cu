#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//  @author raver119@gmail.com
// @author Yurii Shyrma (iuriish@yahoo.com), created on 08.11.2018

#ifndef PAIRWISE_INT_CU
#define PAIRWISE_INT_CU

#include "../pairwise_int.h"


using namespace simdOps;

////////////////////////////////////////////////////////////////////////////////
template <typename X, typename OpType>
SD_KERNEL static void pairwiseSimpleShaped(void const* vx, sd::LongType const* xShapeInfo,
                                            void const* vy, sd::LongType const* yShapeInfo,
                                            void *vz, sd::LongType const* zShapeInfo,
                                            void *vextraParams) {

    auto x = reinterpret_cast<X const*>(vx);
    auto y = reinterpret_cast<X const*>(vy);
    auto z = reinterpret_cast<X*>(vz);
    auto extraParams = reinterpret_cast<X*>(vextraParams);

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ int xEws;
    __shared__ int yEws;
    __shared__ int zEws;
    __shared__ char xOrder;
    __shared__ char yOrder;
    __shared__ char zOrder;
    __shared__ sd::LongType len;

    if (threadIdx.x == 0) {
        xEws = shape::elementWiseStride(xShapeInfo);
        yEws = shape::elementWiseStride(yShapeInfo);
        zEws = shape::elementWiseStride(zShapeInfo);
        xOrder = shape::order(xShapeInfo);
        yOrder = shape::order(yShapeInfo);
        zOrder = shape::order(zShapeInfo);
        len = shape::length(xShapeInfo);
    }
    __syncthreads();


    if (xEws >= 1 && yEws >= 1 && zEws >= 1 && xOrder == yOrder && xOrder == zOrder) {
        for (sd::LongType i = tid; i < len; i += gridDim.x * blockDim.x) {
            z[i * zEws] = OpType::op(x[i * xEws], y[i * yEws], extraParams);
        }
    }
    else if (vx == vz) {
        for (sd::LongType i = tid; i < len; i += gridDim.x * blockDim.x) {
            auto xOffset = shape::getIndexOffset(i, xShapeInfo);
            auto yOffset = shape::getIndexOffset(i, yShapeInfo);

            z[xOffset] = OpType::op(x[xOffset], y[yOffset], extraParams);
        }
    }
    else {
        for (sd::LongType i = tid; i < len; i += gridDim.x * blockDim.x) {
            auto xOffset = shape::getIndexOffset(i, xShapeInfo);
            auto yOffset = shape::getIndexOffset(i, yShapeInfo);
            auto zOffset = shape::getIndexOffset(i, zShapeInfo);

            z[zOffset] = OpType::op(x[xOffset], y[yOffset], extraParams);
        }
    }
}


namespace functions           {
namespace pairwise_transforms {

////////////////////////////////////////////////////////////////////////////////
template<typename X>
template<typename OpType>
void SD_HOST PairWiseIntTransform<X>::intermediateShaped(dim3& launchDims, hipStream_t *stream,
                                                        void const* vx, sd::LongType const* xShapeInfo,
                                                        void const* vy, sd::LongType const* yShapeInfo,
                                                        void *vz, sd::LongType const* zShapeInfo,
                                                        void *vextraParams){

    pairwiseSimpleShaped<X, OpType><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, vextraParams);
}


////////////////////////////////////////////////////////////////////////////////
template<typename X>
void PairWiseIntTransform<X>::executeCudaShaped(dim3& launchDims, hipStream_t *stream, int opNum, void const* vx, sd::LongType const* xShapeInfo, void const* vy, sd::LongType const* yShapeInfo, void *vz, sd::LongType const* zShapeInfo, void *vextraParams) {
    auto xType = sd::DataTypeUtils::fromT<X>();

    DISPATCH_BY_OPNUM_T(intermediateShaped, PARAMS(launchDims, stream, vx, xShapeInfo, vy, yShapeInfo, vz, zShapeInfo, vextraParams), PAIRWISE_INT_OPS);
}

    BUILD_SINGLE_TEMPLATE(template class SD_LIB_HIDDEN PairWiseIntTransform, , SD_INTEGER_TYPES);
}
}

#endif // PAIRWISE_INT_CU