#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 31.08.2018
//
#include <ops/declarable/helpers/histogramFixedWidth.h>
#include <exceptions/cuda_exception.h>
#include <helpers/PointersManager.h>

namespace sd    {
namespace ops     {
namespace helpers {

///////////////////////////////////////////////////////////////////
template<typename X, typename Z>
SD_KERNEL static void histogramFixedWidthCuda( const void* vx, const sd::LongType* xShapeInfo,
                                                      void* vz, const sd::LongType* zShapeInfo,
                                                const X leftEdge, const X rightEdge) {

    const auto x  = reinterpret_cast<const X*>(vx);
    auto z = reinterpret_cast<Z*>(vz);

    __shared__ sd::LongType xLen, zLen, totalThreads, nbins;
    __shared__ X binWidth, secondEdge, lastButOneEdge;

    if (threadIdx.x == 0) {

        xLen  = shape::length(xShapeInfo);
        nbins = shape::length(zShapeInfo);          // nbins = zLen
        totalThreads = gridDim.x * blockDim.x;

        binWidth       = (rightEdge - leftEdge ) / nbins;
        secondEdge     = leftEdge + binWidth;
        lastButOneEdge = rightEdge - binWidth;
    }

    __syncthreads();

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (sd::LongType i = tid; i < xLen; i += totalThreads) {

        const X value = x[shape::getIndexOffset(i, xShapeInfo)];

        sd::LongType zIndex;

        if(value < secondEdge)
            zIndex = 0;
        else if(value >= lastButOneEdge)
            zIndex = nbins - 1;
        else
            zIndex = static_cast<sd::LongType>((value - leftEdge) / binWidth);

        sd::math::atomics::sd_atomicAdd<Z>(&z[shape::getIndexOffset(zIndex, zShapeInfo)], 1);
    }
}

///////////////////////////////////////////////////////////////////
template<typename X, typename Z>
SD_HOST static void histogramFixedWidthCudaLauncher(const hipStream_t *stream, const NDArray& input, const NDArray& range, NDArray& output) {

    const X leftEdge  = range.e<X>(0);
    const X rightEdge = range.e<X>(1);

    histogramFixedWidthCuda<X, Z><<<256, 256, 1024, *stream>>>(input.specialBuffer(), input.specialShapeInfo(), output.specialBuffer(), output.specialShapeInfo(), leftEdge, rightEdge);
}

////////////////////////////////////////////////////////////////////////
void histogramFixedWidth(sd::LaunchContext* context, const NDArray& input, const NDArray& range, NDArray& output) {

    // firstly initialize output with zeros
    output.nullify();

    PointersManager manager(context, "histogramFixedWidth");

    NDArray::prepareSpecialUse({&output}, {&input});
    BUILD_DOUBLE_SELECTOR(input.dataType(), output.dataType(), histogramFixedWidthCudaLauncher, (context->getCudaStream(), input, range, output), SD_COMMON_TYPES, SD_INDEXING_TYPES);
    NDArray::registerSpecialUse({&output}, {&input});

    manager.synchronize();
}


//     template <typename T>
//     SD_KERNEL static void copyBuffers(sd::LongType* destination, void const* source, sd::LongType* sourceShape, sd::LongType bufferLength) {
//         const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
//         const auto step = gridDim.x * blockDim.x;
//         for (int t = tid; t < bufferLength; t += step) {
//             destination[t] = reinterpret_cast<T const*>(source)[shape::getIndexOffset(t, sourceShape)];
//         }
//     }

//     template <typename T>
//     SD_KERNEL static void returnBuffers(void* destination, sd::LongType const* source, sd::LongType* destinationShape, sd::LongType bufferLength) {
//         const auto tid = blockIdx.x * gridDim.x + threadIdx.x;
//         const auto step = gridDim.x * blockDim.x;
//         for (int t = tid; t < bufferLength; t += step) {
//             reinterpret_cast<T*>(destination)[shape::getIndexOffset(t, destinationShape)] = source[t];
//         }
//     }

//     template <typename T>
//     static SD_KERNEL void histogramFixedWidthKernel(void* outputBuffer, sd::LongType outputLength, void const* inputBuffer, sd::LongType* inputShape, sd::LongType inputLength, double const leftEdge, double binWidth, double secondEdge, double lastButOneEdge) {

//         __shared__ T const* x;
//         __shared__ sd::LongType* z; // output buffer

//         if (threadIdx.x == 0) {
//             z = reinterpret_cast<sd::LongType*>(outputBuffer);
//             x = reinterpret_cast<T const*>(inputBuffer);
//         }
//         __syncthreads();
//         auto tid = blockIdx.x * gridDim.x + threadIdx.x;
//         auto step = blockDim.x * gridDim.x;

//         for(auto i = tid; i < inputLength; i += step) {

//             const T value = x[shape::getIndexOffset(i, inputShape)];
//             sd::LongType currInd = static_cast<sd::LongType>((value - leftEdge) / binWidth);

//             if(value < secondEdge)
//                 currInd = 0;
//             else if(value >= lastButOneEdge)
//                 currInd = outputLength - 1;
//             sd::math::atomics::sd_atomicAdd(&z[currInd], 1LL);
//         }
//     }


//     template <typename T>
//     void histogramFixedWidth_(sd::LaunchContext * context, const NDArray& input, const NDArray& range, NDArray& output) {
//         const int nbins = output.lengthOf();
//         auto stream = context->getCudaStream();
//         // firstly initialize output with zeros
//         //if(output.ews() == 1)
//         //    memset(output.buffer(), 0, nbins * output.sizeOfT());
//         //else
//         output.assign(0);
//         if (!input.isActualOnDeviceSide())
//             input.syncToDevice();

//         const double leftEdge  = range.e<double>(0);
//         const double rightEdge = range.e<double>(1);

//         const double binWidth       = (rightEdge - leftEdge ) / nbins;
//         const double secondEdge     = leftEdge + binWidth;
//         double lastButOneEdge = rightEdge - binWidth;
//         sd::LongType* outputBuffer;
//         hipError_t err = hipMalloc(&outputBuffer, output.lengthOf() * sizeof(sd::LongType));
//         if (err != 0)
//             throw cuda_exception::build("helpers::histogramFixedWidth: Cannot allocate memory for output", err);
//         copyBuffers<sd::LongType ><<<256, 512, 8192, *stream>>>(outputBuffer, output.specialBuffer(), output.special(), output.lengthOf());
//         histogramFixedWidthKernel<T><<<256, 512, 8192, *stream>>>(outputBuffer, output.lengthOf(), input.specialBuffer(), input.special(), input.lengthOf(), leftEdge, binWidth, secondEdge, lastButOneEdge);
//         returnBuffers<sd::LongType><<<256, 512, 8192, *stream>>>(output.specialBuffer(), outputBuffer, output.special(), output.lengthOf());
//         //cudaSyncStream(*stream);
//         err = hipFree(outputBuffer);
//         if (err != 0)
//             throw cuda_exception::build("helpers::histogramFixedWidth: Cannot deallocate memory for output buffer", err);
//         output.tickWriteDevice();
// //#pragma omp parallel for schedule(guided)
// //        for(sd::LongType i = 0; i < input.lengthOf(); ++i) {
// //
// //            const T value = input.e<T>(i);
// //
// //            if(value < secondEdge)
// //#pragma omp critical
// //                output.p<sd::LongType>(0, output.e<sd::LongType>(0) + 1);
// //            else if(value >= lastButOneEdge)
// //#pragma omp critical
// //                output.p<sd::LongType>(nbins-1, output.e<sd::LongType>(nbins-1) + 1);
// //            else {
// //                sd::LongType currInd = static_cast<sd::LongType>((value - leftEdge) / binWidth);
// //#pragma omp critical
// //                output.p<sd::LongType>(currInd, output.e<sd::LongType>(currInd) + 1);
// //            }
// //        }
//     }

//     void histogramFixedWidth(sd::LaunchContext * context, const NDArray& input, const NDArray& range, NDArray& output) {
//         BUILD_SINGLE_SELECTOR(input.dataType(), histogramFixedWidth_, (context, input, range, output), SD_COMMON_TYPES);
//     }
//     BUILD_SINGLE_TEMPLATE(template void histogramFixedWidth_, (sd::LaunchContext * context, const NDArray& input, const NDArray& range, NDArray& output), SD_COMMON_TYPES);

}
}
}