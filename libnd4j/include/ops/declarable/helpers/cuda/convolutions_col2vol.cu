#include "hip/hip_runtime.h"
/*
 *  ******************************************************************************
 *  *
 *  *
 *  * This program and the accompanying materials are made available under the
 *  * terms of the Apache License, Version 2.0 which is available at
 *  * https://www.apache.org/licenses/LICENSE-2.0.
 *  *
 *  * See the NOTICE file distributed with this work for additional
 *  * information regarding copyright ownership.
 *  * Unless required by applicable law or agreed to in writing, software
 *  * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 *  * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 *  * License for the specific language governing permissions and limitations
 *  * under the License.
 *  *
 *  * SPDX-License-Identifier: Apache-2.0
 *  *****************************************************************************
 */

//
// @author Yurii Shyrma (iuriish@yahoo.com)
//
#include <ops/declarable/helpers/convolutions.h>
#include <helpers/PointersManager.h>
#include <math/templatemath.h>

namespace sd {
namespace ops  {

//////////////////////////////////////////////////////////////////////////
// columns [bS, iC, kD, kH, kW, oD, oH, oW] to be de-convoluted to volume [bS, iC, iD, iH, iW]
template <typename T>
static SD_KERNEL void col2volCuda(const void* columns, const sd::LongType* colShapeInfo, void* volume, const sd::LongType* volShapeInfo,  const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

    const T* col = reinterpret_cast<const T*>(columns);
          T* vol = reinterpret_cast<T*>(volume);

    __shared__ sd::Unsigned kD, kH, kW, oD, oH, oW, *sharedMem;
    __shared__ sd::LongType volLen;

    if (threadIdx.x == 0) {
        extern __shared__ unsigned char shmem[];
        sharedMem = reinterpret_cast<sd::Unsigned*>(shmem);

        oD = colShapeInfo[6];
        oH = colShapeInfo[7];
        oW = colShapeInfo[8];

        kD = dD * (colShapeInfo[3] - 1) + 1;
        kH = dH * (colShapeInfo[4] - 1) + 1;
        kW = dW * (colShapeInfo[5] - 1) + 1;

        volLen  = shape::length(volShapeInfo);
    }
    __syncthreads();

    auto coords = sharedMem + threadIdx.x * 8;

    const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (sd::LongType i = tid; i < volLen; i += gridDim.x * blockDim.x) {

        shape::index2coords(i, volShapeInfo, coords);

        const auto volOffset = shape::getOffset(volShapeInfo, coords);

        const auto bSiCoffset = coords[0] * colShapeInfo[9] + coords[1] * colShapeInfo[10];

        const sd::Unsigned imD = coords[2] + pD;
        const sd::Unsigned imH = coords[3] + pH;
        const sd::Unsigned imW = coords[4] + pW;

        const sd::Unsigned colDstart = (imD < kD) ? 0 : (imD - kD) / sD + 1;
        const sd::Unsigned colHstart = (imH < kH) ? 0 : (imH - kH) / sH + 1;
        const sd::Unsigned colWstart = (imW < kW) ? 0 : (imW - kW) / sW + 1;

        const sd::Unsigned colDend = sd::math::sd_min<sd::Unsigned>(imD / sD + 1, oD);
        const sd::Unsigned colHend = sd::math::sd_min<sd::Unsigned>(imH / sH + 1, oH);
        const sd::Unsigned colWend = sd::math::sd_min<sd::Unsigned>(imW / sW + 1, oW);

        T val = 0;

        for(sd::Unsigned colD = colDstart; colD < colDend; ++colD) {
            coords[2] = imD - colD * sD;
            if(coords[2] % dD != 0) continue;

            for(sd::Unsigned colH = colHstart; colH < colHend; ++colH) {
                coords[3] = imH - colH * sH;
                if(coords[3] % dH != 0) continue;

                for(sd::Unsigned colW = colWstart; colW < colWend; ++colW) {
                    coords[4] = imW - colW * sW;
                    if(coords[4] % dW != 0) continue;

                    val += col[bSiCoffset + (coords[2]/dD)*colShapeInfo[11] + (coords[3]/dH)*colShapeInfo[12] + (coords[4]/dW)*colShapeInfo[13] + colD*colShapeInfo[14] + colH*colShapeInfo[15] + colW*colShapeInfo[16]];

                }
            }
        }

        vol[volOffset] = val;
    }
}

//////////////////////////////////////////////////////////////////////////
template <typename T>
static void col2volCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const int sharedMem, const hipStream_t *stream,
                                const void* columns, const sd::LongType* colShapeInfo,
                                      void* volume, const sd::LongType* volShapeInfo,
                                const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

    col2volCuda<T><<<blocksPerGrid, threadsPerBlock, sharedMem, *stream>>>(columns, colShapeInfo, volume, volShapeInfo, sD, sH, sW, pD, pH, pW, dD, dH, dW);
}

//////////////////////////////////////////////////////////////////////////
void ConvolutionUtils::col2vol(sd::graph::Context& block, const NDArray& col, NDArray& vol, const int sD, const int sH, const int sW, const int pD, const int pH, const int pW, const int dD, const int dH, const int dW) {

    PointersManager manager(block.launchContext(), "col2vol");

    const int threadsPerBlock = SD_MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (vol.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;
    const int sharedMem = col.rankOf() * sizeof(sd::Unsigned) * threadsPerBlock  + 256;

    NDArray::prepareSpecialUse({&vol}, {&col});
    BUILD_SINGLE_SELECTOR(vol.dataType(), col2volCudaLauncher, (blocksPerGrid, threadsPerBlock, sharedMem, block.launchContext()->getCudaStream(), col.specialBuffer(), col.specialShapeInfo(), vol.specialBuffer(), vol.specialShapeInfo(), sD, sH, sW, pD, pH, pW, dD, dH, dW), SD_FLOAT_TYPES);
    NDArray::registerSpecialUse({&vol}, {&col});

    manager.synchronize();
}

}
}
