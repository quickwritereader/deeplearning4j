#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//
#include <ops/declarable/helpers/segment.h>
#include <ops/declarable/helpers/segment_common.h>
#include <array/NDArrayFactory.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <helpers/PointersManager.h>
#include <helpers/ConstantTadHelper.h>

namespace sd {
    namespace ops {
        namespace helpers {

            // -------------------------------------------------------------------------------------------------------------- //
            // Segment ops linear kernels
            // -------------------------------------------------------------------------------------------------------------- //

            template<typename T, typename I>
            static SD_KERNEL void
            segmentMaxLinearKernel(void *input, sd::LongType const* inputShape, int *starts, int *lengths, sd::LongType numOfClasses,
                                   void *output, sd::LongType const* outputShape) {
                __shared__                 T *val;
                __shared__                sd::LongType xLen, zLen, zIndex;
                __shared__                T *x;
                __shared__                T *z;
                __shared__ int threadsPerSegment, start, finish;

                auto segment = blockIdx.x;
                if (threadIdx.x == 0) {
//                    threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
//                    segment = blockIdx.x / threadsPerSegment;
                    x = reinterpret_cast<T *>(input);
                    z = reinterpret_cast<T *>(output);
                    extern __shared__ unsigned char shmem[];
                    val = reinterpret_cast<T *>(shmem);
                    xLen = shape::length(inputShape);
                    zLen = shape::length(outputShape);

                    if (segment < numOfClasses) {
                        zIndex = shape::getIndexOffset(segment, outputShape);
                        start = starts[segment];
                        finish = start + lengths[segment];
                        z[zIndex] = x[shape::getIndexOffset(start, inputShape)];
                        val[segment] = z[zIndex];
                    }

                }
                __syncthreads();

                for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputShape);
                    sd::math::atomics::sd_atomicMax(&z[zIndex], x[xIndex]);
                }
            }
            // -------------------------------------------------------------------------------------------------------------- //

            template<typename T, typename I>
            static SD_KERNEL void
            unsortedSegmentMaxLinearKernel(void *input, sd::LongType const* inputShape, void *indices, sd::LongType const* indicesShape,
                                           int *starts, int *lengths, sd::LongType numOfClasses, void *output,
                                           sd::LongType const* outputShape) {
                __shared__                 T *val;
                __shared__                sd::LongType xLen, zLen, zIndex;
                __shared__                T *x;
                __shared__                T *z;
                __shared__                I *y; //int threadsPerSegment, start, finish;
                auto segment = blockIdx.x;

                if (threadIdx.x == 0) {
                    x = reinterpret_cast<T *>(input);
                    z = reinterpret_cast<T *>(output);
                    y = reinterpret_cast<I *>(indices);
                    xLen = shape::length(inputShape);
                    zLen = shape::length(outputShape);

                    zIndex = shape::getIndexOffset(segment, outputShape);
                    //start = starts[segment];
                    //finish = start + lengths[segment];
                    if (lengths[segment] > 0)
                        z[zIndex] = x[shape::getIndexOffset(starts[segment], inputShape)];
                    else
                        z[zIndex] = -DataTypeUtils::max<T>();
                }
                __syncthreads();
                if (lengths[segment] > 0)
                    for (auto e = threadIdx.x + 1; e < xLen; e += blockDim.x) {
                        auto xIndex = shape::getIndexOffset(e, inputShape);
                        auto yIndex = shape::getIndexOffset(e, indicesShape);
                        if (y[yIndex] == segment) {
                            sd::math::atomics::sd_atomicMax(&z[zIndex], x[xIndex]);
                        }
                    }
            }
            // -------------------------------------------------------------------------------------------------------------- //
            template <typename T, typename I>
            static SD_KERNEL void segmentMaxTadKernel(void* inputBuf, sd::LongType const* inputShape, sd::LongType const* inputTads,
                                                       sd::LongType const* inputTadOffsets, I* indices, int* starts, int* lengths, sd::LongType numOfClasses, void* outputBuf,
                                                       sd::LongType const* outputShape, sd::LongType const* outputTads, sd::LongType const* outputTadOffsets, T filler = 0) {

                __shared__ T* val;
                __shared__ sd::LongType len, zIndex, total;
                __shared__ T* z;
                __shared__ int start, finish;
                __shared__ I segment;

                if (threadIdx.x == 0) {
                    segment = indices[blockIdx.x]; // / threadsPerSegment;
                    z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
                    len = shape::length(inputTads);

                    start = starts[segment];
                    finish = start + lengths[segment];
                    total = shape::sizeAt(inputShape, 0);
                }
                __syncthreads();

                auto idx = blockIdx.x;
                if (idx <= total) {
                    auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
                    if (blockIdx.x == start) {
                        for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                            auto xIndex = shape::getIndexOffset(e, inputTads);
                            auto zIndex = shape::getIndexOffset(e, outputTads);
                            sd::math::atomics::sd_atomicMax(&z[zIndex], x[xIndex]);
                            //z[zIndex] = x[xIndex];
                        }
                    }
                    else {
                        for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                            auto xIndex = shape::getIndexOffset(e, inputTads);
                            auto zIndex = shape::getIndexOffset(e, outputTads);
                            if (lengths[segment])
                                sd::math::atomics::sd_atomicMax(&z[zIndex], x[xIndex]);
                        }
                    }
                }
            }
            // -------------------------------------------------------------------------------------------------------------- //

            template <typename T, typename I>
            static void segmentMaxFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
                //int numClasses = output->sizeAt(0);
                // if input is a vector: (as if in doc sample)
                //sd::LongType idx = indices->e<sd::LongType>(0);
                output->assign(-DataTypeUtils::infOrMax<T>());
                auto stream = context->getCudaStream();
                indices->syncToHost();
                sd::LongType numOfClasses = indices->e<sd::LongType>(indices->lengthOf() - 1) + 1;
                NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses}, context);
                NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses}, context);

                classesRangesBegs.assign(indices->lengthOf());
                classesRangesLens.assign(0);
                dim3 dims(256, 512, 256);
                int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
                int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
                fillUpSegments(indices, numOfClasses, classesRangesBegs, classesRangesLens);

                NDArray::prepareSpecialUse({output}, {input, indices, &classesRangesBegs, &classesRangesLens});

                if (input->isVector()) {

                    segmentMaxLinearKernel<T,I><<<numOfClasses, input->lengthOf(), numOfClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
                }
                else {
                    std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
                    auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
                    auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
                    auto inputTads = packX.specialShapeInfo();
                    auto inputTadOffsets = packX.specialOffsets();
                    auto outputTads = packZ.specialShapeInfo();
                    auto outputTadOffsets = packZ.specialOffsets();
                    segmentMaxTadKernel<T,I><<<packX.numberOfTads(), 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
                }
                NDArray::registerSpecialUse({output}, {input, indices, &classesRangesBegs, &classesRangesLens});
            }
            // -------------------------------------------------------------------------------------------------------------- //
            void segmentMaxFunctor(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* output) {
                NDArray::prepareSpecialUse({output}, {input, indices});
                BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), segmentMaxFunctor_, (context, input, indices, output), SD_NUMERIC_TYPES, SD_INDEXING_TYPES);
                NDArray::registerSpecialUse({output}, {input, indices});
            }
            // -------------------------------------------------------------------------------------------------------------- //

            template <typename T, typename I>
            static void unsortedSegmentMaxFunctor_(sd::LaunchContext* context, NDArray* input, NDArray* indices, sd::LongType numOfClasses, NDArray* output) {
                auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});
                output->assign(DataTypeUtils::infOrMax<T>());

                NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses}, context);
                NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses}, context);
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(sd::BroadcastOpsTuple::Assign(), row, classes);
                classesRangesBegs.assign(indices->lengthOf());
                classesRangesLens.assign(0);
                dim3 dims(numOfClasses, indices->lengthOf(), numOfClasses * 32 + 32);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
                fillUpSegments(indices, numOfClasses, classesRangesBegs, classesRangesLens);
                int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
                int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());

                if (input->isVector()) {
                    unsortedSegmentMaxLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
                }
                else {
                    std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
                    auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
                    auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
                    auto inputTads = packX.specialShapeInfo();
                    auto inputTadOffsets = packX.specialOffsets();
                    auto outputTads = packZ.specialShapeInfo();
                    auto outputTadOffsets = packZ.specialOffsets();
                    dims.x = input->sizeAt(0);
                    output->assign(-DataTypeUtils::max<T>());
                    segmentMaxTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
                }

            }
            // -------------------------------------------------------------------------------------------------------------- //
            void unsortedSegmentMaxFunctor(sd::LaunchContext* context, NDArray* input, NDArray* indices, sd::LongType numOfClasses, NDArray* output) {
                NDArray::prepareSpecialUse({output}, {input, indices});
                output->nullify();
                BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentMaxFunctor_, (context, input, indices, numOfClasses, output), SD_NUMERIC_TYPES, SD_INDEXING_TYPES);
                NDArray::registerSpecialUse({output}, {input, indices});
            }

            // -------------------------------------------------------------------------------------------------------------- //
            // segment max
            // -------------------------------------------------------------------------------------------------------------- //
            template <typename T, typename I>
            static SD_KERNEL void segmentMaxBPLinearKernel(void* inputBuf, sd::LongType const*  inputShape, void* forwardOutput,
                                                            sd::LongType const*  forwardShape, void* eps, sd::LongType const*  epsShape, void* indicesBuf, sd::LongType const*  indicesShape,
                                                            void* outputBuf, sd::LongType const*  outputShape) {
                __shared__ T* x;
                __shared__ T* gradIn;
                __shared__ T* gradOut;
                __shared__ I* y;
                __shared__ T* z;
                __shared__ sd::LongType xLen, gradLen;

                if (threadIdx.x == 0) {
                    xLen = shape::length(inputShape);
                    x = reinterpret_cast<T*>(inputBuf);
                    y = reinterpret_cast<I*>(indicesBuf);
                    z = reinterpret_cast<T*>(outputBuf);
                    gradIn = reinterpret_cast<T*>(forwardOutput);
                    gradOut = reinterpret_cast<T*>(eps);
                    gradLen = shape::length(epsShape);
                }
                __syncthreads();

                auto start = blockIdx.x * blockDim.x + threadIdx.x;
                auto step = gridDim.x * blockDim.x;

                for (auto e = start; e < xLen; e += step) {

                    auto zOffset = shape::getIndexOffset(e, outputShape);
                    auto xOffset = shape::getIndexOffset(e, inputShape);
                    auto yOffset = shape::getIndexOffset(e, indicesShape);
                    auto classIndex = y[yOffset];
                    auto gradOffsetI = shape::getIndexOffset(classIndex, forwardShape);
                    auto gradOffsetO = shape::getIndexOffset(classIndex, epsShape);

                    if (sd::math::sd_abs(gradIn[gradOffsetI] - x[xOffset]) <= T(1.e-6)) {
                        z[zOffset] = gradOut[gradOffsetO];
                    }
                }
            }

            // -------------------------------------------------------------------------------------------------------------- //
            template <typename T, typename I>
            static SD_KERNEL void segmentMaxBPTadKernel(void* inputBuf, sd::LongType const*  inputShape, void* forwardOutput,
                                                         sd::LongType const*  forwardShape, void* eps, sd::LongType const*  epsShape, void* indicesBuf, sd::LongType const*  indicesShape,
                                                         void* outputBuf, sd::LongType const*  outputShape,sd::LongType const*  inputTad,
                                                         sd::LongType const*  inputOffsets, sd::LongType const*  gradInTad, sd::LongType const*  gradInOffsets,
                                                         sd::LongType const*  gradOutTad, sd::LongType const*  gradOutOffsets, sd::LongType const*  outTad,
                                                         sd::LongType const*  outOffsets) {
                __shared__ T* x;
                __shared__ T* gradIn;
                __shared__ T* gradOut;
                __shared__ I* y;
                __shared__ T* z;
                __shared__ sd::LongType xLen, yLen, gradLen, currentLen;

                if (threadIdx.x == 0) {
                    xLen = shape::length(inputShape);
                    x = reinterpret_cast<T*>(inputBuf);
                    y = reinterpret_cast<I*>(indicesBuf);
                    z = reinterpret_cast<T*>(outputBuf);
                    yLen = shape::length(indicesShape);
                    gradOut = reinterpret_cast<T*>(eps);
                    gradIn = reinterpret_cast<T*>(forwardOutput);
                    gradLen = shape::length(epsShape);
                    currentLen = shape::length(outTad);
                }
                __syncthreads();

                for (auto i = blockIdx.x; i < yLen; i += gridDim.x) {
                    auto yIndex = shape::getIndexOffset(i, indicesShape);
                    auto segment = y[yIndex];
                    T* current = x + inputOffsets[i];
                    T* currentOut = z + outOffsets[i];
                    T* in = gradIn + gradInOffsets[segment];
                    T* outGrad = gradOut + gradOutOffsets[segment];

                    for (auto e = threadIdx.x; e < currentLen; e += blockDim.x) {
                        if (sd::math::sd_abs(in[e] - current[e]) <= T(1.e-6))
                            currentOut[e] = outGrad[e];
                    }
                }
            }
            // -------------------------------------------------------------------------------------------------------------- //
            template <typename T, typename I>
            sd::Status segmentMaxFunctorBP_(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
                //int numOfClasses = gradOut->sizeAt(0);
                // if input is a vector: (as if in doc sample)
                auto stream = context->getCudaStream();
                NDArray tempRes(gradOut->ordering(), gradOut->getShapeAsVector(), DataTypeUtils::fromT<T>(), context);//->shapeInfo(), context);
                segmentMaxFunctor_<T, I>(context, input, indices, &tempRes);
                NDArray::prepareSpecialUse({output}, {input, indices, gradOut, &tempRes});
                if (input->isVector()) {
                    sd::LongType loop_size = input->lengthOf();
                    auto numOfClasses = gradOut->lengthOf(); //indices->e<sd::LongType>(loop_size - 1);
                    segmentMaxBPLinearKernel<T,I><<<1 + gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                            tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                            indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
                }
                else {
                    std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
                    auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
                    auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
                    auto packGradIn = sd::ConstantTadHelper::getInstance().tadForDimensions(tempRes.shapeInfo(), dimensions);
                    auto packGradOut = sd::ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
                    sd::LongType const*  inputTads = packX.specialShapeInfo();
                    sd::LongType const*  inputTadOffsets = packX.specialOffsets();
                    sd::LongType const*  outputTads = packZ.specialShapeInfo();
                    sd::LongType const*  outputTadOffsets = packZ.specialOffsets();
                    sd::LongType const*  gradInTads = packGradIn.specialShapeInfo();
                    sd::LongType const*  gradInTadOffsets = packGradIn.specialOffsets();
                    sd::LongType const*  gradOutTads = packGradOut.specialShapeInfo();
                    sd::LongType const*  gradOutTadOffsets = packGradOut.specialOffsets();

                    segmentMaxBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                            tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                            indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                            inputTads, inputTadOffsets, gradInTads, gradInTadOffsets, gradOutTads, gradOutTadOffsets,
                            outputTads, outputTadOffsets);
                }
                NDArray::registerSpecialUse({output}, {input, indices, gradOut, &tempRes});
                return sd::Status::OK;
            }
            // -------------------------------------------------------------------------------------------------------------- //
            sd::Status segmentMaxFunctorBP(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
                NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
                BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return segmentMaxFunctorBP_, (context, input,
                        indices, gradOut, output), SD_FLOAT_TYPES, SD_INDEXING_TYPES);
                NDArray::registerSpecialUse({output}, {input, indices, gradOut});
            }

            // -------------------------------------------------------------------------------------------------------------- //
            template <typename T, typename I>
            static sd::Status unsortedSegmentMaxFunctorBP_(sd::LaunchContext* context, NDArray* input, NDArray* indices, NDArray* gradOut, sd::LongType numOfClasses, NDArray* output) {
                //int numOfClasses = gradOut->sizeAt(0);
                // if input is a vector: (as if in doc sample)
                auto stream = context->getCudaStream();
                NDArray tempRes(gradOut->ordering(), gradOut->getShapeAsVector(), DataTypeUtils::fromT<T>(), context);//->shapeInfo(), context);
                unsortedSegmentMaxFunctor_<T, I>(context, input, indices, numOfClasses, &tempRes);
                NDArray::prepareSpecialUse({output}, {input, indices, gradOut, &tempRes});
                if (input->isVector()) {
                    sd::LongType loop_size = input->lengthOf();
                    auto numOfClasses = gradOut->lengthOf(); //indices->e<sd::LongType>(loop_size - 1);
                    segmentMaxBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                            tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                            indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo());
                }
                else {
                    std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
                    auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
                    auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
                    auto packGradIn = sd::ConstantTadHelper::getInstance().tadForDimensions(tempRes.shapeInfo(), dimensions);
                    auto packGradOut = sd::ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
                    sd::LongType const*  inputTads = packX.specialShapeInfo();
                    sd::LongType const*  inputTadOffsets = packX.specialOffsets();
                    sd::LongType const*  outputTads = packZ.specialShapeInfo();
                    sd::LongType const*  outputTadOffsets = packZ.specialOffsets();
                    sd::LongType const*  gradInTads = packGradIn.specialShapeInfo();
                    sd::LongType const*  gradInTadOffsets = packGradIn.specialOffsets();
                    sd::LongType const*  gradOutTads = packGradOut.specialShapeInfo();
                    sd::LongType const*  gradOutTadOffsets = packGradOut.specialOffsets();

                    segmentMaxBPTadKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                            tempRes.specialBuffer(), tempRes.specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                            indices->specialBuffer(), indices->specialShapeInfo(), output->specialBuffer(), output->specialShapeInfo(),
                            inputTads, inputTadOffsets, gradInTads, gradInTadOffsets, gradOutTads, gradOutTadOffsets,
                            outputTads, outputTadOffsets);
                }
                NDArray::registerSpecialUse({output}, {input, indices, gradOut, &tempRes});
                return sd::Status::OK;
            }
            // -------------------------------------------------------------------------------------------------------------- //
            sd::Status unsortedSegmentMaxFunctorBP(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, sd::LongType numOfClasses, NDArray* output) {
                NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
                BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentMaxFunctorBP_, (context, input, indices, gradOut, numOfClasses, output), SD_FLOAT_TYPES, SD_INDEXING_TYPES);
                NDArray::registerSpecialUse({output}, {input, indices, gradOut});
            }
        }
    }
}