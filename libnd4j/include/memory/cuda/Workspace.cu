/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// CUDA workspaces implementation
//
// @author raver119@gmail.com
//
#include <system/op_boilerplate.h>
#include <atomic>
#include <stdio.h>
#include <stdlib.h>
#include "../Workspace.h"
#include <helpers/logger.h>
#include <math/templatemath.h>
#include <cstring>
#include <exceptions/cuda_exception.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace sd {
    namespace memory {
        Workspace::Workspace(ExternalWorkspace *external) {
            if (external->sizeHost() > 0) {
                _ptrHost = (char *) external->pointerHost();
                _ptrDevice = (char *) external->pointerDevice();

                _initialSize = external->sizeDevice();
                _currentSize = external->sizeDevice();
                _initialSizeSecondary = external->sizeHost();
                _currentSizeSecondary = external->sizeHost();
                _offset = 0L;
                _offsetSecondary = 0L;
                this->_cycleAllocations = 0;
                this->_cycleAllocationsSecondary = 0;
                this->_spillsSize = 0;
                this->_spillsSizeSecondary = 0;

                _externalized = true;
            }
        }

        Workspace::Workspace(sd::LongType primarySize, sd::LongType secondarySize) {
            if (secondarySize > 0) {
                auto res = hipHostAlloc(reinterpret_cast<void **>(&_ptrHost), secondarySize, hipHostMallocDefault);
                if (res != 0)
                    throw cuda_exception::build("Can't allocate [HOST] memory", res);

                hipMemset(this->_ptrHost, 0, secondarySize);
                this->_allocatedHost = true;
            } else
                this->_allocatedHost = false;

            if (primarySize > 0) {
                auto res = hipMalloc(reinterpret_cast<void **>(&_ptrDevice), primarySize);
                if (res != 0)
                    throw cuda_exception::build("Can't allocate [DEVICE] memory", res);

                hipMemset(this->_ptrDevice, 0, primarySize);
                this->_allocatedDevice = true;
            } else
                this->_allocatedDevice = false;

            this->_initialSize = primarySize;
            this->_initialSizeSecondary = secondarySize;
            this->_currentSize = primarySize;
            this->_currentSizeSecondary = secondarySize;
            this->_offset = 0;
            this->_offsetSecondary = 0;
            this->_cycleAllocations = 0;
            this->_spillsSize = 0;
            this->_spillsSizeSecondary = 0;
        }

        void Workspace::init(sd::LongType primaryBytes, sd::LongType secondaryBytes) {
            if (this->_currentSize < primaryBytes) {
                if (this->_allocatedDevice && !_externalized)
                    hipFree((void *)this->_ptrDevice);

                auto res = hipMalloc(reinterpret_cast<void **>(&_ptrDevice), secondaryBytes);
                if (res != 0)
                    throw cuda_exception::build("Can't allocate [DEVICE] memory", res);

                hipMemset(this->_ptrDevice, 0, primaryBytes);
                this->_currentSize = primaryBytes;
                this->_allocatedDevice = true;
            }

            if (this->_currentSizeSecondary < secondaryBytes) {
                if (this->_allocatedHost && !_externalized)
                    hipHostFree((void *)this->_ptrHost);

                auto res = hipHostAlloc(reinterpret_cast<void **>(&_ptrHost), secondaryBytes, hipHostMallocDefault);
                if (res != 0)
                    throw cuda_exception::build("Can't allocate [HOST] memory", res);


                hipMemset(this->_ptrHost, 0, secondaryBytes);
                this->_currentSizeSecondary = secondaryBytes;
                this->_allocatedHost = true;
            }
        }

        void Workspace::expandBy(sd::LongType numBytes, sd::LongType secondaryBytes) {
            this->init(_currentSize + numBytes, _currentSizeSecondary + secondaryBytes);
        }

        void Workspace::expandTo(sd::LongType numBytes, sd::LongType secondaryBytes) {
            this->init(numBytes, secondaryBytes);
        }

        void Workspace::freeSpills() {
            _spillsSize = 0;
            _spillsSizeSecondary = 0;

            for (auto v:_spills)
                hipFree(v);

            for (auto v:_spillsSecondary)
                hipHostFree(v);

            _spills.clear();
            _spillsSecondary.clear();
        }

        Workspace::~Workspace() {
            if (this->_allocatedHost && !_externalized)
                hipHostFree((void *)this->_ptrHost);

            if (this->_allocatedDevice && !_externalized)
                hipFree((void *)this->_ptrDevice);

            freeSpills();
        }

        sd::LongType Workspace::getUsedSize() {
            return getCurrentOffset();
        }

        sd::LongType Workspace::getCurrentSize() {
            return _currentSize;
        }

        sd::LongType Workspace::getCurrentOffset() {
            return _offset.load();
        }


        void* Workspace::allocateBytes(sd::LongType numBytes) {
            return allocateBytes(sd::memory::MemoryType::HOST, numBytes);
        }

        sd::LongType Workspace::getAllocatedSize() {
            return getCurrentSize() + getSpilledSize();
        }

        void Workspace::scopeIn() {
            freeSpills();
            init(_cycleAllocations.load());
            _cycleAllocations = 0;
        }

        void Workspace::scopeOut() {
            _offset = 0;
        }

        sd::LongType Workspace::getSpilledSize() {
            return _spillsSize.load();
        }

        void* Workspace::allocateBytes(sd::memory::MemoryType type, sd::LongType numBytes) {
            switch (type) {
                case HOST: {
                        if (numBytes < 1)
                            throw allocation_exception::build("Number of [HOST] bytes for allocation should be positive", numBytes);


                        //numBytes += 32;
                        void* result = nullptr;
                        this->_cycleAllocationsSecondary += numBytes;
                        this->_mutexAllocation.lock();

                        if (_offsetSecondary.load() + numBytes > _currentSizeSecondary) {
                            sd_debug("Allocating %lld [HOST] bytes in spills\n", numBytes);
                            this->_mutexAllocation.unlock();

                            sd::Pointer p;
                            auto res = hipHostAlloc(reinterpret_cast<void **>(&p), numBytes, hipHostMallocDefault);
                            if (res != 0)
                                throw cuda_exception::build("Can't allocate [HOST] memory", res);

                            _mutexSpills.lock();
                            _spillsSecondary.push_back(p);
                            _mutexSpills.unlock();

                            _spillsSizeSecondary += numBytes;

                            return p;
                        }

                        result = (void *)(_ptrHost + _offsetSecondary.load());
                        _offsetSecondary += numBytes;
                        //memset(result, 0, (int) numBytes);

                        sd_debug("Allocating %lld bytes from [HOST] workspace; Current PTR: %p; Current offset: %lld\n", numBytes, result, _offset.load());

                        this->_mutexAllocation.unlock();

                        return result;
                    }
                    break;
                case DEVICE: {
                        if (numBytes < 1)
                            throw allocation_exception::build("Number of [DEVICE] bytes for allocation should be positive", numBytes);


                        //numBytes += 32;
                        void* result = nullptr;
                        this->_cycleAllocations += numBytes;
                        this->_mutexAllocation.lock();

                        if (_offset.load() + numBytes > _currentSize) {
                            sd_debug("Allocating %lld [DEVICE] bytes in spills\n", numBytes);
                            this->_mutexAllocation.unlock();

                            sd::Pointer p;
                            auto res = hipMalloc(reinterpret_cast<void **>(&p), numBytes);
                            if (res != 0)
                                throw cuda_exception::build("Can't allocate [DEVICE] memory", res);

                            _mutexSpills.lock();
                            _spills.push_back(p);
                            _mutexSpills.unlock();

                            _spillsSize += numBytes;

                            return p;
                        }

                        result = (void *)(_ptrDevice + _offset.load());
                        _offset += numBytes;
                        //memset(result, 0, (int) numBytes);

                        sd_debug("Allocating %lld bytes from [DEVICE] workspace; Current PTR: %p; Current offset: %lld\n", numBytes, result, _offset.load());

                        this->_mutexAllocation.unlock();

                        return result;
                    }
                    break;
                default:
                    throw std::runtime_error("Unknown MemoryType was passed in");
            }
        }

        Workspace* Workspace::clone() {
            // for clone we take whatever is higher: current allocated size, or allocated size of current loop
            return new Workspace(sd::math::sd_max<sd::LongType >(this->getCurrentSize(), this->_cycleAllocations.load()));
        }

        sd::LongType Workspace::getAllocatedSecondarySize() {
            return getCurrentSecondarySize() + getSpilledSecondarySize();
        }

        sd::LongType Workspace::getCurrentSecondarySize() {
            return _currentSizeSecondary;
        }

        sd::LongType Workspace::getCurrentSecondaryOffset() {
            return _offsetSecondary.load();
        }

        sd::LongType Workspace::getSpilledSecondarySize() {
            return _spillsSizeSecondary;
        }

        sd::LongType Workspace::getUsedSecondarySize() {
            return getCurrentSecondaryOffset();
        }

    }
}
