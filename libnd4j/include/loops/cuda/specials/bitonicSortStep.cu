#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 28.11.2018
//
#include <ops/specials_cuda.h>


//////////////////////////////////////////////////////////////////////////
template <typename X, typename Y>
SD_KERNEL void bitonicSortStepKernelKey(void *vx, sd::LongType const* xShapeInfo, void *vy, sd::LongType const* yShapeInfo, int j, int k, int length, bool descending) {

    auto x = static_cast<X*>(vx);
    auto y = static_cast<Y*>(vy);

    unsigned int i, ixj; /* Sorting partners: i and ixj */
    i = threadIdx.x + blockDim.x * blockIdx.x;

    __shared__ sd::LongType xLength;
    if (threadIdx.x == 0)
        xLength = shape::length(xShapeInfo);

    __syncthreads();


    if (i >= length)
        return;

    ixj = i^j;

    /* The threads with the lowest ids sort the array. */
    if ((ixj)>i) {
        int posI = shape::getIndexOffset(i, xShapeInfo);
        int posIXJ = shape::getIndexOffset(ixj, xShapeInfo);

        if ((i&k)==0) {
            /* Sort ascending */
            if (!descending == (x[posI]>x[posIXJ])) {
                /* exchange(i,ixj); */
                X temp = x[posI];
                x[posI] = x[posIXJ];
                x[posIXJ] = temp;

                Y ytemp = y[posI];
                y[posI] = y[posIXJ];
                y[posIXJ] = ytemp;
            }
        } else if ((i&k)!=0) {
            /* Sort descending */
            if (!descending == (x[posI]<x[posIXJ])) {
                /* exchange(i,ixj); */
                X temp = x[posI];
                x[posI] = x[posIXJ];
                x[posIXJ] = temp;

                Y ytemp = y[posI];
                y[posI] = y[posIXJ];
                y[posIXJ] = ytemp;
            }
        }
    }
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
SD_KERNEL void bitonicSortStepKernel(void *vx, sd::LongType const* xShapeInfo, int j, int k, int length, bool descending) {

    auto x = static_cast<T*>(vx);

    unsigned int i, ixj; /* Sorting partners: i and ixj */
    i = threadIdx.x + blockDim.x * blockIdx.x;

    __shared__ sd::LongType xLength;
    if (threadIdx.x == 0)
        xLength = shape::length(xShapeInfo);

    __syncthreads();


    if (i >= length)
        return;

    ixj = i^j;

    /* The threads with the lowest ids sort the array. */
    if ((ixj)>i) {
        int posI = shape::getIndexOffset(i, xShapeInfo);
        int posIXJ = shape::getIndexOffset(ixj, xShapeInfo);

        if ((i&k)==0) {
            /* Sort ascending */
            if (!descending == (x[posI]>x[posIXJ])) {
                /* exchange(i,ixj); */
                T temp = x[posI];
                x[posI] = x[posIXJ];
                x[posIXJ] = temp;
            }
        } else if ((i&k)!=0) {
            /* Sort descending */
            if (!descending == (x[posI]<x[posIXJ])) {
                /* exchange(i,ixj); */
                T temp = x[posI];
                x[posI] = x[posIXJ];
                x[posIXJ] = temp;
            }
        }
    }
}

//////////////////////////////////////////////////////////////////////////
template<typename T>
SD_HOST void bitonicSortStepGeneric(dim3 &launchDims, hipStream_t *stream, void *vx, sd::LongType const* xShapeInfo, int j, int k, int length, bool descending) {
    bitonicSortStepKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, xShapeInfo, j, k, length, descending);
}

//////////////////////////////////////////////////////////////////////////
template <typename X, typename Y>
SD_HOST void bitonicSortStepGenericKey(dim3 &launchDims, hipStream_t *stream, void *vx, sd::LongType const* xShapeInfo, void *vy, sd::LongType const* yShapeInfo, int j, int k, int length, bool descending) {
    bitonicSortStepKernelKey<X,Y><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, xShapeInfo, vy, yShapeInfo, j, k, length, descending);
}


BUILD_SINGLE_TEMPLATE(template void SD_LIB_HIDDEN bitonicSortStepGeneric, (dim3 &launchDims, hipStream_t *stream, void *vx, sd::LongType const* xShapeInfo, int j, int k, int length, bool descending), SD_COMMON_TYPES);
BUILD_DOUBLE_TEMPLATE(template void SD_LIB_HIDDEN bitonicSortStepGenericKey, (dim3 &launchDims, hipStream_t *stream, void *vx, sd::LongType const* xShapeInfo, void *vy, sd::LongType const* yShapeInfo, int j, int k, int length, bool descending), SD_COMMON_TYPES, SD_COMMON_TYPES);
