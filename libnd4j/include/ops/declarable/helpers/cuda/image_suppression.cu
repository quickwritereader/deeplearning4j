#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author sgazeos@gmail.com
//
#include <ops/declarable/helpers/image_suppression.h>
#include <array/NDArrayFactory.h>
#include <legacy/NativeOps.h>
#include <exceptions/cuda_exception.h>
#include <queue>

namespace sd {
namespace ops {
namespace helpers {
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// needToSuppressWithThreshold - predicate for suppression
//      boxes - boxes tensor buffer
//      boxesShape boxes tensor shape
//      previousIndex - index for current pos value
//      nextIndex - index for neighbor pos value
//      threshold - threashold value to suppress
//
//      return value: true, if threshold is overcome, false otherwise
//
    template <typename T>
    static SD_DEVICE bool needToSuppressWithThreshold(T* boxes, sd::LongType const* boxesShape, int previousIndex, int nextIndex, T threshold) {
        sd::LongType previous0[] = {previousIndex, 0};
        sd::LongType previous1[] = {previousIndex, 1};
        sd::LongType previous2[] = {previousIndex, 2};
        sd::LongType previous3[] = {previousIndex, 3};
        sd::LongType next0[] = {nextIndex, 0};
        sd::LongType next1[] = {nextIndex, 1};
        sd::LongType next2[] = {nextIndex, 2};
        sd::LongType next3[] = {nextIndex, 3};

        // we have rectangle with given max values. Compute vexes of rectangle first

        T minYPrev = sd::math::sd_min(boxes[shape::getOffset(boxesShape, previous0)], boxes[shape::getOffset(boxesShape, previous2)]);
        T minXPrev = sd::math::sd_min(boxes[shape::getOffset(boxesShape, previous1)], boxes[shape::getOffset(boxesShape, previous3)]);
        T maxYPrev = sd::math::sd_max(boxes[shape::getOffset(boxesShape, previous0)], boxes[shape::getOffset(boxesShape, previous2)]);
        T maxXPrev = sd::math::sd_max(boxes[shape::getOffset(boxesShape, previous1)], boxes[shape::getOffset(boxesShape, previous3)]);
        T minYNext = sd::math::sd_min(boxes[shape::getOffset(boxesShape, next0)],     boxes[shape::getOffset(boxesShape, next2)]);
        T minXNext = sd::math::sd_min(boxes[shape::getOffset(boxesShape, next1)],     boxes[shape::getOffset(boxesShape, next3)]);
        T maxYNext = sd::math::sd_max(boxes[shape::getOffset(boxesShape, next0)],     boxes[shape::getOffset(boxesShape, next2)]);
        T maxXNext = sd::math::sd_max(boxes[shape::getOffset(boxesShape, next1)],     boxes[shape::getOffset(boxesShape, next3)]);

        // compute areas for comparation
        T areaPrev = (maxYPrev - minYPrev) * (maxXPrev - minXPrev);
        T areaNext = (maxYNext - minYNext) * (maxXNext - minXNext);

        // of course, areas should be positive
        if (areaNext <= T(0.f) || areaPrev <= T(0.f)) return false;

        // compute intersection of rectangles
        T minIntersectionY = sd::math::sd_max(minYPrev, minYNext);
        T minIntersectionX = sd::math::sd_max(minXPrev, minXNext);
        T maxIntersectionY = sd::math::sd_min(maxYPrev, maxYNext);
        T maxIntersectionX = sd::math::sd_min(maxXPrev, maxXNext);
        T intersectionArea =
                sd::math::sd_max(T(maxIntersectionY - minIntersectionY), T(0.0f)) *
                sd::math::sd_max(T(maxIntersectionX - minIntersectionX), T(0.0f));
        T intersectionValue = intersectionArea / (areaPrev + areaNext - intersectionArea);
        // final check
        return intersectionValue > threshold;
    }

    template <typename T>
    static SD_DEVICE T similirityV3(T* boxes, sd::LongType const* boxesShape, int previousIndex, int nextIndex) {
        sd::LongType previous0[] = {previousIndex, 0};
        sd::LongType previous1[] = {previousIndex, 1};
        sd::LongType previous2[] = {previousIndex, 2};
        sd::LongType previous3[] = {previousIndex, 3};
        sd::LongType next0[] = {nextIndex, 0};
        sd::LongType next1[] = {nextIndex, 1};
        sd::LongType next2[] = {nextIndex, 2};
        sd::LongType next3[] = {nextIndex, 3};

        // we have rectangle with given max values. Compute vexes of rectangle first

        T minYPrev = sd::math::sd_min(boxes[shape::getOffset(boxesShape, previous0)], boxes[shape::getOffset(boxesShape, previous2)]);
        T minXPrev = sd::math::sd_min(boxes[shape::getOffset(boxesShape, previous1)], boxes[shape::getOffset(boxesShape, previous3)]);
        T maxYPrev = sd::math::sd_max(boxes[shape::getOffset(boxesShape, previous0)], boxes[shape::getOffset(boxesShape, previous2)]);
        T maxXPrev = sd::math::sd_max(boxes[shape::getOffset(boxesShape, previous1)], boxes[shape::getOffset(boxesShape, previous3)]);
        T minYNext = sd::math::sd_min(boxes[shape::getOffset(boxesShape, next0)],     boxes[shape::getOffset(boxesShape, next2)]);
        T minXNext = sd::math::sd_min(boxes[shape::getOffset(boxesShape, next1)],     boxes[shape::getOffset(boxesShape, next3)]);
        T maxYNext = sd::math::sd_max(boxes[shape::getOffset(boxesShape, next0)],     boxes[shape::getOffset(boxesShape, next2)]);
        T maxXNext = sd::math::sd_max(boxes[shape::getOffset(boxesShape, next1)],     boxes[shape::getOffset(boxesShape, next3)]);

        // compute areas for comparation
        T areaPrev = (maxYPrev - minYPrev) * (maxXPrev - minXPrev);
        T areaNext = (maxYNext - minYNext) * (maxXNext - minXNext);

        // of course, areas should be positive
        if (areaNext <= T(0.f) || areaPrev <= T(0.f)) return false;

        // compute intersection of rectangles
        T minIntersectionY = sd::math::sd_max(minYPrev, minYNext);
        T minIntersectionX = sd::math::sd_max(minXPrev, minXNext);
        T maxIntersectionY = sd::math::sd_min(maxYPrev, maxYNext);
        T maxIntersectionX = sd::math::sd_min(maxXPrev, maxXNext);
        T intersectionArea =
                sd::math::sd_max(T(maxIntersectionY - minIntersectionY), T(0.0f)) *
                sd::math::sd_max(T(maxIntersectionX - minIntersectionX), T(0.0f));
        T intersectionValue = intersectionArea / (areaPrev + areaNext - intersectionArea);
        // final check
        return intersectionValue;
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// shouldSelectKernel - compute status for all selected rectangles (boxes)
//
// we compute boolean flag as shared uint32 and return it on final only for the first thread
//
    template <typename T, typename I>
    static SD_KERNEL void shouldSelectKernel(T* boxesBuf, sd::LongType const* boxesShape, I* indexBuf, I* selectedIndicesData, double threshold, int numSelected, int i, bool* shouldSelect) {
        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = gridDim.x * blockDim.x;
        __shared__ unsigned int shouldSelectShared;
        if (threadIdx.x == 0) {
            shouldSelectShared = (unsigned int)shouldSelect[0];
        }
        __syncthreads();
        for (int j = numSelected - 1 - tid; j >= 0; j -= step) {
            if (shouldSelectShared) {
                if (needToSuppressWithThreshold(boxesBuf, boxesShape, indexBuf[i],
                                                                  indexBuf[selectedIndicesData[j]], T(threshold)))
                    atomicCAS(&shouldSelectShared, 1, 0); // exchange only when need to suppress
            }
        }
        __syncthreads();

        // final move: collect result
        if (threadIdx.x == 0) {
            *shouldSelect = shouldSelectShared > 0;
        }
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// indices - type depended, indicesLong - type defined (only 64bit integers)
//
    template <typename I>
    static SD_KERNEL void copyIndices(void* indices,  void* indicesLong, sd::LongType len) {
        I* indexBuf = reinterpret_cast<I*>(indices);
        sd::LongType* srcBuf = reinterpret_cast<sd::LongType*>(indicesLong);;

        auto tid = threadIdx.x + blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (auto i = tid; i < len; i += step)
            indexBuf[i] = (I)srcBuf[i];
    }

    template <typename T, typename I>
    static SD_KERNEL void suppressScores(T* scores, I* indices, sd::LongType length, T scoreThreshold) {
        auto start = blockIdx.x * blockDim.x;
        auto step = gridDim.x * blockDim.x;

        for (auto e = start + threadIdx.x; e < (int)length; e += step) {
            if (scores[e] < scoreThreshold) {
                scores[e] = scoreThreshold;
                indices[e] = -1;
            }
            else {
                indices[e] = I(e);
            }
        }
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// nonMaxSuppressionV2 algorithm - given from TF NonMaxSuppressionV2 implementation
//
    template <typename T, typename I>
    static void nonMaxSuppressionV2_(sd::LaunchContext* context, NDArray* boxes, NDArray* scales, int maxSize, double threshold, double scoreThreshold, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {boxes, scales});
        std::unique_ptr<NDArray> indices(NDArrayFactory::create_<I>('c', {scales->lengthOf()}, context)); // - 1, scales->lengthOf()); //, scales->getContext());

        NDArray scores(*scales);
        sd::Pointer extras[2] = {nullptr, stream};
        auto indexBuf = indices->dataBuffer()->specialAsT<I>();///reinterpret_cast<I*>(indices->specialBuffer());
        auto scoreBuf = scores.dataBuffer()->specialAsT<T>();
        suppressScores<T,I><<<128, 128, 128, *stream>>>(scoreBuf, indexBuf, scores.lengthOf(), T(scoreThreshold));
        indices->tickWriteDevice();
        sortByValue(extras, indices->buffer(), indices->shapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), scores.buffer(), scores.shapeInfo(), scores.specialBuffer(), scores.specialShapeInfo(), true);
        indices->tickWriteDevice();
        NDArray selectedIndices = NDArrayFactory::create<I>('c', {output->lengthOf()}, context);
        int numSelected = 0;
        int numBoxes = boxes->sizeAt(0);
        auto boxesBuf = reinterpret_cast<T*>(boxes->specialBuffer());

        auto selectedIndicesData = reinterpret_cast<I*>(selectedIndices.specialBuffer());
        auto outputBuf = reinterpret_cast<I*>(output->specialBuffer());

        bool* shouldSelectD;
        auto err = hipMalloc(&shouldSelectD, sizeof(bool));
        if (err) {
            throw cuda_exception::build("helpers::nonMaxSuppressionV2: Cannot allocate memory for bool flag", err);
        }
        for (I i = 0; i < boxes->sizeAt(0); ++i) {
            bool shouldSelect = numSelected < output->lengthOf();
            if (shouldSelect) {
                err = hipMemcpy(shouldSelectD, &shouldSelect, sizeof(bool), hipMemcpyHostToDevice);
                if (err) {
                    throw cuda_exception::build("helpers::nonMaxSuppressionV2: Cannot set up bool flag to device", err);
                }

                shouldSelectKernel<T,I><<<128, 256, 1024, *stream>>>(boxesBuf, boxes->specialShapeInfo(), indexBuf, selectedIndicesData, threshold, numSelected, i, shouldSelectD);
                err = hipMemcpy(&shouldSelect, shouldSelectD, sizeof(bool), hipMemcpyDeviceToHost);
                if (err) {
                    throw cuda_exception::build("helpers::nonMaxSuppressionV2: Cannot set up bool flag to host", err);
                }
            }

            if (shouldSelect) {
                hipMemcpy(reinterpret_cast<I*>(output->specialBuffer()) + numSelected, indexBuf + i, sizeof(I), hipMemcpyDeviceToDevice);
                hipMemcpy(selectedIndicesData + numSelected, &i, sizeof(I), hipMemcpyHostToDevice);
                numSelected++;
            }
        }

        err = hipFree(shouldSelectD);
        if (err) {
            throw cuda_exception::build("helpers::nonMaxSuppressionV2: Cannot deallocate memory for bool flag", err);
        }

    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template <typename T, typename I>
    static SD_DEVICE bool checkOverlapBoxes(T* boxes, sd::LongType const* shape, T* scores, I* indices, I* selectedIndices, I* startIndices, I selectedSize, I nextCandidateIndex, T overlapThreshold, T scoreThreshold, bool simple) {
        bool shouldHardSuppress = false;
        T& nextCandidateScore = scores[nextCandidateIndex];
        I selectedIndex = indices[nextCandidateIndex];
        I finish = startIndices[nextCandidateIndex];

        for (int j = selectedSize; j > finish; --j) {
            T boxVal;
            if (simple) {
                sd::LongType xPos[] = {selectedIndex, selectedIndices[j - 1]};
                auto xShift = shape::getOffset(shape, xPos, 0);
                boxVal = boxes[xShift];
            }
            else {
                boxVal = similirityV3(boxes, shape, selectedIndex, selectedIndices[j - 1]);
            }
            if (boxVal > static_cast<T>(overlapThreshold))
                nextCandidateScore = static_cast<T>(0.f);

            // First decide whether to perform hard suppression
            if (boxVal >= overlapThreshold) {
                shouldHardSuppress = true;
                break;
            }

            // If nextCandidate survives hard suppression, apply soft suppression
            if (nextCandidateScore <= static_cast<T>(scoreThreshold)) break;
        }

        return shouldHardSuppress;
    }
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template <typename T, typename I>
    static SD_KERNEL void
    suppressNonMaxOverlapKernel(T* boxes, sd::LongType const* boxesShape, T* scoresData, I* indices, I* startIndices, sd::LongType length, I maxOutputLen,
    T overlapThreshold, T scoreThreshold, I* output, sd::LongType const* outputShape, I* outputLength, bool simple) {

        __shared__ I selectedSize;
        __shared__ I* tempOutput;

        if (threadIdx.x == 0) {
            selectedSize = outputLength?*outputLength:maxOutputLen;
            extern __shared__ unsigned char shmem[];
            tempOutput = (I*)shmem;
        }
        __syncthreads();

        auto start = blockIdx.x * blockDim.x;
        auto step = blockDim.x * gridDim.x;

        for (I nextCandidateIndex = start + threadIdx.x; selectedSize < maxOutputLen && nextCandidateIndex < (I)length; ) {
            auto originalScore = scoresData[nextCandidateIndex];//nextCandidate._score;
            I nextCandidateBoxIndex = indices[nextCandidateIndex];
            auto selectedSizeMark = selectedSize;

            // skip for cases when index is less than 0 (under score threshold)
            if (nextCandidateBoxIndex < 0) {
                nextCandidateIndex += step;
                continue;
            }
            // check for overlaps
            bool shouldHardSuppress = checkOverlapBoxes(boxes, boxesShape, scoresData, indices, tempOutput, startIndices, selectedSize,
                    nextCandidateIndex, overlapThreshold, scoreThreshold, simple);//false;
            T nextCandidateScore = scoresData[nextCandidateIndex];

            startIndices[nextCandidateIndex] = selectedSize;
            if (!shouldHardSuppress) {
                if (nextCandidateScore == originalScore) {
                    // Suppression has not occurred, so select nextCandidate
                    if (output)
                        output[selectedSize] = nextCandidateBoxIndex;
                    tempOutput[selectedSize] = nextCandidateBoxIndex;
                    math::atomics::sd_atomicAdd(&selectedSize, (I)1);
                }

                if (nextCandidateScore > scoreThreshold) {
                    // Soft suppression has occurred and current score is still greater than
                    // scoreThreshold; add nextCandidate back onto priority queue.
                    continue;  // in some cases, this index not 0
                }
            }
            nextCandidateIndex += step;
        }

        if (threadIdx.x == 0) {
            if (outputLength)
                *outputLength = selectedSize;
        }
    }

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    template <typename T, typename I>
    static sd::LongType
    nonMaxSuppressionGeneric_(sd::LaunchContext* context, NDArray* boxes, NDArray* scores, int outputSize,
                              double overlapThreshold, double scoreThreshold, NDArray* output, bool simple) {
        auto stream = context->getCudaStream();
        if (output)
            NDArray::prepareSpecialUse({output}, {boxes, scores});
        else {
            if (!boxes->isActualOnDeviceSide())
                boxes->syncToDevice();
            if (!scores->isActualOnDeviceSide())
                scores->syncToDevice();
        }

        NDArray indices = NDArrayFactory::create<I>('c', {scores->lengthOf()}, context); // - 1, scales->lengthOf()); //, scales->getContext());
        NDArray startPositions = NDArrayFactory::create<I>('c', {scores->lengthOf()}, context);
        NDArray selectedScores(*scores);
        sd::Pointer extras[2] = {nullptr, stream};
        auto indexBuf = indices.dataBuffer()->specialAsT<I>();///reinterpret_cast<I*>(indices->specialBuffer());

        suppressScores<<<128, 128, 128, *stream>>>(selectedScores.dataBuffer()->specialAsT<T>(), indexBuf, selectedScores.lengthOf(), T(scoreThreshold));

        sortByValue(extras, indices.buffer(), indices.shapeInfo(), indices.specialBuffer(), indices.specialShapeInfo(), selectedScores.buffer(), selectedScores.shapeInfo(), selectedScores.specialBuffer(), selectedScores.specialShapeInfo(), true);
        indices.tickWriteDevice();
        selectedScores.tickWriteDevice();

        auto scoresData = selectedScores.dataBuffer()->specialAsT<T>();//, numBoxes, scoresData.begin());

        auto startIndices = startPositions.dataBuffer()->specialAsT<I>();
        I selectedSize = 0;
        sd::LongType res = 0;
        if (output) { // this part used when output shape already calculated to fill up values on output
            DataBuffer selectedSizeBuf(&selectedSize, sizeof(I), DataTypeUtils::fromT<I>());
            suppressNonMaxOverlapKernel<<<1, 1, 1024, *stream >>> (boxes->dataBuffer()->specialAsT<T>(),
                    boxes->specialShapeInfo(), scoresData, indexBuf, startIndices, scores->lengthOf(), (I) outputSize,
                    T(overlapThreshold), T(scoreThreshold), output->dataBuffer()->specialAsT<I>(), output->specialShapeInfo(),
                    selectedSizeBuf.specialAsT<I>(), simple);
        }
        else { // this case used on calculation of output shape. Output and output shape shoulde be nullptr.
            DataBuffer selectedSizeBuf(&selectedSize, sizeof(I), DataTypeUtils::fromT<I>());
            suppressNonMaxOverlapKernel<<<1, 1, 1024, *stream >>> (boxes->dataBuffer()->specialAsT<T>(),
                    boxes->specialShapeInfo(), scoresData, indexBuf, startIndices, scores->lengthOf(), (I)outputSize,
                    T(overlapThreshold), T(scoreThreshold), (I*)nullptr, (sd::LongType*) nullptr, selectedSizeBuf.specialAsT<I>(), simple);
            selectedSizeBuf.syncToPrimary(context, true);
            res = *selectedSizeBuf.primaryAsT<I>();
        }

        if (output)
            NDArray::registerSpecialUse({output}, {boxes, scores});

        return res;
    }
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    void nonMaxSuppression(sd::LaunchContext * context, NDArray* boxes, NDArray* scales, int maxSize, double threshold, double scoreThreshold, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(boxes->dataType(), output->dataType(), nonMaxSuppressionV2_,
                (context, boxes, scales, maxSize, threshold, scoreThreshold, output),
                SD_FLOAT_TYPES, SD_INDEXING_TYPES);
    }
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    sd::LongType nonMaxSuppressionGeneric(sd::LaunchContext * context, NDArray* boxes, NDArray* scales, int maxSize, double threshold, double scoreThreshold, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(boxes->dataType(), output ? output->dataType():DataType::INT32, return nonMaxSuppressionGeneric_,
                              (context, boxes, scales, maxSize, threshold, scoreThreshold, output, true),
                              SD_FLOAT_TYPES, SD_INDEXING_TYPES);
        return boxes->sizeAt(0);
    }

    sd::LongType
    nonMaxSuppressionV3(sd::LaunchContext* context, NDArray* boxes, NDArray* scores, int maxSize,
                             double overlapThreshold, double scoreThreshold, NDArray* output) {
        BUILD_DOUBLE_SELECTOR(boxes->dataType(), output ? output->dataType():DataType::INT32, return nonMaxSuppressionGeneric_,
                              (context, boxes, scores, maxSize, overlapThreshold, scoreThreshold, output, false),
                              SD_FLOAT_TYPES, SD_INDEXING_TYPES);
        return boxes->sizeAt(0);
    }

}
}
}
