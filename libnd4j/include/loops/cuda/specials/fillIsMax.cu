/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//
#include <loops/special_kernels.h>

namespace sd {

////////////////////////////////////////////////////////////////////////
    template <typename T>
    SD_KERNEL void execFillIsMax(void *vdZ, const sd::LongType *xShapeInfo, sd::LongType length, long idx) {
        auto dz = reinterpret_cast<T*>(vdZ);
        int tid = blockIdx.x * blockDim.x + threadIdx.x;

        for (sd::LongType i = tid; i < length; i += blockDim.x * gridDim.x)
            dz[shape::getIndexOffset(i, xShapeInfo)] = (i == idx ? (T) 1 : (T) 0);
    }

////////////////////////////////////////////////////////////////////////
    template <typename T>
    SD_HOST void fillIsMaxGeneric(dim3 &launchDims, hipStream_t *stream, void *dx, const sd::LongType *xShapeInfo, sd::LongType length, long idx) {
        execFillIsMax<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(dx, xShapeInfo, length, idx);
        sd::DebugHelper::checkErrorCode(stream, "fillIsMax(...) failed");
    }


    BUILD_SINGLE_TEMPLATE(template void SD_LIB_HIDDEN fillIsMaxGeneric, (dim3& launchDims, hipStream_t *stream, void* dz, const sd::LongType *zShapeInfo, sd::LongType length, long idx), SD_COMMON_TYPES);
}