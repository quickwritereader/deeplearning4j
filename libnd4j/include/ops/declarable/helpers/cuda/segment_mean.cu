#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
//  @author GS <sgazeos@gmail.com>
//
#include <ops/declarable/helpers/segment.h>
#include <ops/declarable/helpers/segment_common.h>
#include <array/NDArrayFactory.h>
#include <helpers/ShapeUtils.h>
#include <helpers/TAD.h>
#include <exceptions/cuda_exception.h>
#include <helpers/PointersManager.h>
#include <helpers/ConstantTadHelper.h>

namespace sd {
namespace ops {
namespace helpers {
    // -------------------------------------------------------------------------------------------------------------- //
    // Segment ops linear kernels
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static SD_KERNEL void segmentMeanLinearKernel(void* input, sd::LongType const*  inputShape, int* starts, int* lengths, sd::LongType numOfClasses, void* output, sd::LongType const*  outputShape) {
        __shared__ T* val;
        __shared__ sd::LongType xLen, zLen, segment, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;

        if (threadIdx.x == 0) {
            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
            segment = blockIdx.x / threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
//            extern __shared__ unsigned char shmem[];
//            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

            //[zIndex] =
            if (segment < numOfClasses) {
                zIndex = shape::getIndexOffset(segment, outputShape);
                start = starts[segment];
                finish = start + lengths[segment];
                //val[segment] = ;
                z[zIndex] = T(x[shape::getIndexOffset(start, inputShape)] / lengths[segment]);
//                val[segment] = z[zIndex];
            }

        }
        __syncthreads();

        for (auto e = start + threadIdx.x + 1; e < finish; e += blockDim.x) {
            auto xIndex = shape::getIndexOffset(e, inputShape);
            if (lengths[segment])
                sd::math::atomics::sd_atomicAdd(&z[zIndex], T(x[xIndex] / lengths[segment]));
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static SD_KERNEL void unsortedSegmentMeanLinearKernel(void* input, sd::LongType const*  inputShape, void* indices, sd::LongType const*  indicesShape, int* starts, int* lengths, sd::LongType numOfClasses, void* output, sd::LongType const*  outputShape) {
        __shared__ T* val;
        __shared__ sd::LongType xLen, zLen, zIndex;
        __shared__ T* x;
        __shared__ T* z;
        __shared__ I* y; //int threadsPerSegment, start, finish;
        auto segment = blockIdx.x;// /
        if (threadIdx.x == 0) {
//            threadsPerSegment = (gridDim.x + numOfClasses - 1) / numOfClasses;
//            threadsPerSegment;
            x = reinterpret_cast<T*>(input);
            z = reinterpret_cast<T*>(output);
            y = reinterpret_cast<I*>(indices);
//            extern __shared__ unsigned char shmem[];
//            val = reinterpret_cast<T*>(shmem);
            xLen = shape::length(inputShape);
            zLen = shape::length(outputShape);

//            if (segment < numOfClasses) {
            zIndex = shape::getIndexOffset(segment, outputShape);
            //start = starts[segment];
            //finish = start + lengths[segment];
            if (lengths[segment] > 0)
                z[zIndex] = T(x[shape::getIndexOffset(starts[segment], inputShape)] / T(lengths[segment]));
            else
                z[zIndex] = 0; //DataTypeUtils::max<T>();
//                val[segment] = z[zIndex];
//            }

        }
        __syncthreads();
        if (lengths[segment] > 0)
            for (auto e = threadIdx.x; e < xLen; e += blockDim.x) {
                auto xIndex = shape::getIndexOffset(e, inputShape);
                auto yIndex = shape::getIndexOffset(e, indicesShape);
                if (y[yIndex] == segment && e != starts[segment]) {
                    sd::math::atomics::sd_atomicAdd(&z[zIndex], T(x[xIndex]/T(lengths[segment])));
                }
            }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    // SegmentMean kernel
    template <typename T, typename I>
    static SD_KERNEL void segmentMeanTadKernel(void* inputBuf, sd::LongType const*  inputShape, sd::LongType const*  inputTads, sd::LongType const*  inputTadOffsets, I* indices, int* starts, int* lengths, sd::LongType numOfClasses, void* outputBuf, sd::LongType const*  outputShape, sd::LongType const*  outputTads, sd::LongType const*  outputTadOffsets) {
        __shared__ T* val;
        __shared__ sd::LongType len, zIndex, total;
        __shared__ T* z;
        __shared__ int threadsPerSegment, start, finish;
        auto segment = indices[blockIdx.x]; // / threadsPerSegment;

        if (threadIdx.x == 0) {
            z = reinterpret_cast<T*>(outputBuf) + outputTadOffsets[segment];
            len = shape::length(inputTads);
            start = starts[segment];
            finish = start + lengths[segment];
            total = shape::sizeAt(inputShape, 0);

        }
        __syncthreads();

        auto idx = blockIdx.x;
        if (blockIdx.x <= total) {
            auto x = reinterpret_cast<T *>(inputBuf) + inputTadOffsets[idx];
            if (blockIdx.x == start) {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads);
                    auto zIndex = shape::getIndexOffset(e, outputTads);
                    sd::math::atomics::sd_atomicAdd(&z[zIndex], T(x[xIndex]/lengths[segment]));
                }
            }
            else {
                for (auto e = threadIdx.x; e < len; e += blockDim.x) {
                    auto xIndex = shape::getIndexOffset(e, inputTads);
                    auto zIndex = shape::getIndexOffset(e, outputTads);
                    if (lengths[segment])
                        sd::math::atomics::sd_atomicAdd(&z[zIndex], T(x[xIndex]/lengths[segment]));
                }
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    // segmen mean
    template <typename T, typename I>
    static void segmentMeanFunctor_(LaunchContext* context, NDArray* input, NDArray* indices, NDArray* output) {
        auto stream = context->getCudaStream();
        sd::LongType numClasses = indices->e<sd::LongType>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses}, context);
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses}, context);

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        NDArray::prepareSpecialUse({output}, {input, indices});
        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());
        fillUpSegments(indices, numClasses, classesRangesBegs, classesRangesLens);

        if (input->isVector()) {
            segmentMeanLinearKernel<T,I><<<numClasses, input->lengthOf(), numClasses * 32 + 32, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            auto inputTads = packX.specialShapeInfo();
            auto inputTadOffsets = packX.specialOffsets();
            auto outputTads = packZ.specialShapeInfo();
            auto outputTadOffsets = packZ.specialOffsets();
            segmentMeanTadKernel<T,I><<<input->sizeAt(0), 512, 2048, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices});

    }
    // -------------------------------------------------------------------------------------------------------------- //
    void segmentMeanFunctor(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), segmentMeanFunctor_, (context, input, indices, output), SD_NUMERIC_TYPES, SD_INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices});
    }

    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static void unsortedSegmentMeanFunctor_(sd::LaunchContext* context, NDArray* input, NDArray* indices, sd::LongType numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
//        NDArray classes = NDArrayFactory::create<int>('c', {numOfClasses, 2});

        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numOfClasses}, context);
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numOfClasses}, context);
//        NDArray row = NDArrayFactory::create<int>('c', {1, 2}, {(int)indices->lengthOf(), (int)0});
//        classes.applyTrueBroadcast(sd::BroadcastOpsTuple::Assign(), &row, &classes);
        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numOfClasses, indices->lengthOf(), numOfClasses * 32 + 32);
//        int* classesBuf = reinterpret_cast<int*>(classes.specialBuffer());
        fillUpSegments(indices, numOfClasses, classesRangesBegs, classesRangesLens);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());

        if (input->isVector()) {
            unsortedSegmentMeanLinearKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            output->assign(0);
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
            sd::LongType const*  inputTads = packX.specialShapeInfo();
            sd::LongType const*  inputTadOffsets = packX.specialOffsets();
            sd::LongType const*  outputTads = packZ.specialShapeInfo();
            sd::LongType const*  outputTadOffsets = packZ.specialOffsets();
            dims.x = input->sizeAt(0);
            segmentMeanTadKernel<T,I><<<dims.x, dims.y, dims.z, *stream>>>(input->specialBuffer(), input->specialShapeInfo(), inputTads, inputTadOffsets, reinterpret_cast<I*>(indices->specialBuffer()), begins, lengths, numOfClasses, output->specialBuffer(), output->specialShapeInfo(), outputTads, outputTadOffsets);
        }

    }
    // -------------------------------------------------------------------------------------------------------------- //
    void unsortedSegmentMeanFunctor(sd::LaunchContext* context , NDArray* input, NDArray* indices, sd::LongType numOfClasses, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices});
        BUILD_DOUBLE_SELECTOR(input->dataType(), indices->dataType(), unsortedSegmentMeanFunctor_, (context, input, indices, numOfClasses, output),
                              SD_NUMERIC_TYPES, SD_INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices});
    }

    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static SD_KERNEL void segmentMeanBPLinearKernel(void* inputBuf, sd::LongType const*  inputShape, void* eps, sd::LongType const*  epsShape, void* indicesBuf, sd::LongType const*  indicesShape,
                                                     int* lengths, void* outputBuf, sd::LongType const*  outputShape) {
        __shared__ T* x;
        __shared__ T* gradIn;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ sd::LongType xLen, gradLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            gradOut = reinterpret_cast<T*>(eps);
            gradLen = shape::length(epsShape);
        }
        __syncthreads();

        auto start = blockIdx.x * blockDim.x + threadIdx.x;
        auto step = gridDim.x * blockDim.x;

        for (auto e = start; e < xLen; e += step) {

            auto zOffset = shape::getIndexOffset(e, outputShape);
            auto xOffset = shape::getIndexOffset(e, inputShape);
            auto yOffset = shape::getIndexOffset(e, indicesShape);
            auto classIndex = y[yOffset];
            auto gradOffsetO = shape::getIndexOffset(classIndex, epsShape);

            z[zOffset] = T(gradOut[gradOffsetO] / float(lengths[classIndex]));
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    template <typename T, typename I>
    static SD_KERNEL void segmentMeanBPTadKernel(void* inputBuf, sd::LongType const*  inputShape, void* eps, sd::LongType const*  epsShape,
                                                  void* indicesBuf, sd::LongType const*  indicesShape, int* lengths, void* outputBuf, sd::LongType const*  outputShape,sd::LongType const*  inputTad,
                                                  sd::LongType const*  inputOffsets, sd::LongType const*  gradOutTad, sd::LongType const*  gradOutOffsets, sd::LongType const*  outTad, sd::LongType const*  outOffsets) {
        __shared__ T* x;
        __shared__ T* gradOut;
        __shared__ I* y;
        __shared__ T* z;
        __shared__ sd::LongType xLen, yLen, gradLen, currentLen;

        if (threadIdx.x == 0) {
            xLen = shape::length(inputShape);
            x = reinterpret_cast<T*>(inputBuf);
            y = reinterpret_cast<I*>(indicesBuf);
            z = reinterpret_cast<T*>(outputBuf);
            yLen = shape::length(indicesShape);
            gradOut = reinterpret_cast<T*>(eps);
            gradLen = shape::length(epsShape);
            currentLen = shape::length(outTad);
        }
        __syncthreads();

        for (auto i = blockIdx.x; i < yLen; i += gridDim.x) {
//            auto yIndex = shape::getIndexOffset(i, indicesShape);
            auto segment = y[i]; //yIndex];
            T* currentOut = z + outOffsets[i];
            T* outGrad = gradOut + gradOutOffsets[segment];

            for (auto e = threadIdx.x; e < currentLen; e += blockDim.x) {
                auto zIndex = shape::getIndexOffset(e, outTad);
                auto gradIndex = shape::getIndexOffset(e, gradOutTad);
                if (lengths[segment] > 0)
                    currentOut[zIndex] = T(outGrad[gradIndex] / float(lengths[segment]));
            }
        }
    }
    // -------------------------------------------------------------------------------------------------------------- //
    // backrop for mean
    template <typename T, typename I>
    sd::Status segmentMeanFunctorBP_(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        auto numClasses = indices->e<int>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses}, context);
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses}, context);

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        fillUpSegments(indices, numClasses, classesRangesBegs, classesRangesLens);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());

        if (input->isVector()) {
            sd::LongType loop_size = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<sd::LongType>(loop_size - 1);
            segmentMeanBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(),
                    input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), lengths, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
//            auto packGradIn = sd::ConstantTadHelper::getInstance().tadForDimensions(tempRes.shapeInfo(), dimensions);
            auto packGradOut = sd::ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
            sd::LongType const*  inputTads = packX.specialShapeInfo();
            sd::LongType const*  inputTadOffsets = packX.specialOffsets();
            sd::LongType const*  outputTads = packZ.specialShapeInfo();
            sd::LongType const*  outputTadOffsets = packZ.specialOffsets();
            sd::LongType const*  gradOutTads = packGradOut.specialShapeInfo();
            sd::LongType const*  gradOutTadOffsets = packGradOut.specialOffsets();

            segmentMeanBPTadKernel<T,I><<<indices->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    gradOut->specialBuffer(), gradOut->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), lengths,
                    output->specialBuffer(), output->specialShapeInfo(), inputTads, inputTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
        return sd::Status::OK;
    }
    // -------------------------------------------------------------------------------------------------------------- //
    // segmen mean bp main
    sd::Status segmentMeanFunctorBP(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return segmentMeanFunctorBP_, (context, input,
                indices, gradOut, output), SD_FLOAT_TYPES, SD_INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
    }
    // -------------------------------------------------------------------------------------------------------------- //

    template <typename T, typename I>
    static sd::Status unsortedSegmentMeanFunctorBP_(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, sd::LongType numOfClasses, NDArray* output) {
        auto stream = context->getCudaStream();
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        auto numClasses = indices->e<int>(indices->lengthOf() - 1) + 1;
        NDArray classesRangesLens = NDArrayFactory::create<int>('c', {numClasses}, context);
        NDArray classesRangesBegs = NDArrayFactory::create<int>('c', {numClasses}, context);

        classesRangesBegs.assign(indices->lengthOf());
        classesRangesLens.assign(0);
        dim3 dims(numClasses, indices->lengthOf(), numClasses * 32 + 32);
        fillUpSegments(indices, numClasses, classesRangesBegs, classesRangesLens);
        int* begins = reinterpret_cast<int*>(classesRangesBegs.specialBuffer());
        int* lengths = reinterpret_cast<int*>(classesRangesLens.specialBuffer());

        if (input->isVector()) {
            sd::LongType loop_size = input->lengthOf();
            auto numOfClasses = gradOut->lengthOf(); //indices->e<sd::LongType>(loop_size - 1);
            segmentMeanBPLinearKernel<T,I><<<gradOut->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(),
                    input->specialShapeInfo(), gradOut->specialBuffer(), gradOut->specialShapeInfo(),
                    indices->specialBuffer(), indices->specialShapeInfo(), lengths, output->specialBuffer(), output->specialShapeInfo());
        }
        else {
            std::vector<int> dimensions = ShapeUtils::evalDimsToExclude(input->rankOf(), {0});
            auto packX = sd::ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), dimensions);
            auto packZ = sd::ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), dimensions);
//            auto packGradIn = sd::ConstantTadHelper::getInstance().tadForDimensions(tempRes.shapeInfo(), dimensions);
            auto packGradOut = sd::ConstantTadHelper::getInstance().tadForDimensions(gradOut->shapeInfo(), dimensions);
            sd::LongType const*  inputTads = packX.specialShapeInfo();
            sd::LongType const*  inputTadOffsets = packX.specialOffsets();
            sd::LongType const*  outputTads = packZ.specialShapeInfo();
            sd::LongType const*  outputTadOffsets = packZ.specialOffsets();
            sd::LongType const*  gradOutTads = packGradOut.specialShapeInfo();
            sd::LongType const*  gradOutTadOffsets = packGradOut.specialOffsets();

            segmentMeanBPTadKernel<T,I><<<indices->lengthOf(), input->lengthOf(), 256, *stream>>>(input->specialBuffer(), input->specialShapeInfo(),
                    gradOut->specialBuffer(), gradOut->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(), lengths,
                    output->specialBuffer(), output->specialShapeInfo(), inputTads, inputTadOffsets, gradOutTads, gradOutTadOffsets,
                    outputTads, outputTadOffsets);
        }
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
        return sd::Status::OK;
    }
    // -------------------------------------------------------------------------------------------------------------- //
    sd::Status unsortedSegmentMeanFunctorBP(sd::LaunchContext* context , NDArray* input, NDArray* indices, NDArray* gradOut, sd::LongType numOfClasses, NDArray* output) {
        NDArray::prepareSpecialUse({output}, {input, indices, gradOut});
        BUILD_DOUBLE_SELECTOR(output->dataType(), indices->dataType(), return unsortedSegmentMeanFunctorBP_, (context, input, indices, gradOut, numOfClasses, output), SD_FLOAT_TYPES, SD_INDEXING_TYPES);
        NDArray::registerSpecialUse({output}, {input, indices, gradOut});
    }

}
}
}