#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Oleh Semeniv (oleg.semeniv@gmail.com)
//
#include <system/op_boilerplate.h>
#include <ops/declarable/helpers/updatersHelpers.h>
#include <helpers/PointersManager.h>
#include <math/platformmath.h>
#include <math/templatemath.h>

namespace sd    {
namespace ops     {
namespace helpers {

///////////////////////////////////////////////////////////////////
template<typename T>
SD_KERNEL void adaGradUpdaterCuda(const void* vx, const sd::LongType* xShapeInfo, const void* vin, const sd::LongType* inShapeInfo, 
                                   void* vz, const sd::LongType* zShapeInfo, void* vst, const sd::LongType* stShapeInfo,
                                   const T lr, const T epsilon) {

    const auto x = reinterpret_cast<const T*>(vx);
    const auto init = reinterpret_cast<const T*>(vin);
    
    auto up = reinterpret_cast<T*>(vz);
    auto st = reinterpret_cast<T*>(vst);

    __shared__ bool bEWS, bOrdering, bXZsame, bXInSame, bXStSame;
    __shared__ sd::LongType xLen;
    
    if (threadIdx.x == 0) {
        xLen = shape::length(xShapeInfo);
        
        bEWS =  1 == shape::elementWiseStride(xShapeInfo) && 1 == shape::elementWiseStride(zShapeInfo) &&
                1 == shape::elementWiseStride(stShapeInfo) && 1 == shape::elementWiseStride(inShapeInfo);
        bOrdering = shape::order(xShapeInfo) == shape::order(zShapeInfo) && shape::order(xShapeInfo) == shape::order(stShapeInfo) &&
                    shape::order(xShapeInfo) == shape::order(inShapeInfo);

        bXZsame = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
        bXInSame = shape::haveSameShapeAndStrides(xShapeInfo, inShapeInfo);
        bXStSame = shape::haveSameShapeAndStrides(xShapeInfo, stShapeInfo);
    }
    __syncthreads();

    int coords[SD_MAX_RANK];

    for (sd::LongType i = blockIdx.x * blockDim.x + threadIdx.x; i < xLen; i += gridDim.x * blockDim.x) {

        auto xOffset = i, zOffset = i, initOffset = i, stOffset = i;

        if (!bEWS || !bOrdering) {

            shape::index2coords(i, xShapeInfo, coords);
            xOffset  = shape::getOffset(xShapeInfo, coords);
            zOffset  = bXZsame ? xOffset : shape::getOffset(zShapeInfo, coords);
            initOffset = bXInSame ? xOffset : shape::getOffset(inShapeInfo, coords);
            stOffset = bXStSame ? xOffset : shape::getOffset(stShapeInfo, coords);
        }

        st[stOffset] = init[initOffset] + x[xOffset] * x[xOffset];
        up[zOffset] = (lr * x[xOffset]) / (math::sd_sqrt<T, T>(st[stOffset]) + epsilon);

    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
void adaGradUpdaterCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t* stream, 
                                        const void* vx, const sd::LongType* xShapeInfo, const void* vin, const sd::LongType* inShapeInfo, 
                                        void* vz, const sd::LongType* zShapeInfo, void* vst, const sd::LongType* stShapeInfo,
                                        const double dLr, const double dEpsilon) {
    
    const T lr = static_cast<T>(dLr);
    const T epsilon = static_cast<T>(dEpsilon);

    adaGradUpdaterCuda<T><<<blocksPerGrid, threadsPerBlock, 256, * stream>>>(vx, xShapeInfo, vin, inShapeInfo,
                                         vz, zShapeInfo, vst, stShapeInfo, lr, epsilon);
}

///////////////////////////////////////////////////////////////////
void updaterAdaGrad(sd::LaunchContext* context, const NDArray& gradient, const NDArray& initState, 
                    NDArray& update, NDArray& stateH, const double dLr, const double dEpsilon) {

    PointersManager manager(context, "adaGradUpdater");

    const int threadsPerBlock = SD_MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (gradient.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

    NDArray::prepareSpecialUse({ &update, &stateH }, { &gradient, &initState });
    BUILD_SINGLE_SELECTOR(gradient.dataType(), adaGradUpdaterCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), 
                          gradient.specialBuffer(), gradient.specialShapeInfo(),
                          initState.specialBuffer(), initState.specialShapeInfo(),
                          update.specialBuffer(), update.specialShapeInfo(),
                          stateH.specialBuffer(), stateH.specialShapeInfo(), dLr, dEpsilon), SD_FLOAT_TYPES);
    NDArray::registerSpecialUse({ &update, &stateH }, { &gradient, &initState });

    manager.synchronize();
}

}
}
}
