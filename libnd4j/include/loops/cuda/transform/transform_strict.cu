#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//
#include <system/Environment.h>
#include <loops/transform_strict.h>
#include <types/types.h>
#include <system/op_boilerplate.h>
#include <loops/legacy_ops.h>
#include <helpers/DebugHelper.h>

using namespace simdOps;

template <typename X, typename OpType>
SD_KERNEL void transformStrictSimple(const void *x, const sd::LongType *xShapeInfo, int xRank,
                                      void *params,
                                      void *z, const sd::LongType *zShapeInfo, int zRank,
                                      int *allocationPointer,
                                      void *reductionPointer,
                                      const sd::LongType *tadShapeInfo, const sd::LongType *tadOffsets) {

    functions::transform::TransformStrict<X>::template transformCuda<OpType>(x,xShapeInfo,params,z,zShapeInfo,allocationPointer,reductionPointer,tadShapeInfo, tadOffsets);
}


namespace functions {
    namespace transform {

        template<typename X>
        SD_HOST void TransformStrict<X>::executeTransformShaped(dim3 launchDims, hipStream_t *stream,
                                                                const int opNum,
                                                                const void *x, const sd::LongType *xShape, int xRank,
                                                                void *extraParams,
                                                                void *z, const sd::LongType *zShape, int zRank,
                                                                int *allocationPointer, void *reductionPointer,
                                                                const sd::LongType *tadShapeInfo, const sd::LongType *tadOffsets) {
            DISPATCH_BY_OPNUM_T(intermediateShaped, PARAMS(launchDims, stream, x, xShape, xRank, extraParams, z, zShape, zRank, allocationPointer, reductionPointer, tadShapeInfo, tadOffsets), TRANSFORM_STRICT_OPS);

            DEBUG_KERNEL(stream, opNum);
        }


        template<typename X>
        template <typename OpType>
        SD_DEVICE void TransformStrict<X>::transformCuda(const void *vx, const sd::LongType *xShapeInfo,
                                                        void *vparams,
                                                        void *vz, const sd::LongType *zShapeInfo,
                                                        int *allocationPointer, void *vreductionPointer,
                                                          const sd::LongType *tadShapeInfo, const sd::LongType *tadOffsets) {

            auto x = static_cast<const X*>(vx);
            auto z = static_cast<X*>(vz);
            auto params = static_cast<X*>(vparams);
            auto reductionPointer = static_cast<X*>(vreductionPointer);


            if(OpType::requiresSpecial) {
                OpType::execSpecialCuda(x,xShapeInfo,z,zShapeInfo,params, allocationPointer, reductionPointer, tadShapeInfo, tadOffsets);
                return;
            }
            else {
                __shared__ sd::LongType xEws;
                __shared__ sd::LongType zEws;
                __shared__ char xOrder;
                __shared__ char zOrder;
                __shared__ sd::LongType length;

                if (threadIdx.x == 0) {

                    xEws = shape::elementWiseStride(xShapeInfo);
                    zEws = shape::elementWiseStride(zShapeInfo);
                    xOrder = shape::order(xShapeInfo);
                    zOrder = shape::order(zShapeInfo);
                    length = shape::length(xShapeInfo);
                }
                __syncthreads();

                auto tid = blockIdx.x * blockDim.x + threadIdx.x;
                int totalThreads = gridDim.x * blockDim.x;

                if(xEws > 0 && zEws > 0 && xOrder == zOrder && xOrder == 'c') {

                    for (int i = tid; i < length; i += totalThreads)
                        z[i * zEws] = OpType::op(x[i * xEws], params);
                }
                else {
                    if(vx == vz) {
                        for (sd::LongType i = tid; i < length; i+= totalThreads) {
                            auto xOffset = shape::getIndexOffset(i, xShapeInfo);
                            z[xOffset] = OpType::op(x[xOffset], params);
                        }
                    }
                    else {
                        for (sd::LongType i = tid; i < length; i+= totalThreads) {
                            auto xOffset = shape::getIndexOffset(i, xShapeInfo);
                            auto zOffset = shape::getIndexOffset(i, zShapeInfo);
                            z[zOffset] = OpType::op(x[xOffset], params);
                        }
                    }
                }
              }
        };

        template<typename X>
        template <typename OpType>
        SD_HOST void TransformStrict<X>::intermediateShaped(dim3 launchDims, hipStream_t *stream,
                                                            const void *x, const sd::LongType *xShape, int xRank,
                                                            void *extraParams,
                                                            void *z, const sd::LongType *zShape, int zRank,
                                                            int *allocationPointer, void *reductionPointer,
                                                            const sd::LongType *tadShapeInfo, const sd::LongType *tadOffsets) {

            transformStrictSimple<X, OpType><<<launchDims.x, launchDims.x, launchDims.z, *stream>>>(x, xShape, xRank, extraParams, z, zShape, zRank, allocationPointer, reductionPointer, tadShapeInfo, tadOffsets);
            sd::DebugHelper::checkErrorCode(stream, "transformStrict(...) failed");
        }

        BUILD_SINGLE_TEMPLATE(template class SD_LIB_HIDDEN TransformStrict, , SD_FLOAT_TYPES);
    }
}
