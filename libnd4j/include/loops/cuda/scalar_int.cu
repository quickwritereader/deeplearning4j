#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Yurii Shyrma (iuriish@yahoo.com), created on 08.11.2018
// @author raver119@gmail.com
//
#include "../scalar_int.h"
#include <system/op_boilerplate.h>
#include <types/types.h>
#include "../legacy_ops.h"

using namespace simdOps;

////////////////////////////////////////////////////////////////////////
template <typename X, typename OpType>
SD_KERNEL void scalarAlongDimension(void const* x, sd::LongType const* xShapeInfo,
                                    void *extraParams,
                                    void *z, sd::LongType const* zShapeInfo,
                                    void const* scalars,
                                    int *dimension, int dimensionLength,
                                    sd::LongType const* tadShapeInfo, sd::LongType const* tadOffsets,
                                    sd::LongType const* tadShapeInfoZ, sd::LongType const* tadOffsetsZ) {

    functions::scalar::ScalarIntTransform<X>::template transformCuda<OpType>(x, xShapeInfo, extraParams, z, zShapeInfo, scalars, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ);
}


////////////////////////////////////////////////////////////////////////
template <typename X, typename OpType>
SD_KERNEL void scalarSimpleShaped(void const* x, void const* y, sd::LongType const* xShapeInfo, void *params, void *z, sd::LongType const* zShapeInfo, int *allocationBuffer) {

    functions::scalar::ScalarIntTransform<X>::template transformCuda<OpType>(y, x, xShapeInfo, params, z, zShapeInfo, allocationBuffer);
}


// *********************************************************************//
// *********************************************************************//
namespace functions {
namespace scalar    {

////////////////////////////////////////////////////////////////////////
template<typename X>
template<typename OpType>
SD_DEVICE void  ScalarIntTransform<X>::transformCuda(void const* vscalar,
                                                        void const* vy, sd::LongType const* yShapeInfo,
                                                        void *vparams,
                                                        void *vz, sd::LongType const* zShapeInfo,
                                                        int *allocationBuffer) {
    auto scalar = reinterpret_cast<X const*>(vscalar)[0];
    auto y      = reinterpret_cast<X const*>(vy);
    auto params = reinterpret_cast<X*>(vparams);
    auto z      = reinterpret_cast<X*>(vz);

    auto yRank   = shape::rank(yShapeInfo);
    auto yEWS    = shape::elementWiseStride(yShapeInfo);
    auto yShape  = shape::shapeOf(yShapeInfo);
    auto yStride = shape::stride(yShapeInfo);

    auto zRank   = shape::rank(zShapeInfo);
    auto zEWS    = shape::elementWiseStride(zShapeInfo);
    auto zShape  = shape::shapeOf(zShapeInfo);
    auto zStride = shape::stride(zShapeInfo);

    int totalThreads = gridDim.x * blockDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ int len;
    if(threadIdx.x == 0)
        len = shape::length(yShapeInfo);
    __syncthreads();

    if(yEWS >= 1 && zEWS >= 1 && shape::order(yShapeInfo) == shape::order(zShapeInfo)) {
            transformCuda<OpType>(len, vscalar, vy, yEWS, vparams, vz, zEWS, allocationBuffer);
    }
    else {
        for (sd::LongType i = tid; i < len; i+= totalThreads)
            z[shape::getIndexOffset(i, zShapeInfo)] = OpType::op(y[shape::getIndexOffset(i, yShapeInfo)], scalar, params);
    }
}

////////////////////////////////////////////////////////////////////////
template<typename X>
template<typename OpType>
SD_DEVICE void  ScalarIntTransform<X>::transformCuda(sd::LongType len,
                                                          void const* vx,
                                                          void const* vy, sd::LongType yEWS,
                                                          void *vparams,
                                                          void *vz, sd::LongType zEWS,
                                                          int *allocationBuffer) {

    auto x = reinterpret_cast<X const*>(vx)[0];
    auto y = reinterpret_cast<X const*>(vy);
    auto z = reinterpret_cast<X*>(vz);
    auto params = reinterpret_cast<X*>(vparams);

    int totalThreads = gridDim.x * blockDim.x;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    sd::LongType i = tid;
    if(yEWS == 1 && zEWS == 1) {
        for (; i < len; i += totalThreads)
            z[i] = OpType::op(y[i], x, params);
    }
    else {
        for (; i < len; i += totalThreads)
            z[i * zEWS] = OpType::op(y[i * yEWS], x, params);
    }
}


////////////////////////////////////////////////////////////////////////
template<typename X>
template<typename OpType>
SD_DEVICE void  ScalarIntTransform<X>::transformCuda(void const* vx, sd::LongType const* xShapeInfo,
                                                        void *vextraParams,
                                                        void *vz, sd::LongType const* zShapeInfo,
                                                        void const* vscalars,
                                                        int *dimension, int dimensionLength,
                                                        sd::LongType const* tadShapeInfo, sd::LongType const* tadOffsets,
                                                        sd::LongType const* tadShapeInfoZ, sd::LongType const* tadOffsetsZ) {
    auto x = reinterpret_cast<X const*>(vx);
    auto scalars = reinterpret_cast<X const*>(vscalars);
    auto z = reinterpret_cast<X*>(vz);
    auto extraParams = reinterpret_cast<X*>(vextraParams);

    if (tadShapeInfoZ == nullptr) {
        tadShapeInfoZ = tadShapeInfo;
        tadOffsetsZ = tadOffsets;
    }

    // tad preparation
    auto tadEws = shape::elementWiseStride(tadShapeInfo);
    auto zEws = shape::elementWiseStride(tadShapeInfoZ);
    auto tadLength = shape::length(tadShapeInfo);//shape::tadLength(xShapeInfo, dimension, dimensionLength);
    auto numTads =shape::length(xShapeInfo) / tadLength;

    if (tadEws > 0 && zEws > 0 && shape::order(tadShapeInfo) == shape::order(zShapeInfo)) {

        // main loop, rolling over tads
        for (int r = blockIdx.x; r < numTads; r += gridDim.x) {
            X *oZ = z + tadOffsetsZ[r];
            auto oX = x + tadOffsets[r];

            auto s = scalars[r];

            for (int f = threadIdx.x; f < tadLength; f += blockDim.x)
                oZ[f * zEws] = OpType::op(oX[f * tadEws], s, extraParams);
        }
    } else {
        // main loop, rolling over tads
        for (int r = blockIdx.x; r < numTads; r += gridDim.x) {
            X *oZ = z + tadOffsetsZ[r];
            auto oX = x + tadOffsets[r];

            auto s = scalars[r];

            for (int f = threadIdx.x; f < tadLength; f += blockDim.x)
                oZ[shape::getIndexOffset(f, tadShapeInfoZ)] = OpType::op(oX[shape::getIndexOffset(f, tadShapeInfo)], s, extraParams);
        }
    }
}


////////////////////////////////////////////////////////////////////////
template<typename X>
template <typename OpType>
SD_HOST void ScalarIntTransform<X>::intermediateAlongDimension(dim3& launchDims, hipStream_t *stream,
                                                                void const* x, sd::LongType const* xShapeInfo,
                                                                void *z, sd::LongType const* zShapeInfo,
                                                                void const* scalars,
                                                                void *extraParams,
                                                                int *dimension, int dimensionLength,
                                                                sd::LongType const* tadShapeInfo, sd::LongType const* tadOffsets,
                                                                sd::LongType const* tadShapeInfoZ, sd::LongType const* tadOffsetsZ) {

    scalarAlongDimension<X, OpType><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(x, xShapeInfo, extraParams, z, zShapeInfo, scalars, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ);
}

////////////////////////////////////////////////////////////////////////
template<typename X>
template<typename OpType>
void SD_HOST ScalarIntTransform<X>::intermediateShaped(dim3& launchDims, hipStream_t *stream,
                                                            void const* vx, sd::LongType const* xShapeInfo,
                                                            void *vz, sd::LongType const* zShapeInfo,
                                                            void const* vscalar,
                                                            void *vextraParams, int *allocPointer){

    scalarSimpleShaped<X, OpType><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, vscalar, xShapeInfo, vextraParams, vz, zShapeInfo, allocPointer);
}

////////////////////////////////////////////////////////////////////////
template<typename X>
void ScalarIntTransform<X>::executeCudaShaped(dim3& launchDims, hipStream_t *stream,
                                                int opNum,
                                                void const* vx, sd::LongType const* xShapeInfo,
                                                void *vz, sd::LongType const* zShapeInfo,
                                                void const* vscalar,
                                                void* vextraParams) {

    if (sd::Environment::getInstance().isDebugAndVerbose())
        printf("H14 opNum:[%i]\n", opNum);

    DISPATCH_BY_OPNUM_T(intermediateShaped, PARAMS(launchDims, stream, vx, xShapeInfo, vz, zShapeInfo, vscalar, vextraParams, nullptr), SCALAR_INT_OPS);
}

////////////////////////////////////////////////////////////////////////
template<typename X>
void ScalarIntTransform<X>::executeCudaAlongDimension(dim3& launchDims, hipStream_t *stream, int opNum, void const* vx, sd::LongType const* xShapeInfo, void *vz, sd::LongType const* zShapeInfo, void const* vscalars, void *vextraParams, int *dimension, int dimensionLength, sd::LongType const* tadShapeInfo, sd::LongType const* tadOffsets, sd::LongType const* tadShapeInfoZ, sd::LongType const* tadOffsetsZ) {
    DISPATCH_BY_OPNUM_T(intermediateAlongDimension, PARAMS(launchDims, stream, vx, xShapeInfo, vz, zShapeInfo, vscalars, vextraParams, dimension, dimensionLength, tadShapeInfo, tadOffsets, tadShapeInfoZ, tadOffsetsZ), SCALAR_INT_OPS);
}

    BUILD_SINGLE_TEMPLATE(template class SD_LIB_HIDDEN ScalarIntTransform, , SD_INTEGER_TYPES);

}
}

