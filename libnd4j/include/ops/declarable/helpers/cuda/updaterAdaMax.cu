#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author Oleh Semeniv (oleg.semeniv@gmail.com)
//
#include <system/op_boilerplate.h>
#include <ops/declarable/helpers/updatersHelpers.h>
#include <helpers/PointersManager.h>
#include <math/platformmath.h>
#include <math/templatemath.h>

namespace sd    {
namespace ops     {
namespace helpers {

///////////////////////////////////////////////////////////////////
template<typename T>
SD_KERNEL void adaMaxUpdaterCuda(const void* vx, const sd::LongType* xShapeInfo, const void* vinv, const sd::LongType* invShapeInfo, 
    const void* vinm, const sd::LongType* inmShapeInfo,  void* vz, const sd::LongType* zShapeInfo, 
    void* vstV, const sd::LongType* stvShapeInfo, void* vstM, const sd::LongType* stmShapeInfo, 
    const T lr, const T beta1, const T beta2, const T epsilon, const T iteration) {

    const auto grad = reinterpret_cast<const T*>(vx);
    const auto initU = reinterpret_cast<const T*>(vinv);
    const auto initM = reinterpret_cast<const T*>(vinm);
    
    auto up = reinterpret_cast<T*>(vz);
    auto stU = reinterpret_cast<T*>(vstV);
    auto stM = reinterpret_cast<T*>(vstM);

    __shared__ sd::LongType xLen;
    __shared__ T beta1T, epsilonT;
    __shared__ bool bEWS, bOrdering, bXZsame, bXInUSame, bXStUSame, bXInMSame, bXStMSame;

    if (threadIdx.x == 0) {
        xLen = shape::length(xShapeInfo);
        beta1T = sd::math::sd_pow<T,T,T>(beta1, (iteration + 1) );

        epsilonT = lr / (1.0 - beta1T);
        if (sd::math::sd_isnan(epsilonT) || 0 == epsilonT || sd::math::sd_isinf(epsilonT))
            epsilonT = epsilon;

        bEWS =  1 == shape::elementWiseStride(xShapeInfo) && 1 == shape::elementWiseStride(zShapeInfo) &&
                1 == shape::elementWiseStride(stmShapeInfo) && 1 == shape::elementWiseStride(inmShapeInfo) &&
                1 == shape::elementWiseStride(stvShapeInfo) && 1 == shape::elementWiseStride(invShapeInfo);
        bOrdering = shape::order(xShapeInfo) == shape::order(zShapeInfo) && shape::order(xShapeInfo) == shape::order(stmShapeInfo) &&
                    shape::order(xShapeInfo) == shape::order(inmShapeInfo) && shape::order(xShapeInfo) == shape::order(invShapeInfo) &&
                    shape::order(xShapeInfo) == shape::order(stvShapeInfo);

        bXZsame = shape::haveSameShapeAndStrides(xShapeInfo, zShapeInfo);
        bXInUSame = shape::haveSameShapeAndStrides(xShapeInfo, invShapeInfo);
        bXStUSame = shape::haveSameShapeAndStrides(xShapeInfo, stvShapeInfo);
        bXInMSame = shape::haveSameShapeAndStrides(xShapeInfo, inmShapeInfo);
        bXStMSame = shape::haveSameShapeAndStrides(xShapeInfo, stmShapeInfo);
    }
    __syncthreads();

    int coords[SD_MAX_RANK];

    for (sd::LongType i = blockIdx.x * blockDim.x + threadIdx.x; i < xLen; i += gridDim.x * blockDim.x) {
        

        auto xOffset = i, zOffset = i, initMOffset = i, initUOffset = i, stMOffset = i, stUOffset = i;

        if (!bEWS || !bOrdering) {

            shape::index2coords(i, xShapeInfo, coords);
            xOffset  = shape::getOffset(xShapeInfo, coords);
            zOffset  = bXZsame ? xOffset : shape::getOffset(zShapeInfo, coords);
            initUOffset = bXInUSame ? xOffset : shape::getOffset(invShapeInfo, coords);
            stUOffset = bXStUSame ? xOffset : shape::getOffset(stvShapeInfo, coords);
            initMOffset = bXInMSame ? xOffset : shape::getOffset(inmShapeInfo, coords);
            stMOffset = bXStMSame ? xOffset : shape::getOffset(stmShapeInfo, coords);
        }

        //m = B_1 * m + (1-B_1)*grad
        stM[stMOffset] = beta1 * initM[initMOffset] + grad[xOffset] * (1 - beta1);
        //u = max(B_2 * u, |grad|)
        stU[stUOffset] = sd::math::sd_max( (beta2* initU[initUOffset]), sd::math::sd_abs(grad[xOffset])) + 1e-32;
        
        up[zOffset] = (stM[stMOffset] * epsilonT) / stU[stUOffset];
    }
}

///////////////////////////////////////////////////////////////////
template<typename T>
void adaMaxUpdaterCudaLauncher(const int blocksPerGrid, const int threadsPerBlock, const hipStream_t* stream, const void* vx, const sd::LongType* xShapeInfo, 
                                       const void* vinv, const sd::LongType* invShapeInfo, const void* vinm, const sd::LongType* inmShapeInfo, 
                                        void* vz, const sd::LongType* zShapeInfo, void* vstV, const sd::LongType* stvShapeInfo, 
                                        void* vstM, const sd::LongType* stmShapeInfo, const double dLr, 
                                        const double dBeta1, const double dBeta2, const double dEpsilon, const int nIteration) {
    
    const T lr = static_cast<T>(dLr);
    const T beta1 = static_cast<T>(dBeta1);
    const T beta2 = static_cast<T>(dBeta2);
    const T epsilon = static_cast<T>(dEpsilon);
    const T iteration = static_cast<T>(nIteration);

    adaMaxUpdaterCuda<T><<<blocksPerGrid, threadsPerBlock, 256, * stream>>>(vx, xShapeInfo, vinv, invShapeInfo, vinm, inmShapeInfo, vz,
         zShapeInfo, vstV, stvShapeInfo, vstM, stmShapeInfo, lr, beta1, beta2, epsilon, iteration);
}

///////////////////////////////////////////////////////////////////
void updaterAdaMax(sd::LaunchContext* context, const NDArray& gradient, const NDArray& initStateU, const NDArray& initStateM,
                   NDArray& update, NDArray& stateU, NDArray& stateM, const double dLr, const double dBeta1, 
                   const double dBeta2, const double dEpsilon, const int nIteration) {

    PointersManager manager(context, "adaMaxUpdater");

    const int threadsPerBlock = SD_MAX_NUM_THREADS / 4;
    const int blocksPerGrid = (gradient.lengthOf() + threadsPerBlock - 1) / threadsPerBlock;

    NDArray::prepareSpecialUse({ &update, &stateU, &stateM }, { &gradient, &initStateU, &initStateM });
    BUILD_SINGLE_SELECTOR(gradient.dataType(), adaMaxUpdaterCudaLauncher, (blocksPerGrid, threadsPerBlock, context->getCudaStream(), 
                          gradient.specialBuffer(), gradient.specialShapeInfo(), initStateU.specialBuffer(),
                          initStateU.specialShapeInfo(), initStateM.specialBuffer(), initStateM.specialShapeInfo(),
                          update.specialBuffer(), update.specialShapeInfo(), stateU.specialBuffer(),
                          stateU.specialShapeInfo(), stateM.specialBuffer(), stateM.specialShapeInfo(),
                          dLr, dBeta1, dBeta2, dEpsilon, nIteration ), SD_FLOAT_TYPES);
    NDArray::registerSpecialUse({ &update, &stateU, &stateM }, { &gradient, &initStateU, &initStateM });

    manager.synchronize();
}

}
}
}
