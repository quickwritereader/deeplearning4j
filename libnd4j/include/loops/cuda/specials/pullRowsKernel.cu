/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
// @author Yurii Shyrma, created on 15.11.2018
//
#include <loops/special_kernels.h>

namespace sd {

///////////////////////////////////////////////////////////////////////
    template<typename T>
    SD_DEVICE void pullRowsKernel(void *vx,
                                   void *vz,
                                   sd::LongType len,
                                   sd::LongType *indexes,
                                   sd::LongType const* tadShapeInfo, sd::LongType const* tadOffsets,
                                   sd::LongType const* zTadShapeInfo, sd::LongType const* zTadOffsets) {

        auto x = reinterpret_cast<T *>(vx);
        auto z = reinterpret_cast<T *>(vz);
        auto xEWS = shape::elementWiseStride(tadShapeInfo);
        auto zEWS = shape::elementWiseStride(zTadShapeInfo);
        auto tadLength = shape::length(tadShapeInfo);

        if (xEWS >= 1 && zEWS >= 1) {
            for (int idx = blockIdx.x; idx < len; idx += gridDim.x) {
                T *rX = x + tadOffsets[indexes[idx]];
                T *rZ = z + zTadOffsets[idx];

                for (int i = threadIdx.x; i < tadLength; i += blockDim.x) {
                    rZ[i * zEWS] = rX[i * xEWS];
                }
            }
        } else {
            for (int idx = blockIdx.x; idx < len; idx += gridDim.x) {
                T *rX = x + tadOffsets[indexes[idx]];
                T *rZ = z + zTadOffsets[idx];

                for (int i = threadIdx.x; i < tadLength; i += blockDim.x) {
                    auto xOffset = shape::getIndexOffset(i, tadShapeInfo);
                    auto zOffset = shape::getIndexOffset(i, zTadShapeInfo);
                    rZ[zOffset] = rX[xOffset];
                }
            }
        }
    }

///////////////////////////////////////////////////////////////////////
    template<typename T>
    SD_KERNEL void execPullRowsKernel(void *vx,
                                       void *vz,
                                       sd::LongType len,
                                       sd::LongType *indexes,
                                       sd::LongType const* tadShapeInfo, sd::LongType const* tadOffsets,
                                       sd::LongType const* zTadShapeInfo, sd::LongType const* zTadOffsets) {

        pullRowsKernel<T>(vx, vz, len, indexes, tadShapeInfo, tadOffsets, zTadShapeInfo, zTadOffsets);
    }

///////////////////////////////////////////////////////////////////////
    template<typename T>
    SD_HOST void pullRowsKernelGeneric(dim3 &launchDims, hipStream_t *stream,
                                        void *vx,
                                        void *vz,
                                        sd::LongType len,
                                        sd::LongType *indexes,
                                        sd::LongType const* tadShapeInfo, sd::LongType const* tadOffsets,
                                        sd::LongType const* zTadShapeInfo, sd::LongType const* zTadOffsets) {

        execPullRowsKernel<T><<<launchDims.x, launchDims.y, launchDims.z, *stream>>>(vx, vz, len, indexes, tadShapeInfo, tadOffsets, zTadShapeInfo, zTadOffsets);
        sd::DebugHelper::checkErrorCode(stream, "pullRows(...) failed");
    }

    BUILD_SINGLE_TEMPLATE(template void SD_LIB_HIDDEN pullRowsKernelGeneric, (dim3 & launchDims, hipStream_t * stream, void * vx, void * vz, sd::LongType len, sd::LongType * indexes, sd::LongType const* tadShapeInfo, sd::LongType const* tadOffsets, sd::LongType const* zTadShapeInfo, sd::LongType const* zTadOffsets), SD_COMMON_TYPES);
}

